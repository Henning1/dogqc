#include "hip/hip_runtime.h"
#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
__global__ void krnl_lineitem1(
    int* iatt6_lquantit, float* iatt7_lextende, float* iatt8_ldiscoun, unsigned* iatt12_lshipdat, float* agg1) {
    int att6_lquantit;
    float att7_lextende;
    float att8_ldiscoun;
    unsigned att12_lshipdat;
    float att18_rev;

    int tid_lineitem1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_lineitem1 = loopVar;
        active = (loopVar < 6001215);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att6_lquantit = iatt6_lquantit[tid_lineitem1];
            att7_lextende = iatt7_lextende[tid_lineitem1];
            att8_ldiscoun = iatt8_ldiscoun[tid_lineitem1];
            att12_lshipdat = iatt12_lshipdat[tid_lineitem1];
        }
        // -------- selection (opId: 2) --------
        if(active) {
            active = ((att12_lshipdat >= 19940101) && ((att12_lshipdat < 19950101) && ((att8_ldiscoun >= 0.05) && ((att8_ldiscoun <= 0.07) && (att6_lquantit < 24)))));
        }
        // -------- map (opId: 3) --------
        if(active) {
            att18_rev = (att7_lextende * att8_ldiscoun);
        }
        // -------- aggregation (opId: 4) --------
        int bucket = 0;
        if(active) {
            atomicAdd(&(agg1[bucket]), ((float)att18_rev));
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation4(
    float* agg1, int* nout_result, float* oatt1_revenue) {
    float att1_revenue;
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_aggregation4 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation4 = loopVar;
        active = (loopVar < 1);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 4) --------
        if(active) {
            att1_revenue = agg1[tid_aggregation4];
        }
        // -------- materialize (opId: 5) --------
        int wp;
        int writeMask;
        int numProj;
        writeMask = __ballot_sync(ALL_LANES,active);
        numProj = __popc(writeMask);
        if((warplane == 0)) {
            wp = atomicAdd(nout_result, numProj);
        }
        wp = __shfl_sync(ALL_LANES,wp,0);
        wp = (wp + __popc((writeMask & prefixlanes)));
        if(active) {
            oatt1_revenue[wp] = att1_revenue;
        }
        loopVar += step;
    }

}

int main() {
    int* iatt6_lquantit;
    iatt6_lquantit = ( int*) map_memory_file ( "mmdb/lineitem_l_quantity" );
    float* iatt7_lextende;
    iatt7_lextende = ( float*) map_memory_file ( "mmdb/lineitem_l_extendedprice" );
    float* iatt8_ldiscoun;
    iatt8_ldiscoun = ( float*) map_memory_file ( "mmdb/lineitem_l_discount" );
    unsigned* iatt12_lshipdat;
    iatt12_lshipdat = ( unsigned*) map_memory_file ( "mmdb/lineitem_l_shipdate" );

    int nout_result;
    std::vector < float > oatt1_revenue(1);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt6_lquantit;
    hipMalloc((void**) &d_iatt6_lquantit, 6001215* sizeof(int) );
    float* d_iatt7_lextende;
    hipMalloc((void**) &d_iatt7_lextende, 6001215* sizeof(float) );
    float* d_iatt8_ldiscoun;
    hipMalloc((void**) &d_iatt8_ldiscoun, 6001215* sizeof(float) );
    unsigned* d_iatt12_lshipdat;
    hipMalloc((void**) &d_iatt12_lshipdat, 6001215* sizeof(unsigned) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    float* d_oatt1_revenue;
    hipMalloc((void**) &d_oatt1_revenue, 1* sizeof(float) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    float* d_agg1;
    hipMalloc((void**) &d_agg1, 1* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg1, 0.0f, 1);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt6_lquantit, iatt6_lquantit, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt7_lextende, iatt7_lextende, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt8_ldiscoun, iatt8_ldiscoun, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt12_lshipdat, iatt12_lshipdat, 6001215 * sizeof(unsigned), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime0 = std::clock();
    std::clock_t start_krnl_lineitem11 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_lineitem1<<<gridsize, blocksize>>>(d_iatt6_lquantit, d_iatt7_lextende, d_iatt8_ldiscoun, d_iatt12_lshipdat, d_agg1);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_lineitem11 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem1")
        }
    }

    std::clock_t start_krnl_aggregation42 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation4<<<gridsize, blocksize>>>(d_agg1, d_nout_result, d_oatt1_revenue);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation42 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation4! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation4")
        }
    }

    std::clock_t stop_totalKernelTime0 = std::clock();
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt1_revenue.data(), d_oatt1_revenue, 1 * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt6_lquantit);
    hipFree( d_iatt7_lextende);
    hipFree( d_iatt8_ldiscoun);
    hipFree( d_iatt12_lshipdat);
    hipFree( d_agg1);
    hipFree( d_nout_result);
    hipFree( d_oatt1_revenue);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish3 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 1)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("revenue: ");
        printf("%15.2f", oatt1_revenue[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish3 = std::clock();

    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "krnl_lineitem1", (stop_krnl_lineitem11 - start_krnl_lineitem11) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation4", (stop_krnl_aggregation42 - start_krnl_aggregation42) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish3 - start_finish3) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime0 - start_totalKernelTime0) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
