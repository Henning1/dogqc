#include "hip/hip_runtime.h"
#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct jpayl3 {
    int att2_ppartkey;
    str_t att5_pbrand;
    int att7_psize;
    str_t att8_pcontain;
};

__global__ void krnl_part1(
    int* iatt2_ppartkey, size_t* iatt5_pbrand_offset, char* iatt5_pbrand_char, int* iatt7_psize, size_t* iatt8_pcontain_offset, char* iatt8_pcontain_char, unique_ht<jpayl3>* jht3) {
    int att2_ppartkey;
    str_t att5_pbrand;
    int att7_psize;
    str_t att8_pcontain;

    int tid_part1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_part1 = loopVar;
        active = (loopVar < 200000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att2_ppartkey = iatt2_ppartkey[tid_part1];
            att5_pbrand = stringScan ( iatt5_pbrand_offset, iatt5_pbrand_char, tid_part1);
            att7_psize = iatt7_psize[tid_part1];
            att8_pcontain = stringScan ( iatt8_pcontain_offset, iatt8_pcontain_char, tid_part1);
        }
        // -------- hash join build (opId: 3) --------
        if(active) {
            jpayl3 payl3;
            payl3.att2_ppartkey = att2_ppartkey;
            payl3.att5_pbrand = att5_pbrand;
            payl3.att7_psize = att7_psize;
            payl3.att8_pcontain = att8_pcontain;
            uint64_t hash3;
            hash3 = 0;
            if(active) {
                hash3 = hash ( (hash3 + ((uint64_t)att2_ppartkey)));
            }
            hashBuildUnique ( jht3, 400000, hash3, &(payl3));
        }
        loopVar += step;
    }

}

__global__ void krnl_lineitem2(
    int* iatt12_lpartkey, int* iatt15_lquantit, float* iatt16_lextende, float* iatt17_ldiscoun, size_t* iatt24_lshipins_offset, char* iatt24_lshipins_char, size_t* iatt25_lshipmod_offset, char* iatt25_lshipmod_char, unique_ht<jpayl3>* jht3, float* agg1) {
    int att12_lpartkey;
    int att15_lquantit;
    float att16_lextende;
    float att17_ldiscoun;
    str_t att24_lshipins;
    str_t att25_lshipmod;
    int att2_ppartkey;
    str_t att5_pbrand;
    int att7_psize;
    str_t att8_pcontain;
    str_t c1 = stringConstant ( "Brand#12", 8);
    str_t c2 = stringConstant ( "SM PKG", 6);
    str_t c3 = stringConstant ( "SM PACK", 7);
    str_t c4 = stringConstant ( "SM BOX", 6);
    str_t c5 = stringConstant ( "SM CASE", 7);
    str_t c6 = stringConstant ( "AIR REG", 7);
    str_t c7 = stringConstant ( "AIR", 3);
    str_t c8 = stringConstant ( "DELIVER IN PERSON", 17);
    str_t c9 = stringConstant ( "Brand#23", 8);
    str_t c10 = stringConstant ( "MED PACK", 8);
    str_t c11 = stringConstant ( "MED PKG", 7);
    str_t c12 = stringConstant ( "MED BOX", 7);
    str_t c13 = stringConstant ( "MED BAG", 7);
    str_t c14 = stringConstant ( "AIR REG", 7);
    str_t c15 = stringConstant ( "AIR", 3);
    str_t c16 = stringConstant ( "DELIVER IN PERSON", 17);
    str_t c17 = stringConstant ( "Brand#34", 8);
    str_t c18 = stringConstant ( "LG PKG", 6);
    str_t c19 = stringConstant ( "LG PACK", 7);
    str_t c20 = stringConstant ( "LG BOX", 6);
    str_t c21 = stringConstant ( "LG CASE", 7);
    str_t c22 = stringConstant ( "AIR REG", 7);
    str_t c23 = stringConstant ( "AIR", 3);
    str_t c24 = stringConstant ( "DELIVER IN PERSON", 17);
    float att27_rev;

    int tid_lineitem1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_lineitem1 = loopVar;
        active = (loopVar < 6001215);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att12_lpartkey = iatt12_lpartkey[tid_lineitem1];
            att15_lquantit = iatt15_lquantit[tid_lineitem1];
            att16_lextende = iatt16_lextende[tid_lineitem1];
            att17_ldiscoun = iatt17_ldiscoun[tid_lineitem1];
            att24_lshipins = stringScan ( iatt24_lshipins_offset, iatt24_lshipins_char, tid_lineitem1);
            att25_lshipmod = stringScan ( iatt25_lshipmod_offset, iatt25_lshipmod_char, tid_lineitem1);
        }
        // -------- hash join probe (opId: 3) --------
        uint64_t hash3 = 0;
        if(active) {
            hash3 = 0;
            if(active) {
                hash3 = hash ( (hash3 + ((uint64_t)att12_lpartkey)));
            }
        }
        jpayl3* probepayl3;
        int numLookups3 = 0;
        if(active) {
            active = hashProbeUnique ( jht3, 400000, hash3, numLookups3, &(probepayl3));
        }
        int bucketFound3 = 0;
        int probeActive3 = active;
        while((probeActive3 && !(bucketFound3))) {
            jpayl3 jprobepayl3 = *(probepayl3);
            att2_ppartkey = jprobepayl3.att2_ppartkey;
            att5_pbrand = jprobepayl3.att5_pbrand;
            att7_psize = jprobepayl3.att7_psize;
            att8_pcontain = jprobepayl3.att8_pcontain;
            bucketFound3 = 1;
            bucketFound3 &= ((att2_ppartkey == att12_lpartkey));
            if(!(bucketFound3)) {
                probeActive3 = hashProbeUnique ( jht3, 400000, hash3, numLookups3, &(probepayl3));
            }
        }
        active = bucketFound3;
        // -------- selection (opId: 4) --------
        if(active) {
            active = (((((((((stringEquals ( att5_pbrand, c1) && (stringEquals ( att8_pcontain, c2) || (stringEquals ( att8_pcontain, c3) || (stringEquals ( att8_pcontain, c4) || stringEquals ( att8_pcontain, c5))))) && (att15_lquantit >= 1.0f)) && (att15_lquantit <= 11.0f)) && (att7_psize >= 1)) && (att7_psize <= 5)) && (stringEquals ( att25_lshipmod, c6) || stringEquals ( att25_lshipmod, c7))) && stringEquals ( att24_lshipins, c8)) || (((((((stringEquals ( att5_pbrand, c9) && (stringEquals ( att8_pcontain, c10) || (stringEquals ( att8_pcontain, c11) || (stringEquals ( att8_pcontain, c12) || stringEquals ( att8_pcontain, c13))))) && (att15_lquantit >= 10.0f)) && (att15_lquantit <= 20.0f)) && (att7_psize >= 1)) && (att7_psize <= 10)) && (stringEquals ( att25_lshipmod, c14) || stringEquals ( att25_lshipmod, c15))) && stringEquals ( att24_lshipins, c16))) || (((((((stringEquals ( att5_pbrand, c17) && (stringEquals ( att8_pcontain, c18) || (stringEquals ( att8_pcontain, c19) || (stringEquals ( att8_pcontain, c20) || stringEquals ( att8_pcontain, c21))))) && (att15_lquantit >= 20.0f)) && (att15_lquantit <= 30.0f)) && (att7_psize >= 1)) && (att7_psize <= 15)) && (stringEquals ( att25_lshipmod, c22) || stringEquals ( att25_lshipmod, c23))) && stringEquals ( att24_lshipins, c24)));
        }
        // -------- map (opId: 5) --------
        if(active) {
            att27_rev = (att16_lextende * (1.0 - att17_ldiscoun));
        }
        // -------- aggregation (opId: 6) --------
        int bucket = 0;
        if(active) {
            atomicAdd(&(agg1[bucket]), ((float)att27_rev));
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation6(
    float* agg1, int* nout_result, float* oatt1_revenue) {
    float att1_revenue;
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_aggregation6 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation6 = loopVar;
        active = (loopVar < 1);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 6) --------
        if(active) {
            att1_revenue = agg1[tid_aggregation6];
        }
        // -------- materialize (opId: 7) --------
        int wp;
        int writeMask;
        int numProj;
        writeMask = __ballot_sync(ALL_LANES,active);
        numProj = __popc(writeMask);
        if((warplane == 0)) {
            wp = atomicAdd(nout_result, numProj);
        }
        wp = __shfl_sync(ALL_LANES,wp,0);
        wp = (wp + __popc((writeMask & prefixlanes)));
        if(active) {
            oatt1_revenue[wp] = att1_revenue;
        }
        loopVar += step;
    }

}

int main() {
    int* iatt2_ppartkey;
    iatt2_ppartkey = ( int*) map_memory_file ( "mmdb/part_p_partkey" );
    size_t* iatt5_pbrand_offset;
    iatt5_pbrand_offset = ( size_t*) map_memory_file ( "mmdb/part_p_brand_offset" );
    char* iatt5_pbrand_char;
    iatt5_pbrand_char = ( char*) map_memory_file ( "mmdb/part_p_brand_char" );
    int* iatt7_psize;
    iatt7_psize = ( int*) map_memory_file ( "mmdb/part_p_size" );
    size_t* iatt8_pcontain_offset;
    iatt8_pcontain_offset = ( size_t*) map_memory_file ( "mmdb/part_p_container_offset" );
    char* iatt8_pcontain_char;
    iatt8_pcontain_char = ( char*) map_memory_file ( "mmdb/part_p_container_char" );
    int* iatt12_lpartkey;
    iatt12_lpartkey = ( int*) map_memory_file ( "mmdb/lineitem_l_partkey" );
    int* iatt15_lquantit;
    iatt15_lquantit = ( int*) map_memory_file ( "mmdb/lineitem_l_quantity" );
    float* iatt16_lextende;
    iatt16_lextende = ( float*) map_memory_file ( "mmdb/lineitem_l_extendedprice" );
    float* iatt17_ldiscoun;
    iatt17_ldiscoun = ( float*) map_memory_file ( "mmdb/lineitem_l_discount" );
    size_t* iatt24_lshipins_offset;
    iatt24_lshipins_offset = ( size_t*) map_memory_file ( "mmdb/lineitem_l_shipinstruct_offset" );
    char* iatt24_lshipins_char;
    iatt24_lshipins_char = ( char*) map_memory_file ( "mmdb/lineitem_l_shipinstruct_char" );
    size_t* iatt25_lshipmod_offset;
    iatt25_lshipmod_offset = ( size_t*) map_memory_file ( "mmdb/lineitem_l_shipmode_offset" );
    char* iatt25_lshipmod_char;
    iatt25_lshipmod_char = ( char*) map_memory_file ( "mmdb/lineitem_l_shipmode_char" );

    int nout_result;
    std::vector < float > oatt1_revenue(1);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt2_ppartkey;
    hipMalloc((void**) &d_iatt2_ppartkey, 200000* sizeof(int) );
    size_t* d_iatt5_pbrand_offset;
    hipMalloc((void**) &d_iatt5_pbrand_offset, (200000 + 1)* sizeof(size_t) );
    char* d_iatt5_pbrand_char;
    hipMalloc((void**) &d_iatt5_pbrand_char, 1600009* sizeof(char) );
    int* d_iatt7_psize;
    hipMalloc((void**) &d_iatt7_psize, 200000* sizeof(int) );
    size_t* d_iatt8_pcontain_offset;
    hipMalloc((void**) &d_iatt8_pcontain_offset, (200000 + 1)* sizeof(size_t) );
    char* d_iatt8_pcontain_char;
    hipMalloc((void**) &d_iatt8_pcontain_char, 1514980* sizeof(char) );
    int* d_iatt12_lpartkey;
    hipMalloc((void**) &d_iatt12_lpartkey, 6001215* sizeof(int) );
    int* d_iatt15_lquantit;
    hipMalloc((void**) &d_iatt15_lquantit, 6001215* sizeof(int) );
    float* d_iatt16_lextende;
    hipMalloc((void**) &d_iatt16_lextende, 6001215* sizeof(float) );
    float* d_iatt17_ldiscoun;
    hipMalloc((void**) &d_iatt17_ldiscoun, 6001215* sizeof(float) );
    size_t* d_iatt24_lshipins_offset;
    hipMalloc((void**) &d_iatt24_lshipins_offset, (6001215 + 1)* sizeof(size_t) );
    char* d_iatt24_lshipins_char;
    hipMalloc((void**) &d_iatt24_lshipins_char, 72006418* sizeof(char) );
    size_t* d_iatt25_lshipmod_offset;
    hipMalloc((void**) &d_iatt25_lshipmod_offset, (6001215 + 1)* sizeof(size_t) );
    char* d_iatt25_lshipmod_char;
    hipMalloc((void**) &d_iatt25_lshipmod_char, 25717043* sizeof(char) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    float* d_oatt1_revenue;
    hipMalloc((void**) &d_oatt1_revenue, 1* sizeof(float) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    unique_ht<jpayl3>* d_jht3;
    hipMalloc((void**) &d_jht3, 400000* sizeof(unique_ht<jpayl3>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht3, 400000);
    }
    float* d_agg1;
    hipMalloc((void**) &d_agg1, 1* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg1, 0.0f, 1);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt2_ppartkey, iatt2_ppartkey, 200000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt5_pbrand_offset, iatt5_pbrand_offset, (200000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt5_pbrand_char, iatt5_pbrand_char, 1600009 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt7_psize, iatt7_psize, 200000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt8_pcontain_offset, iatt8_pcontain_offset, (200000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt8_pcontain_char, iatt8_pcontain_char, 1514980 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt12_lpartkey, iatt12_lpartkey, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt15_lquantit, iatt15_lquantit, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt16_lextende, iatt16_lextende, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt17_ldiscoun, iatt17_ldiscoun, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt24_lshipins_offset, iatt24_lshipins_offset, (6001215 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt24_lshipins_char, iatt24_lshipins_char, 72006418 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt25_lshipmod_offset, iatt25_lshipmod_offset, (6001215 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt25_lshipmod_char, iatt25_lshipmod_char, 25717043 * sizeof(char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime161 = std::clock();
    std::clock_t start_krnl_part1162 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_part1<<<gridsize, blocksize>>>(d_iatt2_ppartkey, d_iatt5_pbrand_offset, d_iatt5_pbrand_char, d_iatt7_psize, d_iatt8_pcontain_offset, d_iatt8_pcontain_char, d_jht3);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_part1162 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_part1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_part1")
        }
    }

    std::clock_t start_krnl_lineitem2163 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_lineitem2<<<gridsize, blocksize>>>(d_iatt12_lpartkey, d_iatt15_lquantit, d_iatt16_lextende, d_iatt17_ldiscoun, d_iatt24_lshipins_offset, d_iatt24_lshipins_char, d_iatt25_lshipmod_offset, d_iatt25_lshipmod_char, d_jht3, d_agg1);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_lineitem2163 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem2! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem2")
        }
    }

    std::clock_t start_krnl_aggregation6164 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation6<<<gridsize, blocksize>>>(d_agg1, d_nout_result, d_oatt1_revenue);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation6164 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation6! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation6")
        }
    }

    std::clock_t stop_totalKernelTime161 = std::clock();
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt1_revenue.data(), d_oatt1_revenue, 1 * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt2_ppartkey);
    hipFree( d_iatt5_pbrand_offset);
    hipFree( d_iatt5_pbrand_char);
    hipFree( d_iatt7_psize);
    hipFree( d_iatt8_pcontain_offset);
    hipFree( d_iatt8_pcontain_char);
    hipFree( d_jht3);
    hipFree( d_iatt12_lpartkey);
    hipFree( d_iatt15_lquantit);
    hipFree( d_iatt16_lextende);
    hipFree( d_iatt17_ldiscoun);
    hipFree( d_iatt24_lshipins_offset);
    hipFree( d_iatt24_lshipins_char);
    hipFree( d_iatt25_lshipmod_offset);
    hipFree( d_iatt25_lshipmod_char);
    hipFree( d_agg1);
    hipFree( d_nout_result);
    hipFree( d_oatt1_revenue);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish165 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 1)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("revenue: ");
        printf("%15.2f", oatt1_revenue[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish165 = std::clock();

    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "krnl_part1", (stop_krnl_part1162 - start_krnl_part1162) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_lineitem2", (stop_krnl_lineitem2163 - start_krnl_lineitem2163) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation6", (stop_krnl_aggregation6164 - start_krnl_aggregation6164) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish165 - start_finish165) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime161 - start_totalKernelTime161) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
