#include "hip/hip_runtime.h"
#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct jpayl12 {
    int att4_ssuppkey;
    int att7_snationk;
};
struct jpayl6 {
    int att11_nnationk;
    str_t att12_nname;
    int att15_nnationk;
    str_t att16_nname;
};
struct jpayl8 {
    int att11_nnationk;
    str_t att12_nname;
    str_t att16_nname;
    int att19_ccustkey;
};
struct jpayl11 {
    int att11_nnationk;
    str_t att12_nname;
    str_t att16_nname;
    int att27_oorderke;
};
struct apayl15 {
    str_t att12_nname;
    str_t att16_nname;
    unsigned att53_lyear;
};

__global__ void krnl_supplier1(
    int* iatt4_ssuppkey, int* iatt7_snationk, unique_ht<jpayl12>* jht12) {
    int att4_ssuppkey;
    int att7_snationk;

    int tid_supplier1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_supplier1 = loopVar;
        active = (loopVar < 10000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att4_ssuppkey = iatt4_ssuppkey[tid_supplier1];
            att7_snationk = iatt7_snationk[tid_supplier1];
        }
        // -------- hash join build (opId: 12) --------
        if(active) {
            jpayl12 payl12;
            payl12.att4_ssuppkey = att4_ssuppkey;
            payl12.att7_snationk = att7_snationk;
            uint64_t hash12;
            hash12 = 0;
            if(active) {
                hash12 = hash ( (hash12 + ((uint64_t)att7_snationk)));
            }
            if(active) {
                hash12 = hash ( (hash12 + ((uint64_t)att4_ssuppkey)));
            }
            hashBuildUnique ( jht12, 20000, hash12, &(payl12));
        }
        loopVar += step;
    }

}

__global__ void krnl_nation2(
    int* iatt11_nnationk, size_t* iatt12_nname_offset, char* iatt12_nname_char, int* nout_inner4, int* itm_inner4_n_nationkey, str_t* itm_inner4_n_name) {
    int att11_nnationk;
    str_t att12_nname;
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_nation1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation1 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att11_nnationk = iatt11_nnationk[tid_nation1];
            att12_nname = stringScan ( iatt12_nname_offset, iatt12_nname_char, tid_nation1);
        }
        // -------- nested join: materialize inner  (opId: 4) --------
        int wp;
        int writeMask;
        int numProj;
        writeMask = __ballot_sync(ALL_LANES,active);
        numProj = __popc(writeMask);
        if((warplane == 0)) {
            wp = atomicAdd(nout_inner4, numProj);
        }
        wp = __shfl_sync(ALL_LANES,wp,0);
        wp = (wp + __popc((writeMask & prefixlanes)));
        if(active) {
            itm_inner4_n_nationkey[wp] = att11_nnationk;
            itm_inner4_n_name[wp] = att12_nname;
        }
        loopVar += step;
    }

}

__global__ void krnl_nation23(
    int* iatt15_nnationk, size_t* iatt16_nname_offset, char* iatt16_nname_char, int* nout_inner4, int* itm_inner4_n_nationkey, str_t* itm_inner4_n_name, unique_ht<jpayl6>* jht6) {
    int att15_nnationk;
    str_t att16_nname;
    int att11_nnationk;
    str_t att12_nname;
    str_t c1 = stringConstant ( "GERMANY", 7);
    str_t c2 = stringConstant ( "FRANCE", 6);
    str_t c3 = stringConstant ( "FRANCE", 6);
    str_t c4 = stringConstant ( "GERMANY", 7);

    int tid_nation2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation2 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att15_nnationk = iatt15_nnationk[tid_nation2];
            att16_nname = stringScan ( iatt16_nname_offset, iatt16_nname_char, tid_nation2);
        }
        // -------- nested join: loop inner  (opId: 4) --------
        int outerActive4 = active;
        for ( int tid_inner40 = 0; (tid_inner40 < *(nout_inner4)); (tid_inner40++)) {
            active = outerActive4;
            if(active) {
                att11_nnationk = itm_inner4_n_nationkey[tid_inner40];
                att12_nname = itm_inner4_n_name[tid_inner40];
            }
            if(active) {
                active = ((stringEquals ( att12_nname, c1) && stringEquals ( att16_nname, c2)) || (stringEquals ( att12_nname, c3) && stringEquals ( att16_nname, c4)));
            }
            // -------- hash join build (opId: 6) --------
            if(active) {
                jpayl6 payl6;
                payl6.att11_nnationk = att11_nnationk;
                payl6.att12_nname = att12_nname;
                payl6.att15_nnationk = att15_nnationk;
                payl6.att16_nname = att16_nname;
                uint64_t hash6;
                hash6 = 0;
                if(active) {
                    hash6 = hash ( (hash6 + ((uint64_t)att15_nnationk)));
                }
                hashBuildUnique ( jht6, 1250, hash6, &(payl6));
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_customer5(
    int* iatt19_ccustkey, int* iatt22_cnationk, unique_ht<jpayl6>* jht6, unique_ht<jpayl8>* jht8) {
    int att19_ccustkey;
    int att22_cnationk;
    int att11_nnationk;
    str_t att12_nname;
    int att15_nnationk;
    str_t att16_nname;

    int tid_customer1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_customer1 = loopVar;
        active = (loopVar < 150000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att19_ccustkey = iatt19_ccustkey[tid_customer1];
            att22_cnationk = iatt22_cnationk[tid_customer1];
        }
        // -------- hash join probe (opId: 6) --------
        uint64_t hash6 = 0;
        if(active) {
            hash6 = 0;
            if(active) {
                hash6 = hash ( (hash6 + ((uint64_t)att22_cnationk)));
            }
        }
        jpayl6* probepayl6;
        int numLookups6 = 0;
        if(active) {
            active = hashProbeUnique ( jht6, 1250, hash6, numLookups6, &(probepayl6));
        }
        int bucketFound6 = 0;
        int probeActive6 = active;
        while((probeActive6 && !(bucketFound6))) {
            jpayl6 jprobepayl6 = *(probepayl6);
            att11_nnationk = jprobepayl6.att11_nnationk;
            att12_nname = jprobepayl6.att12_nname;
            att15_nnationk = jprobepayl6.att15_nnationk;
            att16_nname = jprobepayl6.att16_nname;
            bucketFound6 = 1;
            bucketFound6 &= ((att15_nnationk == att22_cnationk));
            if(!(bucketFound6)) {
                probeActive6 = hashProbeUnique ( jht6, 1250, hash6, numLookups6, &(probepayl6));
            }
        }
        active = bucketFound6;
        // -------- hash join build (opId: 8) --------
        if(active) {
            jpayl8 payl8;
            payl8.att11_nnationk = att11_nnationk;
            payl8.att12_nname = att12_nname;
            payl8.att16_nname = att16_nname;
            payl8.att19_ccustkey = att19_ccustkey;
            uint64_t hash8;
            hash8 = 0;
            if(active) {
                hash8 = hash ( (hash8 + ((uint64_t)att19_ccustkey)));
            }
            hashBuildUnique ( jht8, 30000, hash8, &(payl8));
        }
        loopVar += step;
    }

}

__global__ void krnl_orders7(
    int* iatt27_oorderke, int* iatt28_ocustkey, unique_ht<jpayl8>* jht8, unique_ht<jpayl11>* jht11) {
    int att27_oorderke;
    int att28_ocustkey;
    int att11_nnationk;
    str_t att12_nname;
    str_t att16_nname;
    int att19_ccustkey;

    int tid_orders1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_orders1 = loopVar;
        active = (loopVar < 1500000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att27_oorderke = iatt27_oorderke[tid_orders1];
            att28_ocustkey = iatt28_ocustkey[tid_orders1];
        }
        // -------- hash join probe (opId: 8) --------
        uint64_t hash8 = 0;
        if(active) {
            hash8 = 0;
            if(active) {
                hash8 = hash ( (hash8 + ((uint64_t)att28_ocustkey)));
            }
        }
        jpayl8* probepayl8;
        int numLookups8 = 0;
        if(active) {
            active = hashProbeUnique ( jht8, 30000, hash8, numLookups8, &(probepayl8));
        }
        int bucketFound8 = 0;
        int probeActive8 = active;
        while((probeActive8 && !(bucketFound8))) {
            jpayl8 jprobepayl8 = *(probepayl8);
            att11_nnationk = jprobepayl8.att11_nnationk;
            att12_nname = jprobepayl8.att12_nname;
            att16_nname = jprobepayl8.att16_nname;
            att19_ccustkey = jprobepayl8.att19_ccustkey;
            bucketFound8 = 1;
            bucketFound8 &= ((att19_ccustkey == att28_ocustkey));
            if(!(bucketFound8)) {
                probeActive8 = hashProbeUnique ( jht8, 30000, hash8, numLookups8, &(probepayl8));
            }
        }
        active = bucketFound8;
        // -------- hash join build (opId: 11) --------
        if(active) {
            jpayl11 payl11;
            payl11.att11_nnationk = att11_nnationk;
            payl11.att12_nname = att12_nname;
            payl11.att16_nname = att16_nname;
            payl11.att27_oorderke = att27_oorderke;
            uint64_t hash11;
            hash11 = 0;
            if(active) {
                hash11 = hash ( (hash11 + ((uint64_t)att27_oorderke)));
            }
            hashBuildUnique ( jht11, 300000, hash11, &(payl11));
        }
        loopVar += step;
    }

}

__global__ void krnl_lineitem9(
    int* iatt36_lorderke, int* iatt38_lsuppkey, float* iatt41_lextende, float* iatt42_ldiscoun, unsigned* iatt46_lshipdat, unique_ht<jpayl11>* jht11, unique_ht<jpayl12>* jht12, agg_ht<apayl15>* aht15, float* agg1, float* agg2, int* agg3) {
    int att36_lorderke;
    int att38_lsuppkey;
    float att41_lextende;
    float att42_ldiscoun;
    unsigned att46_lshipdat;
    int att11_nnationk;
    str_t att12_nname;
    str_t att16_nname;
    int att27_oorderke;
    int att4_ssuppkey;
    int att7_snationk;
    float att52_volume;
    unsigned att53_lyear;

    int tid_lineitem1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_lineitem1 = loopVar;
        active = (loopVar < 6001215);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att36_lorderke = iatt36_lorderke[tid_lineitem1];
            att38_lsuppkey = iatt38_lsuppkey[tid_lineitem1];
            att41_lextende = iatt41_lextende[tid_lineitem1];
            att42_ldiscoun = iatt42_ldiscoun[tid_lineitem1];
            att46_lshipdat = iatt46_lshipdat[tid_lineitem1];
        }
        // -------- selection (opId: 10) --------
        if(active) {
            active = ((att46_lshipdat >= 19950101) && (att46_lshipdat <= 19961231));
        }
        // -------- hash join probe (opId: 11) --------
        uint64_t hash11 = 0;
        if(active) {
            hash11 = 0;
            if(active) {
                hash11 = hash ( (hash11 + ((uint64_t)att36_lorderke)));
            }
        }
        jpayl11* probepayl11;
        int numLookups11 = 0;
        if(active) {
            active = hashProbeUnique ( jht11, 300000, hash11, numLookups11, &(probepayl11));
        }
        int bucketFound11 = 0;
        int probeActive11 = active;
        while((probeActive11 && !(bucketFound11))) {
            jpayl11 jprobepayl11 = *(probepayl11);
            att11_nnationk = jprobepayl11.att11_nnationk;
            att12_nname = jprobepayl11.att12_nname;
            att16_nname = jprobepayl11.att16_nname;
            att27_oorderke = jprobepayl11.att27_oorderke;
            bucketFound11 = 1;
            bucketFound11 &= ((att27_oorderke == att36_lorderke));
            if(!(bucketFound11)) {
                probeActive11 = hashProbeUnique ( jht11, 300000, hash11, numLookups11, &(probepayl11));
            }
        }
        active = bucketFound11;
        // -------- hash join probe (opId: 12) --------
        uint64_t hash12 = 0;
        if(active) {
            hash12 = 0;
            if(active) {
                hash12 = hash ( (hash12 + ((uint64_t)att11_nnationk)));
            }
            if(active) {
                hash12 = hash ( (hash12 + ((uint64_t)att38_lsuppkey)));
            }
        }
        jpayl12* probepayl12;
        int numLookups12 = 0;
        if(active) {
            active = hashProbeUnique ( jht12, 20000, hash12, numLookups12, &(probepayl12));
        }
        int bucketFound12 = 0;
        int probeActive12 = active;
        while((probeActive12 && !(bucketFound12))) {
            jpayl12 jprobepayl12 = *(probepayl12);
            att4_ssuppkey = jprobepayl12.att4_ssuppkey;
            att7_snationk = jprobepayl12.att7_snationk;
            bucketFound12 = 1;
            bucketFound12 &= ((att7_snationk == att11_nnationk));
            bucketFound12 &= ((att4_ssuppkey == att38_lsuppkey));
            if(!(bucketFound12)) {
                probeActive12 = hashProbeUnique ( jht12, 20000, hash12, numLookups12, &(probepayl12));
            }
        }
        active = bucketFound12;
        // -------- map (opId: 13) --------
        if(active) {
            att52_volume = (att41_lextende * (1 - att42_ldiscoun));
        }
        // -------- map (opId: 14) --------
        if(active) {
            att53_lyear = (att46_lshipdat / 10000);
        }
        // -------- aggregation (opId: 15) --------
        int bucket = 0;
        if(active) {
            uint64_t hash15 = 0;
            hash15 = 0;
            hash15 = hash ( (hash15 + stringHash ( att12_nname)));
            hash15 = hash ( (hash15 + stringHash ( att16_nname)));
            if(active) {
                hash15 = hash ( (hash15 + ((uint64_t)att53_lyear)));
            }
            apayl15 payl;
            payl.att12_nname = att12_nname;
            payl.att16_nname = att16_nname;
            payl.att53_lyear = att53_lyear;
            int bucketFound = 0;
            int numLookups = 0;
            while(!(bucketFound)) {
                bucket = hashAggregateGetBucket ( aht15, 24004, hash15, numLookups, &(payl));
                apayl15 probepayl = aht15[bucket].payload;
                bucketFound = 1;
                bucketFound &= (stringEquals ( payl.att12_nname, probepayl.att12_nname));
                bucketFound &= (stringEquals ( payl.att16_nname, probepayl.att16_nname));
                bucketFound &= ((payl.att53_lyear == probepayl.att53_lyear));
            }
        }
        if(active) {
            atomicAdd(&(agg1[bucket]), ((float)att52_volume));
            atomicAdd(&(agg2[bucket]), ((float)att52_volume));
            atomicAdd(&(agg3[bucket]), ((int)1));
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation15(
    agg_ht<apayl15>* aht15, float* agg1, float* agg2, int* agg3, int* nout_result, str_offs* oatt12_nname_offset, char* iatt12_nname_char, str_offs* oatt16_nname_offset, char* iatt16_nname_char, unsigned* oatt53_lyear, float* oatt1_sumvolum, float* oatt2_avgvolum) {
    str_t att12_nname;
    str_t att16_nname;
    unsigned att53_lyear;
    float att1_sumvolum;
    float att2_avgvolum;
    int att3_countagg;
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_aggregation15 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation15 = loopVar;
        active = (loopVar < 24004);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 15) --------
        if(active) {
            active &= ((aht15[tid_aggregation15].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl15 payl = aht15[tid_aggregation15].payload;
            att12_nname = payl.att12_nname;
            att16_nname = payl.att16_nname;
            att53_lyear = payl.att53_lyear;
        }
        if(active) {
            att1_sumvolum = agg1[tid_aggregation15];
            att2_avgvolum = agg2[tid_aggregation15];
            att3_countagg = agg3[tid_aggregation15];
            att2_avgvolum = (att2_avgvolum / ((float)att3_countagg));
        }
        // -------- projection (no code) (opId: 16) --------
        // -------- materialize (opId: 17) --------
        int wp;
        int writeMask;
        int numProj;
        writeMask = __ballot_sync(ALL_LANES,active);
        numProj = __popc(writeMask);
        if((warplane == 0)) {
            wp = atomicAdd(nout_result, numProj);
        }
        wp = __shfl_sync(ALL_LANES,wp,0);
        wp = (wp + __popc((writeMask & prefixlanes)));
        if(active) {
            oatt12_nname_offset[wp] = toStringOffset ( iatt12_nname_char, att12_nname);
            oatt16_nname_offset[wp] = toStringOffset ( iatt16_nname_char, att16_nname);
            oatt53_lyear[wp] = att53_lyear;
            oatt1_sumvolum[wp] = att1_sumvolum;
            oatt2_avgvolum[wp] = att2_avgvolum;
        }
        loopVar += step;
    }

}

int main() {
    int* iatt4_ssuppkey;
    iatt4_ssuppkey = ( int*) map_memory_file ( "mmdb/supplier_s_suppkey" );
    int* iatt7_snationk;
    iatt7_snationk = ( int*) map_memory_file ( "mmdb/supplier_s_nationkey" );
    int* iatt11_nnationk;
    iatt11_nnationk = ( int*) map_memory_file ( "mmdb/nation_n_nationkey" );
    size_t* iatt12_nname_offset;
    iatt12_nname_offset = ( size_t*) map_memory_file ( "mmdb/nation_n_name_offset" );
    char* iatt12_nname_char;
    iatt12_nname_char = ( char*) map_memory_file ( "mmdb/nation_n_name_char" );
    int* iatt15_nnationk;
    iatt15_nnationk = ( int*) map_memory_file ( "mmdb/nation_n_nationkey" );
    size_t* iatt16_nname_offset;
    iatt16_nname_offset = ( size_t*) map_memory_file ( "mmdb/nation_n_name_offset" );
    char* iatt16_nname_char;
    iatt16_nname_char = ( char*) map_memory_file ( "mmdb/nation_n_name_char" );
    int* iatt19_ccustkey;
    iatt19_ccustkey = ( int*) map_memory_file ( "mmdb/customer_c_custkey" );
    int* iatt22_cnationk;
    iatt22_cnationk = ( int*) map_memory_file ( "mmdb/customer_c_nationkey" );
    int* iatt27_oorderke;
    iatt27_oorderke = ( int*) map_memory_file ( "mmdb/orders_o_orderkey" );
    int* iatt28_ocustkey;
    iatt28_ocustkey = ( int*) map_memory_file ( "mmdb/orders_o_custkey" );
    int* iatt36_lorderke;
    iatt36_lorderke = ( int*) map_memory_file ( "mmdb/lineitem_l_orderkey" );
    int* iatt38_lsuppkey;
    iatt38_lsuppkey = ( int*) map_memory_file ( "mmdb/lineitem_l_suppkey" );
    float* iatt41_lextende;
    iatt41_lextende = ( float*) map_memory_file ( "mmdb/lineitem_l_extendedprice" );
    float* iatt42_ldiscoun;
    iatt42_ldiscoun = ( float*) map_memory_file ( "mmdb/lineitem_l_discount" );
    unsigned* iatt46_lshipdat;
    iatt46_lshipdat = ( unsigned*) map_memory_file ( "mmdb/lineitem_l_shipdate" );

    int nout_inner4;
    int nout_result;
    std::vector < str_offs > oatt12_nname_offset(12002);
    std::vector < str_offs > oatt16_nname_offset(12002);
    std::vector < unsigned > oatt53_lyear(12002);
    std::vector < float > oatt1_sumvolum(12002);
    std::vector < float > oatt2_avgvolum(12002);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt4_ssuppkey;
    hipMalloc((void**) &d_iatt4_ssuppkey, 10000* sizeof(int) );
    int* d_iatt7_snationk;
    hipMalloc((void**) &d_iatt7_snationk, 10000* sizeof(int) );
    int* d_iatt11_nnationk;
    hipMalloc((void**) &d_iatt11_nnationk, 25* sizeof(int) );
    size_t* d_iatt12_nname_offset;
    hipMalloc((void**) &d_iatt12_nname_offset, (25 + 1)* sizeof(size_t) );
    char* d_iatt12_nname_char;
    hipMalloc((void**) &d_iatt12_nname_char, 186* sizeof(char) );
    int* d_nout_inner4;
    hipMalloc((void**) &d_nout_inner4, 1* sizeof(int) );
    int* d_itm_inner4_n_nationkey;
    hipMalloc((void**) &d_itm_inner4_n_nationkey, 25* sizeof(int) );
    str_t* d_itm_inner4_n_name;
    hipMalloc((void**) &d_itm_inner4_n_name, 25* sizeof(str_t) );
    int* d_iatt15_nnationk;
    d_iatt15_nnationk = d_iatt11_nnationk;
    size_t* d_iatt16_nname_offset;
    d_iatt16_nname_offset = d_iatt12_nname_offset;
    char* d_iatt16_nname_char;
    d_iatt16_nname_char = d_iatt12_nname_char;
    int* d_iatt19_ccustkey;
    hipMalloc((void**) &d_iatt19_ccustkey, 150000* sizeof(int) );
    int* d_iatt22_cnationk;
    hipMalloc((void**) &d_iatt22_cnationk, 150000* sizeof(int) );
    int* d_iatt27_oorderke;
    hipMalloc((void**) &d_iatt27_oorderke, 1500000* sizeof(int) );
    int* d_iatt28_ocustkey;
    hipMalloc((void**) &d_iatt28_ocustkey, 1500000* sizeof(int) );
    int* d_iatt36_lorderke;
    hipMalloc((void**) &d_iatt36_lorderke, 6001215* sizeof(int) );
    int* d_iatt38_lsuppkey;
    hipMalloc((void**) &d_iatt38_lsuppkey, 6001215* sizeof(int) );
    float* d_iatt41_lextende;
    hipMalloc((void**) &d_iatt41_lextende, 6001215* sizeof(float) );
    float* d_iatt42_ldiscoun;
    hipMalloc((void**) &d_iatt42_ldiscoun, 6001215* sizeof(float) );
    unsigned* d_iatt46_lshipdat;
    hipMalloc((void**) &d_iatt46_lshipdat, 6001215* sizeof(unsigned) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    str_offs* d_oatt12_nname_offset;
    hipMalloc((void**) &d_oatt12_nname_offset, 12002* sizeof(str_offs) );
    str_offs* d_oatt16_nname_offset;
    hipMalloc((void**) &d_oatt16_nname_offset, 12002* sizeof(str_offs) );
    unsigned* d_oatt53_lyear;
    hipMalloc((void**) &d_oatt53_lyear, 12002* sizeof(unsigned) );
    float* d_oatt1_sumvolum;
    hipMalloc((void**) &d_oatt1_sumvolum, 12002* sizeof(float) );
    float* d_oatt2_avgvolum;
    hipMalloc((void**) &d_oatt2_avgvolum, 12002* sizeof(float) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    unique_ht<jpayl12>* d_jht12;
    hipMalloc((void**) &d_jht12, 20000* sizeof(unique_ht<jpayl12>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht12, 20000);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_inner4, 0, 1);
    }
    unique_ht<jpayl6>* d_jht6;
    hipMalloc((void**) &d_jht6, 1250* sizeof(unique_ht<jpayl6>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht6, 1250);
    }
    unique_ht<jpayl8>* d_jht8;
    hipMalloc((void**) &d_jht8, 30000* sizeof(unique_ht<jpayl8>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht8, 30000);
    }
    unique_ht<jpayl11>* d_jht11;
    hipMalloc((void**) &d_jht11, 300000* sizeof(unique_ht<jpayl11>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht11, 300000);
    }
    agg_ht<apayl15>* d_aht15;
    hipMalloc((void**) &d_aht15, 24004* sizeof(agg_ht<apayl15>) );
    {
        int gridsize=920;
        int blocksize=128;
        initAggHT<<<gridsize, blocksize>>>(d_aht15, 24004);
    }
    float* d_agg1;
    hipMalloc((void**) &d_agg1, 24004* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg1, 0.0f, 24004);
    }
    float* d_agg2;
    hipMalloc((void**) &d_agg2, 24004* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg2, 0.0f, 24004);
    }
    int* d_agg3;
    hipMalloc((void**) &d_agg3, 24004* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg3, 0, 24004);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt4_ssuppkey, iatt4_ssuppkey, 10000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt7_snationk, iatt7_snationk, 10000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt11_nnationk, iatt11_nnationk, 25 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt12_nname_offset, iatt12_nname_offset, (25 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt12_nname_char, iatt12_nname_char, 186 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt19_ccustkey, iatt19_ccustkey, 150000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt22_cnationk, iatt22_cnationk, 150000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt27_oorderke, iatt27_oorderke, 1500000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt28_ocustkey, iatt28_ocustkey, 1500000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt36_lorderke, iatt36_lorderke, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt38_lsuppkey, iatt38_lsuppkey, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt41_lextende, iatt41_lextende, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt42_ldiscoun, iatt42_ldiscoun, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt46_lshipdat, iatt46_lshipdat, 6001215 * sizeof(unsigned), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime47 = std::clock();
    std::clock_t start_krnl_supplier148 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_supplier1<<<gridsize, blocksize>>>(d_iatt4_ssuppkey, d_iatt7_snationk, d_jht12);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_supplier148 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_supplier1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_supplier1")
        }
    }

    std::clock_t start_krnl_nation249 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_nation2<<<gridsize, blocksize>>>(d_iatt11_nnationk, d_iatt12_nname_offset, d_iatt12_nname_char, d_nout_inner4, d_itm_inner4_n_nationkey, d_itm_inner4_n_name);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation249 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation2! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation2")
        }
    }

    std::clock_t start_krnl_nation2350 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_nation23<<<gridsize, blocksize>>>(d_iatt15_nnationk, d_iatt16_nname_offset, d_iatt16_nname_char, d_nout_inner4, d_itm_inner4_n_nationkey, d_itm_inner4_n_name, d_jht6);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation2350 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation23! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation23")
        }
    }

    std::clock_t start_krnl_customer551 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_customer5<<<gridsize, blocksize>>>(d_iatt19_ccustkey, d_iatt22_cnationk, d_jht6, d_jht8);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_customer551 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_customer5! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_customer5")
        }
    }

    std::clock_t start_krnl_orders752 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_orders7<<<gridsize, blocksize>>>(d_iatt27_oorderke, d_iatt28_ocustkey, d_jht8, d_jht11);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_orders752 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_orders7! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_orders7")
        }
    }

    std::clock_t start_krnl_lineitem953 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_lineitem9<<<gridsize, blocksize>>>(d_iatt36_lorderke, d_iatt38_lsuppkey, d_iatt41_lextende, d_iatt42_ldiscoun, d_iatt46_lshipdat, d_jht11, d_jht12, d_aht15, d_agg1, d_agg2, d_agg3);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_lineitem953 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem9! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem9")
        }
    }

    std::clock_t start_krnl_aggregation1554 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation15<<<gridsize, blocksize>>>(d_aht15, d_agg1, d_agg2, d_agg3, d_nout_result, d_oatt12_nname_offset, d_iatt12_nname_char, d_oatt16_nname_offset, d_iatt16_nname_char, d_oatt53_lyear, d_oatt1_sumvolum, d_oatt2_avgvolum);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation1554 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation15! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation15")
        }
    }

    std::clock_t stop_totalKernelTime47 = std::clock();
    hipMemcpy( &nout_inner4, d_nout_inner4, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt12_nname_offset.data(), d_oatt12_nname_offset, 12002 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt16_nname_offset.data(), d_oatt16_nname_offset, 12002 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt53_lyear.data(), d_oatt53_lyear, 12002 * sizeof(unsigned), hipMemcpyDeviceToHost);
    hipMemcpy( oatt1_sumvolum.data(), d_oatt1_sumvolum, 12002 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( oatt2_avgvolum.data(), d_oatt2_avgvolum, 12002 * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt4_ssuppkey);
    hipFree( d_iatt7_snationk);
    hipFree( d_jht12);
    hipFree( d_iatt11_nnationk);
    hipFree( d_iatt12_nname_offset);
    hipFree( d_iatt12_nname_char);
    hipFree( d_nout_inner4);
    hipFree( d_itm_inner4_n_nationkey);
    hipFree( d_itm_inner4_n_name);
    hipFree( d_jht6);
    hipFree( d_iatt19_ccustkey);
    hipFree( d_iatt22_cnationk);
    hipFree( d_jht8);
    hipFree( d_iatt27_oorderke);
    hipFree( d_iatt28_ocustkey);
    hipFree( d_jht11);
    hipFree( d_iatt36_lorderke);
    hipFree( d_iatt38_lsuppkey);
    hipFree( d_iatt41_lextende);
    hipFree( d_iatt42_ldiscoun);
    hipFree( d_iatt46_lshipdat);
    hipFree( d_aht15);
    hipFree( d_agg1);
    hipFree( d_agg2);
    hipFree( d_agg3);
    hipFree( d_nout_result);
    hipFree( d_oatt12_nname_offset);
    hipFree( d_oatt16_nname_offset);
    hipFree( d_oatt53_lyear);
    hipFree( d_oatt1_sumvolum);
    hipFree( d_oatt2_avgvolum);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish55 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 12002)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("n_name: ");
        stringPrint ( iatt12_nname_char, oatt12_nname_offset[pv]);
        printf("  ");
        printf("n_name: ");
        stringPrint ( iatt16_nname_char, oatt16_nname_offset[pv]);
        printf("  ");
        printf("l_year: ");
        printf("%10i", oatt53_lyear[pv]);
        printf("  ");
        printf("sum_volume: ");
        printf("%15.2f", oatt1_sumvolum[pv]);
        printf("  ");
        printf("avg_volume: ");
        printf("%15.2f", oatt2_avgvolum[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish55 = std::clock();

    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "krnl_supplier1", (stop_krnl_supplier148 - start_krnl_supplier148) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_nation2", (stop_krnl_nation249 - start_krnl_nation249) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_nation23", (stop_krnl_nation2350 - start_krnl_nation2350) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_customer5", (stop_krnl_customer551 - start_krnl_customer551) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_orders7", (stop_krnl_orders752 - start_krnl_orders752) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_lineitem9", (stop_krnl_lineitem953 - start_krnl_lineitem953) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation15", (stop_krnl_aggregation1554 - start_krnl_aggregation1554) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish55 - start_finish55) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime47 - start_totalKernelTime47) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
