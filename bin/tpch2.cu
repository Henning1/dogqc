#include "hip/hip_runtime.h"
#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct jpayl4 {
    int att2_rregionk;
};
struct jpayl6 {
    int att5_nnationk;
    str_t att6_nname;
};
struct jpayl25 {
    str_t att6_nname;
    int att9_ssuppkey;
    str_t att10_sname;
    str_t att11_saddress;
    str_t att13_sphone;
    float att14_sacctbal;
    str_t att15_scomment;
};
struct jpayl10 {
    int att16_rregionk;
};
struct jpayl12 {
    int att19_nnationk;
};
struct jpayl17 {
    int att23_ssuppkey;
};
struct jpayl16 {
    int att30_ppartkey;
};
struct apayl18 {
    int att39_pspartke;
};
struct jpayl21 {
    int att39_pspartke;
    float att1_minsuppl;
};
struct jpayl23 {
    int att39_pspartke;
    float att1_minsuppl;
    int att44_ppartkey;
    str_t att46_pmfgr;
};

__global__ void krnl_region1(
    int* iatt2_rregionk, size_t* iatt3_rname_offset, char* iatt3_rname_char, unique_ht<jpayl4>* jht4) {
    int att2_rregionk;
    str_t att3_rname;
    str_t c1 = stringConstant ( "EUROPE", 6);

    int tid_region1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_region1 = loopVar;
        active = (loopVar < 5);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att2_rregionk = iatt2_rregionk[tid_region1];
            att3_rname = stringScan ( iatt3_rname_offset, iatt3_rname_char, tid_region1);
        }
        // -------- selection (opId: 2) --------
        if(active) {
            active = stringEquals ( att3_rname, c1);
        }
        // -------- hash join build (opId: 4) --------
        if(active) {
            jpayl4 payl4;
            payl4.att2_rregionk = att2_rregionk;
            uint64_t hash4;
            hash4 = 0;
            if(active) {
                hash4 = hash ( (hash4 + ((uint64_t)att2_rregionk)));
            }
            hashBuildUnique ( jht4, 10, hash4, &(payl4));
        }
        loopVar += step;
    }

}

__global__ void krnl_nation3(
    int* iatt5_nnationk, size_t* iatt6_nname_offset, char* iatt6_nname_char, int* iatt7_nregionk, unique_ht<jpayl4>* jht4, multi_ht* jht6, jpayl6* jht6_payload) {
    int att5_nnationk;
    str_t att6_nname;
    int att7_nregionk;
    int att2_rregionk;

    int tid_nation1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation1 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att5_nnationk = iatt5_nnationk[tid_nation1];
            att6_nname = stringScan ( iatt6_nname_offset, iatt6_nname_char, tid_nation1);
            att7_nregionk = iatt7_nregionk[tid_nation1];
        }
        // -------- hash join probe (opId: 4) --------
        uint64_t hash4 = 0;
        if(active) {
            hash4 = 0;
            if(active) {
                hash4 = hash ( (hash4 + ((uint64_t)att7_nregionk)));
            }
        }
        jpayl4* probepayl4;
        int numLookups4 = 0;
        if(active) {
            active = hashProbeUnique ( jht4, 10, hash4, numLookups4, &(probepayl4));
        }
        int bucketFound4 = 0;
        int probeActive4 = active;
        while((probeActive4 && !(bucketFound4))) {
            jpayl4 jprobepayl4 = *(probepayl4);
            att2_rregionk = jprobepayl4.att2_rregionk;
            bucketFound4 = 1;
            bucketFound4 &= ((att2_rregionk == att7_nregionk));
            if(!(bucketFound4)) {
                probeActive4 = hashProbeUnique ( jht4, 10, hash4, numLookups4, &(probepayl4));
            }
        }
        active = bucketFound4;
        // -------- hash join build (opId: 6) --------
        if(active) {
            uint64_t hash6 = 0;
            if(active) {
                hash6 = 0;
                if(active) {
                    hash6 = hash ( (hash6 + ((uint64_t)att5_nnationk)));
                }
            }
            hashCountMulti ( jht6, 50, hash6);
        }
        loopVar += step;
    }

}

__global__ void krnl_nation3_ins(
    int* iatt5_nnationk, size_t* iatt6_nname_offset, char* iatt6_nname_char, int* iatt7_nregionk, unique_ht<jpayl4>* jht4, multi_ht* jht6, jpayl6* jht6_payload, int* offs6) {
    int att5_nnationk;
    str_t att6_nname;
    int att7_nregionk;
    int att2_rregionk;

    int tid_nation1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation1 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att5_nnationk = iatt5_nnationk[tid_nation1];
            att6_nname = stringScan ( iatt6_nname_offset, iatt6_nname_char, tid_nation1);
            att7_nregionk = iatt7_nregionk[tid_nation1];
        }
        // -------- hash join probe (opId: 4) --------
        uint64_t hash4 = 0;
        if(active) {
            hash4 = 0;
            if(active) {
                hash4 = hash ( (hash4 + ((uint64_t)att7_nregionk)));
            }
        }
        jpayl4* probepayl4;
        int numLookups4 = 0;
        if(active) {
            active = hashProbeUnique ( jht4, 10, hash4, numLookups4, &(probepayl4));
        }
        int bucketFound4 = 0;
        int probeActive4 = active;
        while((probeActive4 && !(bucketFound4))) {
            jpayl4 jprobepayl4 = *(probepayl4);
            att2_rregionk = jprobepayl4.att2_rregionk;
            bucketFound4 = 1;
            bucketFound4 &= ((att2_rregionk == att7_nregionk));
            if(!(bucketFound4)) {
                probeActive4 = hashProbeUnique ( jht4, 10, hash4, numLookups4, &(probepayl4));
            }
        }
        active = bucketFound4;
        // -------- hash join build (opId: 6) --------
        if(active) {
            uint64_t hash6 = 0;
            if(active) {
                hash6 = 0;
                if(active) {
                    hash6 = hash ( (hash6 + ((uint64_t)att5_nnationk)));
                }
            }
            jpayl6 payl;
            payl.att5_nnationk = att5_nnationk;
            payl.att6_nname = att6_nname;
            hashInsertMulti ( jht6, jht6_payload, offs6, 50, hash6, &(payl));
        }
        loopVar += step;
    }

}

__global__ void krnl_supplier5(
    int* iatt9_ssuppkey, size_t* iatt10_sname_offset, char* iatt10_sname_char, size_t* iatt11_saddress_offset, char* iatt11_saddress_char, int* iatt12_snationk, size_t* iatt13_sphone_offset, char* iatt13_sphone_char, float* iatt14_sacctbal, size_t* iatt15_scomment_offset, char* iatt15_scomment_char, multi_ht* jht6, jpayl6* jht6_payload, unique_ht<jpayl25>* jht25) {
    int att9_ssuppkey;
    str_t att10_sname;
    str_t att11_saddress;
    int att12_snationk;
    str_t att13_sphone;
    float att14_sacctbal;
    str_t att15_scomment;
    unsigned warplane = (threadIdx.x % 32);
    int att5_nnationk;
    str_t att6_nname;

    int tid_supplier1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_supplier1 = loopVar;
        active = (loopVar < 10000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att9_ssuppkey = iatt9_ssuppkey[tid_supplier1];
            att10_sname = stringScan ( iatt10_sname_offset, iatt10_sname_char, tid_supplier1);
            att11_saddress = stringScan ( iatt11_saddress_offset, iatt11_saddress_char, tid_supplier1);
            att12_snationk = iatt12_snationk[tid_supplier1];
            att13_sphone = stringScan ( iatt13_sphone_offset, iatt13_sphone_char, tid_supplier1);
            att14_sacctbal = iatt14_sacctbal[tid_supplier1];
            att15_scomment = stringScan ( iatt15_scomment_offset, iatt15_scomment_char, tid_supplier1);
        }
        // -------- hash join probe (opId: 6) --------
        // -------- multiprobe multi broadcast (opId: 6) --------
        int matchEnd6 = 0;
        int matchEndBuf6 = 0;
        int matchOffset6 = 0;
        int matchOffsetBuf6 = 0;
        int probeActive6 = active;
        int att9_ssuppkey_bcbuf6;
        str_t att10_sname_bcbuf6;
        str_t att11_saddress_bcbuf6;
        int att12_snationk_bcbuf6;
        str_t att13_sphone_bcbuf6;
        float att14_sacctbal_bcbuf6;
        str_t att15_scomment_bcbuf6;
        uint64_t hash6 = 0;
        if(probeActive6) {
            hash6 = 0;
            if(active) {
                hash6 = hash ( (hash6 + ((uint64_t)att12_snationk)));
            }
            probeActive6 = hashProbeMulti ( jht6, 50, hash6, matchOffsetBuf6, matchEndBuf6);
        }
        unsigned activeProbes6 = __ballot_sync(ALL_LANES,probeActive6);
        int num6 = 0;
        num6 = (matchEndBuf6 - matchOffsetBuf6);
        unsigned wideProbes6 = __ballot_sync(ALL_LANES,(num6 >= 32));
        att9_ssuppkey_bcbuf6 = att9_ssuppkey;
        att10_sname_bcbuf6 = att10_sname;
        att11_saddress_bcbuf6 = att11_saddress;
        att12_snationk_bcbuf6 = att12_snationk;
        att13_sphone_bcbuf6 = att13_sphone;
        att14_sacctbal_bcbuf6 = att14_sacctbal;
        att15_scomment_bcbuf6 = att15_scomment;
        while((activeProbes6 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes6)) {
                if((wideProbes6 > 0)) {
                    tupleLane = (__ffs(wideProbes6) - 1);
                    wideProbes6 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes6) - 1);
                }
                num = __shfl_sync(ALL_LANES,num6,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset6 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes6 -= (1 << tupleLane);
            }
            matchOffset6 += __shfl_sync(ALL_LANES,matchOffsetBuf6,broadcastLane);
            matchEnd6 = __shfl_sync(ALL_LANES,matchEndBuf6,broadcastLane);
            att9_ssuppkey = __shfl_sync(ALL_LANES,att9_ssuppkey_bcbuf6,broadcastLane);
            att10_sname = __shfl_sync(ALL_LANES,att10_sname_bcbuf6,broadcastLane);
            att11_saddress = __shfl_sync(ALL_LANES,att11_saddress_bcbuf6,broadcastLane);
            att12_snationk = __shfl_sync(ALL_LANES,att12_snationk_bcbuf6,broadcastLane);
            att13_sphone = __shfl_sync(ALL_LANES,att13_sphone_bcbuf6,broadcastLane);
            att14_sacctbal = __shfl_sync(ALL_LANES,att14_sacctbal_bcbuf6,broadcastLane);
            att15_scomment = __shfl_sync(ALL_LANES,att15_scomment_bcbuf6,broadcastLane);
            probeActive6 = (matchOffset6 < matchEnd6);
            while(__any_sync(ALL_LANES,probeActive6)) {
                active = probeActive6;
                active = 0;
                jpayl6 payl;
                if(probeActive6) {
                    payl = jht6_payload[matchOffset6];
                    att5_nnationk = payl.att5_nnationk;
                    att6_nname = payl.att6_nname;
                    active = 1;
                    active &= ((att5_nnationk == att12_snationk));
                    matchOffset6 += 32;
                    probeActive6 &= ((matchOffset6 < matchEnd6));
                }
                // -------- hash join build (opId: 25) --------
                if(active) {
                    jpayl25 payl25;
                    payl25.att6_nname = att6_nname;
                    payl25.att9_ssuppkey = att9_ssuppkey;
                    payl25.att10_sname = att10_sname;
                    payl25.att11_saddress = att11_saddress;
                    payl25.att13_sphone = att13_sphone;
                    payl25.att14_sacctbal = att14_sacctbal;
                    payl25.att15_scomment = att15_scomment;
                    uint64_t hash25;
                    hash25 = 0;
                    if(active) {
                        hash25 = hash ( (hash25 + ((uint64_t)att9_ssuppkey)));
                    }
                    hashBuildUnique ( jht25, 20000, hash25, &(payl25));
                }
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_region27(
    int* iatt16_rregionk, size_t* iatt17_rname_offset, char* iatt17_rname_char, unique_ht<jpayl10>* jht10) {
    int att16_rregionk;
    str_t att17_rname;
    str_t c2 = stringConstant ( "EUROPE", 6);

    int tid_region2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_region2 = loopVar;
        active = (loopVar < 5);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att16_rregionk = iatt16_rregionk[tid_region2];
            att17_rname = stringScan ( iatt17_rname_offset, iatt17_rname_char, tid_region2);
        }
        // -------- selection (opId: 8) --------
        if(active) {
            active = stringEquals ( att17_rname, c2);
        }
        // -------- hash join build (opId: 10) --------
        if(active) {
            jpayl10 payl10;
            payl10.att16_rregionk = att16_rregionk;
            uint64_t hash10;
            hash10 = 0;
            if(active) {
                hash10 = hash ( (hash10 + ((uint64_t)att16_rregionk)));
            }
            hashBuildUnique ( jht10, 10, hash10, &(payl10));
        }
        loopVar += step;
    }

}

__global__ void krnl_nation29(
    int* iatt19_nnationk, int* iatt21_nregionk, unique_ht<jpayl10>* jht10, unique_ht<jpayl12>* jht12) {
    int att19_nnationk;
    int att21_nregionk;
    int att16_rregionk;

    int tid_nation2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation2 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att19_nnationk = iatt19_nnationk[tid_nation2];
            att21_nregionk = iatt21_nregionk[tid_nation2];
        }
        // -------- hash join probe (opId: 10) --------
        uint64_t hash10 = 0;
        if(active) {
            hash10 = 0;
            if(active) {
                hash10 = hash ( (hash10 + ((uint64_t)att21_nregionk)));
            }
        }
        jpayl10* probepayl10;
        int numLookups10 = 0;
        if(active) {
            active = hashProbeUnique ( jht10, 10, hash10, numLookups10, &(probepayl10));
        }
        int bucketFound10 = 0;
        int probeActive10 = active;
        while((probeActive10 && !(bucketFound10))) {
            jpayl10 jprobepayl10 = *(probepayl10);
            att16_rregionk = jprobepayl10.att16_rregionk;
            bucketFound10 = 1;
            bucketFound10 &= ((att16_rregionk == att21_nregionk));
            if(!(bucketFound10)) {
                probeActive10 = hashProbeUnique ( jht10, 10, hash10, numLookups10, &(probepayl10));
            }
        }
        active = bucketFound10;
        // -------- hash join build (opId: 12) --------
        if(active) {
            jpayl12 payl12;
            payl12.att19_nnationk = att19_nnationk;
            uint64_t hash12;
            hash12 = 0;
            if(active) {
                hash12 = hash ( (hash12 + ((uint64_t)att19_nnationk)));
            }
            hashBuildUnique ( jht12, 50, hash12, &(payl12));
        }
        loopVar += step;
    }

}

__global__ void krnl_supplier211(
    int* iatt23_ssuppkey, int* iatt26_snationk, unique_ht<jpayl12>* jht12, unique_ht<jpayl17>* jht17) {
    int att23_ssuppkey;
    int att26_snationk;
    int att19_nnationk;

    int tid_supplier2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_supplier2 = loopVar;
        active = (loopVar < 10000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att23_ssuppkey = iatt23_ssuppkey[tid_supplier2];
            att26_snationk = iatt26_snationk[tid_supplier2];
        }
        // -------- hash join probe (opId: 12) --------
        uint64_t hash12 = 0;
        if(active) {
            hash12 = 0;
            if(active) {
                hash12 = hash ( (hash12 + ((uint64_t)att26_snationk)));
            }
        }
        jpayl12* probepayl12;
        int numLookups12 = 0;
        if(active) {
            active = hashProbeUnique ( jht12, 50, hash12, numLookups12, &(probepayl12));
        }
        int bucketFound12 = 0;
        int probeActive12 = active;
        while((probeActive12 && !(bucketFound12))) {
            jpayl12 jprobepayl12 = *(probepayl12);
            att19_nnationk = jprobepayl12.att19_nnationk;
            bucketFound12 = 1;
            bucketFound12 &= ((att19_nnationk == att26_snationk));
            if(!(bucketFound12)) {
                probeActive12 = hashProbeUnique ( jht12, 50, hash12, numLookups12, &(probepayl12));
            }
        }
        active = bucketFound12;
        // -------- hash join build (opId: 17) --------
        if(active) {
            jpayl17 payl17;
            payl17.att23_ssuppkey = att23_ssuppkey;
            uint64_t hash17;
            hash17 = 0;
            if(active) {
                hash17 = hash ( (hash17 + ((uint64_t)att23_ssuppkey)));
            }
            hashBuildUnique ( jht17, 20000, hash17, &(payl17));
        }
        loopVar += step;
    }

}

__global__ void krnl_part13(
    int* iatt30_ppartkey, size_t* iatt34_ptype_offset, char* iatt34_ptype_char, int* iatt35_psize, agg_ht<jpayl16>* jht16) {
    int att30_ppartkey;
    str_t att34_ptype;
    int att35_psize;
    str_t c3 = stringConstant ( "%BRASS", 6);

    int tid_part1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_part1 = loopVar;
        active = (loopVar < 200000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att30_ppartkey = iatt30_ppartkey[tid_part1];
            att34_ptype = stringScan ( iatt34_ptype_offset, iatt34_ptype_char, tid_part1);
            att35_psize = iatt35_psize[tid_part1];
        }
        // -------- selection (opId: 14) --------
        if(active) {
            active = (stringLikeCheck ( att34_ptype, c3) && (att35_psize == 15));
        }
        // -------- hash join build (opId: 16) --------
        if(active) {
            uint64_t hash16;
            hash16 = 0;
            if(active) {
                hash16 = hash ( (hash16 + ((uint64_t)att30_ppartkey)));
            }
            int bucket = 0;
            jpayl16 payl16;
            payl16.att30_ppartkey = att30_ppartkey;
            int bucketFound = 0;
            int numLookups = 0;
            while(!(bucketFound)) {
                bucket = hashAggregateGetBucket ( jht16, 400000, hash16, numLookups, &(payl16));
                jpayl16 probepayl = jht16[bucket].payload;
                bucketFound = 1;
                bucketFound &= ((payl16.att30_ppartkey == probepayl.att30_ppartkey));
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_partsupp15(
    int* iatt39_pspartke, int* iatt40_pssuppke, float* iatt42_pssupply, agg_ht<jpayl16>* jht16, unique_ht<jpayl17>* jht17, agg_ht<apayl18>* aht18, float* agg1) {
    int att39_pspartke;
    int att40_pssuppke;
    float att42_pssupply;
    int att30_ppartkey;
    int att23_ssuppkey;

    int tid_partsupp1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_partsupp1 = loopVar;
        active = (loopVar < 800000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att39_pspartke = iatt39_pspartke[tid_partsupp1];
            att40_pssuppke = iatt40_pssuppke[tid_partsupp1];
            att42_pssupply = iatt42_pssupply[tid_partsupp1];
        }
        // -------- hash join probe (opId: 16) --------
        if(active) {
            uint64_t hash16 = 0;
            hash16 = 0;
            if(active) {
                hash16 = hash ( (hash16 + ((uint64_t)att39_pspartke)));
            }
            int numLookups16 = 0;
            int location16 = 0;
            int filterMatch16 = 0;
            int activeProbe16 = 1;
            while((!(filterMatch16) && activeProbe16)) {
                activeProbe16 = hashAggregateFindBucket ( jht16, 400000, hash16, numLookups16, location16);
                if(activeProbe16) {
                    jpayl16 probepayl = jht16[location16].payload;
                    att30_ppartkey = probepayl.att30_ppartkey;
                    filterMatch16 = 1;
                    filterMatch16 &= ((att30_ppartkey == att39_pspartke));
                }
            }
            active &= (filterMatch16);
        }
        // -------- hash join probe (opId: 17) --------
        uint64_t hash17 = 0;
        if(active) {
            hash17 = 0;
            if(active) {
                hash17 = hash ( (hash17 + ((uint64_t)att40_pssuppke)));
            }
        }
        jpayl17* probepayl17;
        int numLookups17 = 0;
        if(active) {
            active = hashProbeUnique ( jht17, 20000, hash17, numLookups17, &(probepayl17));
        }
        int bucketFound17 = 0;
        int probeActive17 = active;
        while((probeActive17 && !(bucketFound17))) {
            jpayl17 jprobepayl17 = *(probepayl17);
            att23_ssuppkey = jprobepayl17.att23_ssuppkey;
            bucketFound17 = 1;
            bucketFound17 &= ((att23_ssuppkey == att40_pssuppke));
            if(!(bucketFound17)) {
                probeActive17 = hashProbeUnique ( jht17, 20000, hash17, numLookups17, &(probepayl17));
            }
        }
        active = bucketFound17;
        // -------- aggregation (opId: 18) --------
        int bucket = 0;
        if(active) {
            uint64_t hash18 = 0;
            hash18 = 0;
            if(active) {
                hash18 = hash ( (hash18 + ((uint64_t)att39_pspartke)));
            }
            apayl18 payl;
            payl.att39_pspartke = att39_pspartke;
            int bucketFound = 0;
            int numLookups = 0;
            while(!(bucketFound)) {
                bucket = hashAggregateGetBucket ( aht18, 1600000, hash18, numLookups, &(payl));
                apayl18 probepayl = aht18[bucket].payload;
                bucketFound = 1;
                bucketFound &= ((payl.att39_pspartke == probepayl.att39_pspartke));
            }
        }
        if(active) {
            atomicMin(&(agg1[bucket]), ((float)att42_pssupply));
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation18(
    agg_ht<apayl18>* aht18, float* agg1, multi_ht* jht21, jpayl21* jht21_payload) {
    int att39_pspartke;
    float att1_minsuppl;

    int tid_aggregation18 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation18 = loopVar;
        active = (loopVar < 1600000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 18) --------
        if(active) {
            active &= ((aht18[tid_aggregation18].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl18 payl = aht18[tid_aggregation18].payload;
            att39_pspartke = payl.att39_pspartke;
        }
        if(active) {
            att1_minsuppl = agg1[tid_aggregation18];
        }
        // -------- hash join build (opId: 21) --------
        if(active) {
            uint64_t hash21 = 0;
            if(active) {
                hash21 = 0;
                if(active) {
                    hash21 = hash ( (hash21 + ((uint64_t)att39_pspartke)));
                }
            }
            hashCountMulti ( jht21, 1600000, hash21);
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation18_ins(
    agg_ht<apayl18>* aht18, float* agg1, multi_ht* jht21, jpayl21* jht21_payload, int* offs21) {
    int att39_pspartke;
    float att1_minsuppl;

    int tid_aggregation18 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation18 = loopVar;
        active = (loopVar < 1600000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 18) --------
        if(active) {
            active &= ((aht18[tid_aggregation18].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl18 payl = aht18[tid_aggregation18].payload;
            att39_pspartke = payl.att39_pspartke;
        }
        if(active) {
            att1_minsuppl = agg1[tid_aggregation18];
        }
        // -------- hash join build (opId: 21) --------
        if(active) {
            uint64_t hash21 = 0;
            if(active) {
                hash21 = 0;
                if(active) {
                    hash21 = hash ( (hash21 + ((uint64_t)att39_pspartke)));
                }
            }
            jpayl21 payl;
            payl.att39_pspartke = att39_pspartke;
            payl.att1_minsuppl = att1_minsuppl;
            hashInsertMulti ( jht21, jht21_payload, offs21, 1600000, hash21, &(payl));
        }
        loopVar += step;
    }

}

__global__ void krnl_part219(
    int* iatt44_ppartkey, size_t* iatt46_pmfgr_offset, char* iatt46_pmfgr_char, size_t* iatt48_ptype_offset, char* iatt48_ptype_char, int* iatt49_psize, multi_ht* jht21, jpayl21* jht21_payload, multi_ht* jht23, jpayl23* jht23_payload) {
    int att44_ppartkey;
    str_t att46_pmfgr;
    str_t att48_ptype;
    int att49_psize;
    str_t c4 = stringConstant ( "%BRASS", 6);
    unsigned warplane = (threadIdx.x % 32);
    int att39_pspartke;
    float att1_minsuppl;

    int tid_part2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_part2 = loopVar;
        active = (loopVar < 200000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att44_ppartkey = iatt44_ppartkey[tid_part2];
            att46_pmfgr = stringScan ( iatt46_pmfgr_offset, iatt46_pmfgr_char, tid_part2);
            att48_ptype = stringScan ( iatt48_ptype_offset, iatt48_ptype_char, tid_part2);
            att49_psize = iatt49_psize[tid_part2];
        }
        // -------- selection (opId: 20) --------
        if(active) {
            active = (stringLikeCheck ( att48_ptype, c4) && (att49_psize == 15));
        }
        // -------- hash join probe (opId: 21) --------
        // -------- multiprobe multi broadcast (opId: 21) --------
        int matchEnd21 = 0;
        int matchEndBuf21 = 0;
        int matchOffset21 = 0;
        int matchOffsetBuf21 = 0;
        int probeActive21 = active;
        int att44_ppartkey_bcbuf21;
        str_t att46_pmfgr_bcbuf21;
        uint64_t hash21 = 0;
        if(probeActive21) {
            hash21 = 0;
            if(active) {
                hash21 = hash ( (hash21 + ((uint64_t)att44_ppartkey)));
            }
            probeActive21 = hashProbeMulti ( jht21, 1600000, hash21, matchOffsetBuf21, matchEndBuf21);
        }
        unsigned activeProbes21 = __ballot_sync(ALL_LANES,probeActive21);
        int num21 = 0;
        num21 = (matchEndBuf21 - matchOffsetBuf21);
        unsigned wideProbes21 = __ballot_sync(ALL_LANES,(num21 >= 32));
        att44_ppartkey_bcbuf21 = att44_ppartkey;
        att46_pmfgr_bcbuf21 = att46_pmfgr;
        while((activeProbes21 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes21)) {
                if((wideProbes21 > 0)) {
                    tupleLane = (__ffs(wideProbes21) - 1);
                    wideProbes21 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes21) - 1);
                }
                num = __shfl_sync(ALL_LANES,num21,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset21 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes21 -= (1 << tupleLane);
            }
            matchOffset21 += __shfl_sync(ALL_LANES,matchOffsetBuf21,broadcastLane);
            matchEnd21 = __shfl_sync(ALL_LANES,matchEndBuf21,broadcastLane);
            att44_ppartkey = __shfl_sync(ALL_LANES,att44_ppartkey_bcbuf21,broadcastLane);
            att46_pmfgr = __shfl_sync(ALL_LANES,att46_pmfgr_bcbuf21,broadcastLane);
            probeActive21 = (matchOffset21 < matchEnd21);
            while(__any_sync(ALL_LANES,probeActive21)) {
                active = probeActive21;
                active = 0;
                jpayl21 payl;
                if(probeActive21) {
                    payl = jht21_payload[matchOffset21];
                    att39_pspartke = payl.att39_pspartke;
                    att1_minsuppl = payl.att1_minsuppl;
                    active = 1;
                    active &= ((att39_pspartke == att44_ppartkey));
                    matchOffset21 += 32;
                    probeActive21 &= ((matchOffset21 < matchEnd21));
                }
                // -------- hash join build (opId: 23) --------
                if(active) {
                    uint64_t hash23 = 0;
                    if(active) {
                        hash23 = 0;
                        if(active) {
                            hash23 = hash ( (hash23 + ((uint64_t)att39_pspartke)));
                        }
                    }
                    hashCountMulti ( jht23, 1600000, hash23);
                }
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_part219_ins(
    int* iatt44_ppartkey, size_t* iatt46_pmfgr_offset, char* iatt46_pmfgr_char, size_t* iatt48_ptype_offset, char* iatt48_ptype_char, int* iatt49_psize, multi_ht* jht21, jpayl21* jht21_payload, multi_ht* jht23, jpayl23* jht23_payload, int* offs23) {
    int att44_ppartkey;
    str_t att46_pmfgr;
    str_t att48_ptype;
    int att49_psize;
    str_t c4 = stringConstant ( "%BRASS", 6);
    unsigned warplane = (threadIdx.x % 32);
    int att39_pspartke;
    float att1_minsuppl;

    int tid_part2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_part2 = loopVar;
        active = (loopVar < 200000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att44_ppartkey = iatt44_ppartkey[tid_part2];
            att46_pmfgr = stringScan ( iatt46_pmfgr_offset, iatt46_pmfgr_char, tid_part2);
            att48_ptype = stringScan ( iatt48_ptype_offset, iatt48_ptype_char, tid_part2);
            att49_psize = iatt49_psize[tid_part2];
        }
        // -------- selection (opId: 20) --------
        if(active) {
            active = (stringLikeCheck ( att48_ptype, c4) && (att49_psize == 15));
        }
        // -------- hash join probe (opId: 21) --------
        // -------- multiprobe multi broadcast (opId: 21) --------
        int matchEnd21 = 0;
        int matchEndBuf21 = 0;
        int matchOffset21 = 0;
        int matchOffsetBuf21 = 0;
        int probeActive21 = active;
        int att44_ppartkey_bcbuf21;
        str_t att46_pmfgr_bcbuf21;
        uint64_t hash21 = 0;
        if(probeActive21) {
            hash21 = 0;
            if(active) {
                hash21 = hash ( (hash21 + ((uint64_t)att44_ppartkey)));
            }
            probeActive21 = hashProbeMulti ( jht21, 1600000, hash21, matchOffsetBuf21, matchEndBuf21);
        }
        unsigned activeProbes21 = __ballot_sync(ALL_LANES,probeActive21);
        int num21 = 0;
        num21 = (matchEndBuf21 - matchOffsetBuf21);
        unsigned wideProbes21 = __ballot_sync(ALL_LANES,(num21 >= 32));
        att44_ppartkey_bcbuf21 = att44_ppartkey;
        att46_pmfgr_bcbuf21 = att46_pmfgr;
        while((activeProbes21 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes21)) {
                if((wideProbes21 > 0)) {
                    tupleLane = (__ffs(wideProbes21) - 1);
                    wideProbes21 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes21) - 1);
                }
                num = __shfl_sync(ALL_LANES,num21,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset21 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes21 -= (1 << tupleLane);
            }
            matchOffset21 += __shfl_sync(ALL_LANES,matchOffsetBuf21,broadcastLane);
            matchEnd21 = __shfl_sync(ALL_LANES,matchEndBuf21,broadcastLane);
            att44_ppartkey = __shfl_sync(ALL_LANES,att44_ppartkey_bcbuf21,broadcastLane);
            att46_pmfgr = __shfl_sync(ALL_LANES,att46_pmfgr_bcbuf21,broadcastLane);
            probeActive21 = (matchOffset21 < matchEnd21);
            while(__any_sync(ALL_LANES,probeActive21)) {
                active = probeActive21;
                active = 0;
                jpayl21 payl;
                if(probeActive21) {
                    payl = jht21_payload[matchOffset21];
                    att39_pspartke = payl.att39_pspartke;
                    att1_minsuppl = payl.att1_minsuppl;
                    active = 1;
                    active &= ((att39_pspartke == att44_ppartkey));
                    matchOffset21 += 32;
                    probeActive21 &= ((matchOffset21 < matchEnd21));
                }
                // -------- hash join build (opId: 23) --------
                if(active) {
                    uint64_t hash23 = 0;
                    if(active) {
                        hash23 = 0;
                        if(active) {
                            hash23 = hash ( (hash23 + ((uint64_t)att39_pspartke)));
                        }
                    }
                    jpayl23 payl;
                    payl.att39_pspartke = att39_pspartke;
                    payl.att1_minsuppl = att1_minsuppl;
                    payl.att44_ppartkey = att44_ppartkey;
                    payl.att46_pmfgr = att46_pmfgr;
                    hashInsertMulti ( jht23, jht23_payload, offs23, 1600000, hash23, &(payl));
                }
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_partsupp222(
    int* iatt53_pspartke, int* iatt54_pssuppke, float* iatt56_pssupply, multi_ht* jht23, jpayl23* jht23_payload, unique_ht<jpayl25>* jht25, int* nout_result, float* oatt14_sacctbal, str_offs* oatt10_sname_offset, char* iatt10_sname_char, str_offs* oatt6_nname_offset, char* iatt6_nname_char, int* oatt44_ppartkey, str_offs* oatt46_pmfgr_offset, char* iatt46_pmfgr_char, str_offs* oatt11_saddress_offset, char* iatt11_saddress_char, str_offs* oatt13_sphone_offset, char* iatt13_sphone_char, str_offs* oatt15_scomment_offset, char* iatt15_scomment_char) {
    int att53_pspartke;
    int att54_pssuppke;
    float att56_pssupply;
    unsigned warplane = (threadIdx.x % 32);
    int att39_pspartke;
    float att1_minsuppl;
    int att44_ppartkey;
    str_t att46_pmfgr;
    str_t att6_nname;
    int att9_ssuppkey;
    str_t att10_sname;
    str_t att11_saddress;
    str_t att13_sphone;
    float att14_sacctbal;
    str_t att15_scomment;
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_partsupp2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_partsupp2 = loopVar;
        active = (loopVar < 800000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att53_pspartke = iatt53_pspartke[tid_partsupp2];
            att54_pssuppke = iatt54_pssuppke[tid_partsupp2];
            att56_pssupply = iatt56_pssupply[tid_partsupp2];
        }
        // -------- hash join probe (opId: 23) --------
        // -------- multiprobe multi broadcast (opId: 23) --------
        int matchEnd23 = 0;
        int matchEndBuf23 = 0;
        int matchOffset23 = 0;
        int matchOffsetBuf23 = 0;
        int probeActive23 = active;
        int att53_pspartke_bcbuf23;
        int att54_pssuppke_bcbuf23;
        float att56_pssupply_bcbuf23;
        uint64_t hash23 = 0;
        if(probeActive23) {
            hash23 = 0;
            if(active) {
                hash23 = hash ( (hash23 + ((uint64_t)att53_pspartke)));
            }
            probeActive23 = hashProbeMulti ( jht23, 1600000, hash23, matchOffsetBuf23, matchEndBuf23);
        }
        unsigned activeProbes23 = __ballot_sync(ALL_LANES,probeActive23);
        int num23 = 0;
        num23 = (matchEndBuf23 - matchOffsetBuf23);
        unsigned wideProbes23 = __ballot_sync(ALL_LANES,(num23 >= 32));
        att53_pspartke_bcbuf23 = att53_pspartke;
        att54_pssuppke_bcbuf23 = att54_pssuppke;
        att56_pssupply_bcbuf23 = att56_pssupply;
        while((activeProbes23 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes23)) {
                if((wideProbes23 > 0)) {
                    tupleLane = (__ffs(wideProbes23) - 1);
                    wideProbes23 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes23) - 1);
                }
                num = __shfl_sync(ALL_LANES,num23,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset23 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes23 -= (1 << tupleLane);
            }
            matchOffset23 += __shfl_sync(ALL_LANES,matchOffsetBuf23,broadcastLane);
            matchEnd23 = __shfl_sync(ALL_LANES,matchEndBuf23,broadcastLane);
            att53_pspartke = __shfl_sync(ALL_LANES,att53_pspartke_bcbuf23,broadcastLane);
            att54_pssuppke = __shfl_sync(ALL_LANES,att54_pssuppke_bcbuf23,broadcastLane);
            att56_pssupply = __shfl_sync(ALL_LANES,att56_pssupply_bcbuf23,broadcastLane);
            probeActive23 = (matchOffset23 < matchEnd23);
            while(__any_sync(ALL_LANES,probeActive23)) {
                active = probeActive23;
                active = 0;
                jpayl23 payl;
                if(probeActive23) {
                    payl = jht23_payload[matchOffset23];
                    att39_pspartke = payl.att39_pspartke;
                    att1_minsuppl = payl.att1_minsuppl;
                    att44_ppartkey = payl.att44_ppartkey;
                    att46_pmfgr = payl.att46_pmfgr;
                    active = 1;
                    active &= ((att39_pspartke == att53_pspartke));
                    matchOffset23 += 32;
                    probeActive23 &= ((matchOffset23 < matchEnd23));
                }
                // -------- selection (opId: 24) --------
                if(active) {
                    active = (att1_minsuppl == att56_pssupply);
                }
                // -------- hash join probe (opId: 25) --------
                uint64_t hash25 = 0;
                if(active) {
                    hash25 = 0;
                    if(active) {
                        hash25 = hash ( (hash25 + ((uint64_t)att54_pssuppke)));
                    }
                }
                jpayl25* probepayl25;
                int numLookups25 = 0;
                if(active) {
                    active = hashProbeUnique ( jht25, 20000, hash25, numLookups25, &(probepayl25));
                }
                int bucketFound25 = 0;
                int probeActive25 = active;
                while((probeActive25 && !(bucketFound25))) {
                    jpayl25 jprobepayl25 = *(probepayl25);
                    att6_nname = jprobepayl25.att6_nname;
                    att9_ssuppkey = jprobepayl25.att9_ssuppkey;
                    att10_sname = jprobepayl25.att10_sname;
                    att11_saddress = jprobepayl25.att11_saddress;
                    att13_sphone = jprobepayl25.att13_sphone;
                    att14_sacctbal = jprobepayl25.att14_sacctbal;
                    att15_scomment = jprobepayl25.att15_scomment;
                    bucketFound25 = 1;
                    bucketFound25 &= ((att9_ssuppkey == att54_pssuppke));
                    if(!(bucketFound25)) {
                        probeActive25 = hashProbeUnique ( jht25, 20000, hash25, numLookups25, &(probepayl25));
                    }
                }
                active = bucketFound25;
                // -------- projection (no code) (opId: 26) --------
                // -------- materialize (opId: 27) --------
                int wp;
                int writeMask;
                int numProj;
                writeMask = __ballot_sync(ALL_LANES,active);
                numProj = __popc(writeMask);
                if((warplane == 0)) {
                    wp = atomicAdd(nout_result, numProj);
                }
                wp = __shfl_sync(ALL_LANES,wp,0);
                wp = (wp + __popc((writeMask & prefixlanes)));
                if(active) {
                    oatt14_sacctbal[wp] = att14_sacctbal;
                    oatt10_sname_offset[wp] = toStringOffset ( iatt10_sname_char, att10_sname);
                    oatt6_nname_offset[wp] = toStringOffset ( iatt6_nname_char, att6_nname);
                    oatt44_ppartkey[wp] = att44_ppartkey;
                    oatt46_pmfgr_offset[wp] = toStringOffset ( iatt46_pmfgr_char, att46_pmfgr);
                    oatt11_saddress_offset[wp] = toStringOffset ( iatt11_saddress_char, att11_saddress);
                    oatt13_sphone_offset[wp] = toStringOffset ( iatt13_sphone_char, att13_sphone);
                    oatt15_scomment_offset[wp] = toStringOffset ( iatt15_scomment_char, att15_scomment);
                }
            }
        }
        loopVar += step;
    }

}

int main() {
    int* iatt2_rregionk;
    iatt2_rregionk = ( int*) map_memory_file ( "mmdb/region_r_regionkey" );
    size_t* iatt3_rname_offset;
    iatt3_rname_offset = ( size_t*) map_memory_file ( "mmdb/region_r_name_offset" );
    char* iatt3_rname_char;
    iatt3_rname_char = ( char*) map_memory_file ( "mmdb/region_r_name_char" );
    int* iatt5_nnationk;
    iatt5_nnationk = ( int*) map_memory_file ( "mmdb/nation_n_nationkey" );
    size_t* iatt6_nname_offset;
    iatt6_nname_offset = ( size_t*) map_memory_file ( "mmdb/nation_n_name_offset" );
    char* iatt6_nname_char;
    iatt6_nname_char = ( char*) map_memory_file ( "mmdb/nation_n_name_char" );
    int* iatt7_nregionk;
    iatt7_nregionk = ( int*) map_memory_file ( "mmdb/nation_n_regionkey" );
    int* iatt9_ssuppkey;
    iatt9_ssuppkey = ( int*) map_memory_file ( "mmdb/supplier_s_suppkey" );
    size_t* iatt10_sname_offset;
    iatt10_sname_offset = ( size_t*) map_memory_file ( "mmdb/supplier_s_name_offset" );
    char* iatt10_sname_char;
    iatt10_sname_char = ( char*) map_memory_file ( "mmdb/supplier_s_name_char" );
    size_t* iatt11_saddress_offset;
    iatt11_saddress_offset = ( size_t*) map_memory_file ( "mmdb/supplier_s_address_offset" );
    char* iatt11_saddress_char;
    iatt11_saddress_char = ( char*) map_memory_file ( "mmdb/supplier_s_address_char" );
    int* iatt12_snationk;
    iatt12_snationk = ( int*) map_memory_file ( "mmdb/supplier_s_nationkey" );
    size_t* iatt13_sphone_offset;
    iatt13_sphone_offset = ( size_t*) map_memory_file ( "mmdb/supplier_s_phone_offset" );
    char* iatt13_sphone_char;
    iatt13_sphone_char = ( char*) map_memory_file ( "mmdb/supplier_s_phone_char" );
    float* iatt14_sacctbal;
    iatt14_sacctbal = ( float*) map_memory_file ( "mmdb/supplier_s_acctbal" );
    size_t* iatt15_scomment_offset;
    iatt15_scomment_offset = ( size_t*) map_memory_file ( "mmdb/supplier_s_comment_offset" );
    char* iatt15_scomment_char;
    iatt15_scomment_char = ( char*) map_memory_file ( "mmdb/supplier_s_comment_char" );
    int* iatt16_rregionk;
    iatt16_rregionk = ( int*) map_memory_file ( "mmdb/region_r_regionkey" );
    size_t* iatt17_rname_offset;
    iatt17_rname_offset = ( size_t*) map_memory_file ( "mmdb/region_r_name_offset" );
    char* iatt17_rname_char;
    iatt17_rname_char = ( char*) map_memory_file ( "mmdb/region_r_name_char" );
    int* iatt19_nnationk;
    iatt19_nnationk = ( int*) map_memory_file ( "mmdb/nation_n_nationkey" );
    int* iatt21_nregionk;
    iatt21_nregionk = ( int*) map_memory_file ( "mmdb/nation_n_regionkey" );
    int* iatt23_ssuppkey;
    iatt23_ssuppkey = ( int*) map_memory_file ( "mmdb/supplier_s_suppkey" );
    int* iatt26_snationk;
    iatt26_snationk = ( int*) map_memory_file ( "mmdb/supplier_s_nationkey" );
    int* iatt30_ppartkey;
    iatt30_ppartkey = ( int*) map_memory_file ( "mmdb/part_p_partkey" );
    size_t* iatt34_ptype_offset;
    iatt34_ptype_offset = ( size_t*) map_memory_file ( "mmdb/part_p_type_offset" );
    char* iatt34_ptype_char;
    iatt34_ptype_char = ( char*) map_memory_file ( "mmdb/part_p_type_char" );
    int* iatt35_psize;
    iatt35_psize = ( int*) map_memory_file ( "mmdb/part_p_size" );
    int* iatt39_pspartke;
    iatt39_pspartke = ( int*) map_memory_file ( "mmdb/partsupp_ps_partkey" );
    int* iatt40_pssuppke;
    iatt40_pssuppke = ( int*) map_memory_file ( "mmdb/partsupp_ps_suppkey" );
    float* iatt42_pssupply;
    iatt42_pssupply = ( float*) map_memory_file ( "mmdb/partsupp_ps_supplycost" );
    int* iatt44_ppartkey;
    iatt44_ppartkey = ( int*) map_memory_file ( "mmdb/part_p_partkey" );
    size_t* iatt46_pmfgr_offset;
    iatt46_pmfgr_offset = ( size_t*) map_memory_file ( "mmdb/part_p_mfgr_offset" );
    char* iatt46_pmfgr_char;
    iatt46_pmfgr_char = ( char*) map_memory_file ( "mmdb/part_p_mfgr_char" );
    size_t* iatt48_ptype_offset;
    iatt48_ptype_offset = ( size_t*) map_memory_file ( "mmdb/part_p_type_offset" );
    char* iatt48_ptype_char;
    iatt48_ptype_char = ( char*) map_memory_file ( "mmdb/part_p_type_char" );
    int* iatt49_psize;
    iatt49_psize = ( int*) map_memory_file ( "mmdb/part_p_size" );
    int* iatt53_pspartke;
    iatt53_pspartke = ( int*) map_memory_file ( "mmdb/partsupp_ps_partkey" );
    int* iatt54_pssuppke;
    iatt54_pssuppke = ( int*) map_memory_file ( "mmdb/partsupp_ps_suppkey" );
    float* iatt56_pssupply;
    iatt56_pssupply = ( float*) map_memory_file ( "mmdb/partsupp_ps_supplycost" );

    int nout_result;
    std::vector < float > oatt14_sacctbal(800000);
    std::vector < str_offs > oatt10_sname_offset(800000);
    std::vector < str_offs > oatt6_nname_offset(800000);
    std::vector < int > oatt44_ppartkey(800000);
    std::vector < str_offs > oatt46_pmfgr_offset(800000);
    std::vector < str_offs > oatt11_saddress_offset(800000);
    std::vector < str_offs > oatt13_sphone_offset(800000);
    std::vector < str_offs > oatt15_scomment_offset(800000);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt2_rregionk;
    hipMalloc((void**) &d_iatt2_rregionk, 5* sizeof(int) );
    size_t* d_iatt3_rname_offset;
    hipMalloc((void**) &d_iatt3_rname_offset, (5 + 1)* sizeof(size_t) );
    char* d_iatt3_rname_char;
    hipMalloc((void**) &d_iatt3_rname_char, 43* sizeof(char) );
    int* d_iatt5_nnationk;
    hipMalloc((void**) &d_iatt5_nnationk, 25* sizeof(int) );
    size_t* d_iatt6_nname_offset;
    hipMalloc((void**) &d_iatt6_nname_offset, (25 + 1)* sizeof(size_t) );
    char* d_iatt6_nname_char;
    hipMalloc((void**) &d_iatt6_nname_char, 186* sizeof(char) );
    int* d_iatt7_nregionk;
    hipMalloc((void**) &d_iatt7_nregionk, 25* sizeof(int) );
    int* d_iatt9_ssuppkey;
    hipMalloc((void**) &d_iatt9_ssuppkey, 10000* sizeof(int) );
    size_t* d_iatt10_sname_offset;
    hipMalloc((void**) &d_iatt10_sname_offset, (10000 + 1)* sizeof(size_t) );
    char* d_iatt10_sname_char;
    hipMalloc((void**) &d_iatt10_sname_char, 180009* sizeof(char) );
    size_t* d_iatt11_saddress_offset;
    hipMalloc((void**) &d_iatt11_saddress_offset, (10000 + 1)* sizeof(size_t) );
    char* d_iatt11_saddress_char;
    hipMalloc((void**) &d_iatt11_saddress_char, 249461* sizeof(char) );
    int* d_iatt12_snationk;
    hipMalloc((void**) &d_iatt12_snationk, 10000* sizeof(int) );
    size_t* d_iatt13_sphone_offset;
    hipMalloc((void**) &d_iatt13_sphone_offset, (10000 + 1)* sizeof(size_t) );
    char* d_iatt13_sphone_char;
    hipMalloc((void**) &d_iatt13_sphone_char, 150009* sizeof(char) );
    float* d_iatt14_sacctbal;
    hipMalloc((void**) &d_iatt14_sacctbal, 10000* sizeof(float) );
    size_t* d_iatt15_scomment_offset;
    hipMalloc((void**) &d_iatt15_scomment_offset, (10000 + 1)* sizeof(size_t) );
    char* d_iatt15_scomment_char;
    hipMalloc((void**) &d_iatt15_scomment_char, 623073* sizeof(char) );
    int* d_iatt16_rregionk;
    d_iatt16_rregionk = d_iatt2_rregionk;
    size_t* d_iatt17_rname_offset;
    d_iatt17_rname_offset = d_iatt3_rname_offset;
    char* d_iatt17_rname_char;
    d_iatt17_rname_char = d_iatt3_rname_char;
    int* d_iatt19_nnationk;
    d_iatt19_nnationk = d_iatt5_nnationk;
    int* d_iatt21_nregionk;
    d_iatt21_nregionk = d_iatt7_nregionk;
    int* d_iatt23_ssuppkey;
    d_iatt23_ssuppkey = d_iatt9_ssuppkey;
    int* d_iatt26_snationk;
    d_iatt26_snationk = d_iatt12_snationk;
    int* d_iatt30_ppartkey;
    hipMalloc((void**) &d_iatt30_ppartkey, 200000* sizeof(int) );
    size_t* d_iatt34_ptype_offset;
    hipMalloc((void**) &d_iatt34_ptype_offset, (200000 + 1)* sizeof(size_t) );
    char* d_iatt34_ptype_char;
    hipMalloc((void**) &d_iatt34_ptype_char, 4119955* sizeof(char) );
    int* d_iatt35_psize;
    hipMalloc((void**) &d_iatt35_psize, 200000* sizeof(int) );
    int* d_iatt39_pspartke;
    hipMalloc((void**) &d_iatt39_pspartke, 800000* sizeof(int) );
    int* d_iatt40_pssuppke;
    hipMalloc((void**) &d_iatt40_pssuppke, 800000* sizeof(int) );
    float* d_iatt42_pssupply;
    hipMalloc((void**) &d_iatt42_pssupply, 800000* sizeof(float) );
    int* d_iatt44_ppartkey;
    d_iatt44_ppartkey = d_iatt30_ppartkey;
    size_t* d_iatt46_pmfgr_offset;
    hipMalloc((void**) &d_iatt46_pmfgr_offset, (200000 + 1)* sizeof(size_t) );
    char* d_iatt46_pmfgr_char;
    hipMalloc((void**) &d_iatt46_pmfgr_char, 2800009* sizeof(char) );
    size_t* d_iatt48_ptype_offset;
    d_iatt48_ptype_offset = d_iatt34_ptype_offset;
    char* d_iatt48_ptype_char;
    d_iatt48_ptype_char = d_iatt34_ptype_char;
    int* d_iatt49_psize;
    d_iatt49_psize = d_iatt35_psize;
    int* d_iatt53_pspartke;
    d_iatt53_pspartke = d_iatt39_pspartke;
    int* d_iatt54_pssuppke;
    d_iatt54_pssuppke = d_iatt40_pssuppke;
    float* d_iatt56_pssupply;
    d_iatt56_pssupply = d_iatt42_pssupply;
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    float* d_oatt14_sacctbal;
    hipMalloc((void**) &d_oatt14_sacctbal, 800000* sizeof(float) );
    str_offs* d_oatt10_sname_offset;
    hipMalloc((void**) &d_oatt10_sname_offset, 800000* sizeof(str_offs) );
    str_offs* d_oatt6_nname_offset;
    hipMalloc((void**) &d_oatt6_nname_offset, 800000* sizeof(str_offs) );
    int* d_oatt44_ppartkey;
    hipMalloc((void**) &d_oatt44_ppartkey, 800000* sizeof(int) );
    str_offs* d_oatt46_pmfgr_offset;
    hipMalloc((void**) &d_oatt46_pmfgr_offset, 800000* sizeof(str_offs) );
    str_offs* d_oatt11_saddress_offset;
    hipMalloc((void**) &d_oatt11_saddress_offset, 800000* sizeof(str_offs) );
    str_offs* d_oatt13_sphone_offset;
    hipMalloc((void**) &d_oatt13_sphone_offset, 800000* sizeof(str_offs) );
    str_offs* d_oatt15_scomment_offset;
    hipMalloc((void**) &d_oatt15_scomment_offset, 800000* sizeof(str_offs) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    unique_ht<jpayl4>* d_jht4;
    hipMalloc((void**) &d_jht4, 10* sizeof(unique_ht<jpayl4>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht4, 10);
    }
    multi_ht* d_jht6;
    hipMalloc((void**) &d_jht6, 50* sizeof(multi_ht) );
    jpayl6* d_jht6_payload;
    hipMalloc((void**) &d_jht6_payload, 50* sizeof(jpayl6) );
    {
        int gridsize=920;
        int blocksize=128;
        initMultiHT<<<gridsize, blocksize>>>(d_jht6, 50);
    }
    int* d_offs6;
    hipMalloc((void**) &d_offs6, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs6, 0, 1);
    }
    unique_ht<jpayl25>* d_jht25;
    hipMalloc((void**) &d_jht25, 20000* sizeof(unique_ht<jpayl25>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht25, 20000);
    }
    unique_ht<jpayl10>* d_jht10;
    hipMalloc((void**) &d_jht10, 10* sizeof(unique_ht<jpayl10>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht10, 10);
    }
    unique_ht<jpayl12>* d_jht12;
    hipMalloc((void**) &d_jht12, 50* sizeof(unique_ht<jpayl12>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht12, 50);
    }
    unique_ht<jpayl17>* d_jht17;
    hipMalloc((void**) &d_jht17, 20000* sizeof(unique_ht<jpayl17>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht17, 20000);
    }
    agg_ht<jpayl16>* d_jht16;
    hipMalloc((void**) &d_jht16, 400000* sizeof(agg_ht<jpayl16>) );
    {
        int gridsize=920;
        int blocksize=128;
        initAggHT<<<gridsize, blocksize>>>(d_jht16, 400000);
    }
    agg_ht<apayl18>* d_aht18;
    hipMalloc((void**) &d_aht18, 1600000* sizeof(agg_ht<apayl18>) );
    {
        int gridsize=920;
        int blocksize=128;
        initAggHT<<<gridsize, blocksize>>>(d_aht18, 1600000);
    }
    float* d_agg1;
    hipMalloc((void**) &d_agg1, 1600000* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg1, FLT_MAX, 1600000);
    }
    multi_ht* d_jht21;
    hipMalloc((void**) &d_jht21, 1600000* sizeof(multi_ht) );
    jpayl21* d_jht21_payload;
    hipMalloc((void**) &d_jht21_payload, 1600000* sizeof(jpayl21) );
    {
        int gridsize=920;
        int blocksize=128;
        initMultiHT<<<gridsize, blocksize>>>(d_jht21, 1600000);
    }
    int* d_offs21;
    hipMalloc((void**) &d_offs21, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs21, 0, 1);
    }
    multi_ht* d_jht23;
    hipMalloc((void**) &d_jht23, 1600000* sizeof(multi_ht) );
    jpayl23* d_jht23_payload;
    hipMalloc((void**) &d_jht23_payload, 1600000* sizeof(jpayl23) );
    {
        int gridsize=920;
        int blocksize=128;
        initMultiHT<<<gridsize, blocksize>>>(d_jht23, 1600000);
    }
    int* d_offs23;
    hipMalloc((void**) &d_offs23, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs23, 0, 1);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt2_rregionk, iatt2_rregionk, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt3_rname_offset, iatt3_rname_offset, (5 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt3_rname_char, iatt3_rname_char, 43 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt5_nnationk, iatt5_nnationk, 25 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt6_nname_offset, iatt6_nname_offset, (25 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt6_nname_char, iatt6_nname_char, 186 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt7_nregionk, iatt7_nregionk, 25 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt9_ssuppkey, iatt9_ssuppkey, 10000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt10_sname_offset, iatt10_sname_offset, (10000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt10_sname_char, iatt10_sname_char, 180009 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt11_saddress_offset, iatt11_saddress_offset, (10000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt11_saddress_char, iatt11_saddress_char, 249461 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt12_snationk, iatt12_snationk, 10000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt13_sphone_offset, iatt13_sphone_offset, (10000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt13_sphone_char, iatt13_sphone_char, 150009 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt14_sacctbal, iatt14_sacctbal, 10000 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt15_scomment_offset, iatt15_scomment_offset, (10000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt15_scomment_char, iatt15_scomment_char, 623073 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt30_ppartkey, iatt30_ppartkey, 200000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt34_ptype_offset, iatt34_ptype_offset, (200000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt34_ptype_char, iatt34_ptype_char, 4119955 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt35_psize, iatt35_psize, 200000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt39_pspartke, iatt39_pspartke, 800000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt40_pssuppke, iatt40_pssuppke, 800000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt42_pssupply, iatt42_pssupply, 800000 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt46_pmfgr_offset, iatt46_pmfgr_offset, (200000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt46_pmfgr_char, iatt46_pmfgr_char, 2800009 * sizeof(char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime4 = std::clock();
    std::clock_t start_krnl_region15 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_region1<<<gridsize, blocksize>>>(d_iatt2_rregionk, d_iatt3_rname_offset, d_iatt3_rname_char, d_jht4);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_region15 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_region1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_region1")
        }
    }

    std::clock_t start_krnl_nation36 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_nation3<<<gridsize, blocksize>>>(d_iatt5_nnationk, d_iatt6_nname_offset, d_iatt6_nname_char, d_iatt7_nregionk, d_jht4, d_jht6, d_jht6_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation36 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation3! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation3")
        }
    }

    std::clock_t start_scanMultiHT7 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht6, 50, d_offs6);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT7 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT")
        }
    }

    std::clock_t start_krnl_nation3_ins8 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_nation3_ins<<<gridsize, blocksize>>>(d_iatt5_nnationk, d_iatt6_nname_offset, d_iatt6_nname_char, d_iatt7_nregionk, d_jht4, d_jht6, d_jht6_payload, d_offs6);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation3_ins8 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation3_ins! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation3_ins")
        }
    }

    std::clock_t start_krnl_supplier59 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_supplier5<<<gridsize, blocksize>>>(d_iatt9_ssuppkey, d_iatt10_sname_offset, d_iatt10_sname_char, d_iatt11_saddress_offset, d_iatt11_saddress_char, d_iatt12_snationk, d_iatt13_sphone_offset, d_iatt13_sphone_char, d_iatt14_sacctbal, d_iatt15_scomment_offset, d_iatt15_scomment_char, d_jht6, d_jht6_payload, d_jht25);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_supplier59 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_supplier5! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_supplier5")
        }
    }

    std::clock_t start_krnl_region2710 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_region27<<<gridsize, blocksize>>>(d_iatt16_rregionk, d_iatt17_rname_offset, d_iatt17_rname_char, d_jht10);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_region2710 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_region27! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_region27")
        }
    }

    std::clock_t start_krnl_nation2911 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_nation29<<<gridsize, blocksize>>>(d_iatt19_nnationk, d_iatt21_nregionk, d_jht10, d_jht12);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation2911 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation29! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation29")
        }
    }

    std::clock_t start_krnl_supplier21112 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_supplier211<<<gridsize, blocksize>>>(d_iatt23_ssuppkey, d_iatt26_snationk, d_jht12, d_jht17);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_supplier21112 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_supplier211! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_supplier211")
        }
    }

    std::clock_t start_krnl_part1313 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_part13<<<gridsize, blocksize>>>(d_iatt30_ppartkey, d_iatt34_ptype_offset, d_iatt34_ptype_char, d_iatt35_psize, d_jht16);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_part1313 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_part13! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_part13")
        }
    }

    std::clock_t start_krnl_partsupp1514 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_partsupp15<<<gridsize, blocksize>>>(d_iatt39_pspartke, d_iatt40_pssuppke, d_iatt42_pssupply, d_jht16, d_jht17, d_aht18, d_agg1);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_partsupp1514 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_partsupp15! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_partsupp15")
        }
    }

    std::clock_t start_krnl_aggregation1815 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation18<<<gridsize, blocksize>>>(d_aht18, d_agg1, d_jht21, d_jht21_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation1815 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation18! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation18")
        }
    }

    std::clock_t start_scanMultiHT16 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht21, 1600000, d_offs21);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT16 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT")
        }
    }

    std::clock_t start_krnl_aggregation18_ins17 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation18_ins<<<gridsize, blocksize>>>(d_aht18, d_agg1, d_jht21, d_jht21_payload, d_offs21);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation18_ins17 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation18_ins! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation18_ins")
        }
    }

    std::clock_t start_krnl_part21918 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_part219<<<gridsize, blocksize>>>(d_iatt44_ppartkey, d_iatt46_pmfgr_offset, d_iatt46_pmfgr_char, d_iatt48_ptype_offset, d_iatt48_ptype_char, d_iatt49_psize, d_jht21, d_jht21_payload, d_jht23, d_jht23_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_part21918 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_part219! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_part219")
        }
    }

    std::clock_t start_scanMultiHT19 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht23, 1600000, d_offs23);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT19 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT")
        }
    }

    std::clock_t start_krnl_part219_ins20 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_part219_ins<<<gridsize, blocksize>>>(d_iatt44_ppartkey, d_iatt46_pmfgr_offset, d_iatt46_pmfgr_char, d_iatt48_ptype_offset, d_iatt48_ptype_char, d_iatt49_psize, d_jht21, d_jht21_payload, d_jht23, d_jht23_payload, d_offs23);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_part219_ins20 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_part219_ins! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_part219_ins")
        }
    }

    std::clock_t start_krnl_partsupp22221 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_partsupp222<<<gridsize, blocksize>>>(d_iatt53_pspartke, d_iatt54_pssuppke, d_iatt56_pssupply, d_jht23, d_jht23_payload, d_jht25, d_nout_result, d_oatt14_sacctbal, d_oatt10_sname_offset, d_iatt10_sname_char, d_oatt6_nname_offset, d_iatt6_nname_char, d_oatt44_ppartkey, d_oatt46_pmfgr_offset, d_iatt46_pmfgr_char, d_oatt11_saddress_offset, d_iatt11_saddress_char, d_oatt13_sphone_offset, d_iatt13_sphone_char, d_oatt15_scomment_offset, d_iatt15_scomment_char);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_partsupp22221 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_partsupp222! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_partsupp222")
        }
    }

    std::clock_t stop_totalKernelTime4 = std::clock();
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt14_sacctbal.data(), d_oatt14_sacctbal, 800000 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( oatt10_sname_offset.data(), d_oatt10_sname_offset, 800000 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt6_nname_offset.data(), d_oatt6_nname_offset, 800000 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt44_ppartkey.data(), d_oatt44_ppartkey, 800000 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt46_pmfgr_offset.data(), d_oatt46_pmfgr_offset, 800000 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt11_saddress_offset.data(), d_oatt11_saddress_offset, 800000 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt13_sphone_offset.data(), d_oatt13_sphone_offset, 800000 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt15_scomment_offset.data(), d_oatt15_scomment_offset, 800000 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt2_rregionk);
    hipFree( d_iatt3_rname_offset);
    hipFree( d_iatt3_rname_char);
    hipFree( d_jht4);
    hipFree( d_iatt5_nnationk);
    hipFree( d_iatt6_nname_offset);
    hipFree( d_iatt6_nname_char);
    hipFree( d_iatt7_nregionk);
    hipFree( d_jht6);
    hipFree( d_jht6_payload);
    hipFree( d_offs6);
    hipFree( d_iatt9_ssuppkey);
    hipFree( d_iatt10_sname_offset);
    hipFree( d_iatt10_sname_char);
    hipFree( d_iatt11_saddress_offset);
    hipFree( d_iatt11_saddress_char);
    hipFree( d_iatt12_snationk);
    hipFree( d_iatt13_sphone_offset);
    hipFree( d_iatt13_sphone_char);
    hipFree( d_iatt14_sacctbal);
    hipFree( d_iatt15_scomment_offset);
    hipFree( d_iatt15_scomment_char);
    hipFree( d_jht25);
    hipFree( d_jht10);
    hipFree( d_jht12);
    hipFree( d_jht17);
    hipFree( d_iatt30_ppartkey);
    hipFree( d_iatt34_ptype_offset);
    hipFree( d_iatt34_ptype_char);
    hipFree( d_iatt35_psize);
    hipFree( d_jht16);
    hipFree( d_iatt39_pspartke);
    hipFree( d_iatt40_pssuppke);
    hipFree( d_iatt42_pssupply);
    hipFree( d_aht18);
    hipFree( d_agg1);
    hipFree( d_jht21);
    hipFree( d_jht21_payload);
    hipFree( d_offs21);
    hipFree( d_iatt46_pmfgr_offset);
    hipFree( d_iatt46_pmfgr_char);
    hipFree( d_jht23);
    hipFree( d_jht23_payload);
    hipFree( d_offs23);
    hipFree( d_nout_result);
    hipFree( d_oatt14_sacctbal);
    hipFree( d_oatt10_sname_offset);
    hipFree( d_oatt6_nname_offset);
    hipFree( d_oatt44_ppartkey);
    hipFree( d_oatt46_pmfgr_offset);
    hipFree( d_oatt11_saddress_offset);
    hipFree( d_oatt13_sphone_offset);
    hipFree( d_oatt15_scomment_offset);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish22 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 800000)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("s_acctbal: ");
        printf("%15.2f", oatt14_sacctbal[pv]);
        printf("  ");
        printf("s_name: ");
        stringPrint ( iatt10_sname_char, oatt10_sname_offset[pv]);
        printf("  ");
        printf("n_name: ");
        stringPrint ( iatt6_nname_char, oatt6_nname_offset[pv]);
        printf("  ");
        printf("p_partkey: ");
        printf("%8i", oatt44_ppartkey[pv]);
        printf("  ");
        printf("p_mfgr: ");
        stringPrint ( iatt46_pmfgr_char, oatt46_pmfgr_offset[pv]);
        printf("  ");
        printf("s_address: ");
        stringPrint ( iatt11_saddress_char, oatt11_saddress_offset[pv]);
        printf("  ");
        printf("s_phone: ");
        stringPrint ( iatt13_sphone_char, oatt13_sphone_offset[pv]);
        printf("  ");
        printf("s_comment: ");
        stringPrint ( iatt15_scomment_char, oatt15_scomment_offset[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish22 = std::clock();

    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "krnl_region1", (stop_krnl_region15 - start_krnl_region15) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_nation3", (stop_krnl_nation36 - start_krnl_nation36) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "scanMultiHT", (stop_scanMultiHT7 - start_scanMultiHT7) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_nation3_ins", (stop_krnl_nation3_ins8 - start_krnl_nation3_ins8) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_supplier5", (stop_krnl_supplier59 - start_krnl_supplier59) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_region27", (stop_krnl_region2710 - start_krnl_region2710) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_nation29", (stop_krnl_nation2911 - start_krnl_nation2911) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_supplier211", (stop_krnl_supplier21112 - start_krnl_supplier21112) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_part13", (stop_krnl_part1313 - start_krnl_part1313) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_partsupp15", (stop_krnl_partsupp1514 - start_krnl_partsupp1514) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation18", (stop_krnl_aggregation1815 - start_krnl_aggregation1815) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "scanMultiHT", (stop_scanMultiHT16 - start_scanMultiHT16) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation18_ins", (stop_krnl_aggregation18_ins17 - start_krnl_aggregation18_ins17) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_part219", (stop_krnl_part21918 - start_krnl_part21918) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "scanMultiHT", (stop_scanMultiHT19 - start_scanMultiHT19) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_part219_ins", (stop_krnl_part219_ins20 - start_krnl_part219_ins20) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_partsupp222", (stop_krnl_partsupp22221 - start_krnl_partsupp22221) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish22 - start_finish22) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime4 - start_totalKernelTime4) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
