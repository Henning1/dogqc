#include "hip/hip_runtime.h"
#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct jpayl18 {
    int att4_nnationk;
    str_t att5_nname;
};
struct jpayl5 {
    int att8_rregionk;
};
struct jpayl15 {
    int att11_nnationk;
};
struct jpayl9 {
    int att15_ppartkey;
};
struct jpayl12 {
    int att24_lorderke;
    int att26_lsuppkey;
    float att29_lextende;
    float att30_ldiscoun;
};
struct jpayl14 {
    int att26_lsuppkey;
    float att29_lextende;
    float att30_ldiscoun;
    int att41_ocustkey;
    unsigned att44_oorderda;
};
struct jpayl17 {
    int att26_lsuppkey;
    float att29_lextende;
    float att30_ldiscoun;
    unsigned att44_oorderda;
};
struct apayl22 {
    unsigned att64_oyear;
};

__global__ void krnl_nation1(
    int* iatt4_nnationk, size_t* iatt5_nname_offset, char* iatt5_nname_char, multi_ht* jht18, jpayl18* jht18_payload) {
    int att4_nnationk;
    str_t att5_nname;

    int tid_nation1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation1 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att4_nnationk = iatt4_nnationk[tid_nation1];
            att5_nname = stringScan ( iatt5_nname_offset, iatt5_nname_char, tid_nation1);
        }
        // -------- hash join build (opId: 18) --------
        if(active) {
            uint64_t hash18 = 0;
            if(active) {
                hash18 = 0;
                if(active) {
                    hash18 = hash ( (hash18 + ((uint64_t)att4_nnationk)));
                }
            }
            hashCountMulti ( jht18, 50, hash18);
        }
        loopVar += step;
    }

}

__global__ void krnl_nation1_ins(
    int* iatt4_nnationk, size_t* iatt5_nname_offset, char* iatt5_nname_char, multi_ht* jht18, jpayl18* jht18_payload, int* offs18) {
    int att4_nnationk;
    str_t att5_nname;

    int tid_nation1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation1 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att4_nnationk = iatt4_nnationk[tid_nation1];
            att5_nname = stringScan ( iatt5_nname_offset, iatt5_nname_char, tid_nation1);
        }
        // -------- hash join build (opId: 18) --------
        if(active) {
            uint64_t hash18 = 0;
            if(active) {
                hash18 = 0;
                if(active) {
                    hash18 = hash ( (hash18 + ((uint64_t)att4_nnationk)));
                }
            }
            jpayl18 payl;
            payl.att4_nnationk = att4_nnationk;
            payl.att5_nname = att5_nname;
            hashInsertMulti ( jht18, jht18_payload, offs18, 50, hash18, &(payl));
        }
        loopVar += step;
    }

}

__global__ void krnl_region2(
    int* iatt8_rregionk, size_t* iatt9_rname_offset, char* iatt9_rname_char, unique_ht<jpayl5>* jht5) {
    int att8_rregionk;
    str_t att9_rname;
    str_t c1 = stringConstant ( "AMERICA", 7);

    int tid_region1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_region1 = loopVar;
        active = (loopVar < 5);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att8_rregionk = iatt8_rregionk[tid_region1];
            att9_rname = stringScan ( iatt9_rname_offset, iatt9_rname_char, tid_region1);
        }
        // -------- selection (opId: 3) --------
        if(active) {
            active = stringEquals ( att9_rname, c1);
        }
        // -------- hash join build (opId: 5) --------
        if(active) {
            jpayl5 payl5;
            payl5.att8_rregionk = att8_rregionk;
            uint64_t hash5;
            hash5 = 0;
            if(active) {
                hash5 = hash ( (hash5 + ((uint64_t)att8_rregionk)));
            }
            hashBuildUnique ( jht5, 10, hash5, &(payl5));
        }
        loopVar += step;
    }

}

__global__ void krnl_nation24(
    int* iatt11_nnationk, int* iatt13_nregionk, unique_ht<jpayl5>* jht5, unique_ht<jpayl15>* jht15) {
    int att11_nnationk;
    int att13_nregionk;
    int att8_rregionk;

    int tid_nation2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation2 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att11_nnationk = iatt11_nnationk[tid_nation2];
            att13_nregionk = iatt13_nregionk[tid_nation2];
        }
        // -------- hash join probe (opId: 5) --------
        uint64_t hash5 = 0;
        if(active) {
            hash5 = 0;
            if(active) {
                hash5 = hash ( (hash5 + ((uint64_t)att13_nregionk)));
            }
        }
        jpayl5* probepayl5;
        int numLookups5 = 0;
        if(active) {
            active = hashProbeUnique ( jht5, 10, hash5, numLookups5, &(probepayl5));
        }
        int bucketFound5 = 0;
        int probeActive5 = active;
        while((probeActive5 && !(bucketFound5))) {
            jpayl5 jprobepayl5 = *(probepayl5);
            att8_rregionk = jprobepayl5.att8_rregionk;
            bucketFound5 = 1;
            bucketFound5 &= ((att8_rregionk == att13_nregionk));
            if(!(bucketFound5)) {
                probeActive5 = hashProbeUnique ( jht5, 10, hash5, numLookups5, &(probepayl5));
            }
        }
        active = bucketFound5;
        // -------- hash join build (opId: 15) --------
        if(active) {
            jpayl15 payl15;
            payl15.att11_nnationk = att11_nnationk;
            uint64_t hash15;
            hash15 = 0;
            if(active) {
                hash15 = hash ( (hash15 + ((uint64_t)att11_nnationk)));
            }
            hashBuildUnique ( jht15, 50, hash15, &(payl15));
        }
        loopVar += step;
    }

}

__global__ void krnl_part6(
    int* iatt15_ppartkey, size_t* iatt19_ptype_offset, char* iatt19_ptype_char, unique_ht<jpayl9>* jht9) {
    int att15_ppartkey;
    str_t att19_ptype;
    str_t c2 = stringConstant ( "ECONOMY ANODIZED STEEL", 22);

    int tid_part1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_part1 = loopVar;
        active = (loopVar < 200000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att15_ppartkey = iatt15_ppartkey[tid_part1];
            att19_ptype = stringScan ( iatt19_ptype_offset, iatt19_ptype_char, tid_part1);
        }
        // -------- selection (opId: 7) --------
        if(active) {
            active = stringEquals ( att19_ptype, c2);
        }
        // -------- hash join build (opId: 9) --------
        if(active) {
            jpayl9 payl9;
            payl9.att15_ppartkey = att15_ppartkey;
            uint64_t hash9;
            hash9 = 0;
            if(active) {
                hash9 = hash ( (hash9 + ((uint64_t)att15_ppartkey)));
            }
            hashBuildUnique ( jht9, 400000, hash9, &(payl9));
        }
        loopVar += step;
    }

}

__global__ void krnl_lineitem8(
    int* iatt24_lorderke, int* iatt25_lpartkey, int* iatt26_lsuppkey, float* iatt29_lextende, float* iatt30_ldiscoun, unique_ht<jpayl9>* jht9, multi_ht* jht12, jpayl12* jht12_payload) {
    int att24_lorderke;
    int att25_lpartkey;
    int att26_lsuppkey;
    float att29_lextende;
    float att30_ldiscoun;
    int att15_ppartkey;

    int tid_lineitem1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_lineitem1 = loopVar;
        active = (loopVar < 6001215);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att24_lorderke = iatt24_lorderke[tid_lineitem1];
            att25_lpartkey = iatt25_lpartkey[tid_lineitem1];
            att26_lsuppkey = iatt26_lsuppkey[tid_lineitem1];
            att29_lextende = iatt29_lextende[tid_lineitem1];
            att30_ldiscoun = iatt30_ldiscoun[tid_lineitem1];
        }
        // -------- hash join probe (opId: 9) --------
        uint64_t hash9 = 0;
        if(active) {
            hash9 = 0;
            if(active) {
                hash9 = hash ( (hash9 + ((uint64_t)att25_lpartkey)));
            }
        }
        jpayl9* probepayl9;
        int numLookups9 = 0;
        if(active) {
            active = hashProbeUnique ( jht9, 400000, hash9, numLookups9, &(probepayl9));
        }
        int bucketFound9 = 0;
        int probeActive9 = active;
        while((probeActive9 && !(bucketFound9))) {
            jpayl9 jprobepayl9 = *(probepayl9);
            att15_ppartkey = jprobepayl9.att15_ppartkey;
            bucketFound9 = 1;
            bucketFound9 &= ((att15_ppartkey == att25_lpartkey));
            if(!(bucketFound9)) {
                probeActive9 = hashProbeUnique ( jht9, 400000, hash9, numLookups9, &(probepayl9));
            }
        }
        active = bucketFound9;
        // -------- hash join build (opId: 12) --------
        if(active) {
            uint64_t hash12 = 0;
            if(active) {
                hash12 = 0;
                if(active) {
                    hash12 = hash ( (hash12 + ((uint64_t)att24_lorderke)));
                }
            }
            hashCountMulti ( jht12, 120024, hash12);
        }
        loopVar += step;
    }

}

__global__ void krnl_lineitem8_ins(
    int* iatt24_lorderke, int* iatt25_lpartkey, int* iatt26_lsuppkey, float* iatt29_lextende, float* iatt30_ldiscoun, unique_ht<jpayl9>* jht9, multi_ht* jht12, jpayl12* jht12_payload, int* offs12) {
    int att24_lorderke;
    int att25_lpartkey;
    int att26_lsuppkey;
    float att29_lextende;
    float att30_ldiscoun;
    int att15_ppartkey;

    int tid_lineitem1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_lineitem1 = loopVar;
        active = (loopVar < 6001215);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att24_lorderke = iatt24_lorderke[tid_lineitem1];
            att25_lpartkey = iatt25_lpartkey[tid_lineitem1];
            att26_lsuppkey = iatt26_lsuppkey[tid_lineitem1];
            att29_lextende = iatt29_lextende[tid_lineitem1];
            att30_ldiscoun = iatt30_ldiscoun[tid_lineitem1];
        }
        // -------- hash join probe (opId: 9) --------
        uint64_t hash9 = 0;
        if(active) {
            hash9 = 0;
            if(active) {
                hash9 = hash ( (hash9 + ((uint64_t)att25_lpartkey)));
            }
        }
        jpayl9* probepayl9;
        int numLookups9 = 0;
        if(active) {
            active = hashProbeUnique ( jht9, 400000, hash9, numLookups9, &(probepayl9));
        }
        int bucketFound9 = 0;
        int probeActive9 = active;
        while((probeActive9 && !(bucketFound9))) {
            jpayl9 jprobepayl9 = *(probepayl9);
            att15_ppartkey = jprobepayl9.att15_ppartkey;
            bucketFound9 = 1;
            bucketFound9 &= ((att15_ppartkey == att25_lpartkey));
            if(!(bucketFound9)) {
                probeActive9 = hashProbeUnique ( jht9, 400000, hash9, numLookups9, &(probepayl9));
            }
        }
        active = bucketFound9;
        // -------- hash join build (opId: 12) --------
        if(active) {
            uint64_t hash12 = 0;
            if(active) {
                hash12 = 0;
                if(active) {
                    hash12 = hash ( (hash12 + ((uint64_t)att24_lorderke)));
                }
            }
            jpayl12 payl;
            payl.att24_lorderke = att24_lorderke;
            payl.att26_lsuppkey = att26_lsuppkey;
            payl.att29_lextende = att29_lextende;
            payl.att30_ldiscoun = att30_ldiscoun;
            hashInsertMulti ( jht12, jht12_payload, offs12, 120024, hash12, &(payl));
        }
        loopVar += step;
    }

}

__global__ void krnl_orders10(
    int* iatt40_oorderke, int* iatt41_ocustkey, unsigned* iatt44_oorderda, multi_ht* jht12, jpayl12* jht12_payload, multi_ht* jht14, jpayl14* jht14_payload) {
    int att40_oorderke;
    int att41_ocustkey;
    unsigned att44_oorderda;
    unsigned warplane = (threadIdx.x % 32);
    int att24_lorderke;
    int att26_lsuppkey;
    float att29_lextende;
    float att30_ldiscoun;

    int tid_orders1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_orders1 = loopVar;
        active = (loopVar < 1500000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att40_oorderke = iatt40_oorderke[tid_orders1];
            att41_ocustkey = iatt41_ocustkey[tid_orders1];
            att44_oorderda = iatt44_oorderda[tid_orders1];
        }
        // -------- selection (opId: 11) --------
        if(active) {
            active = ((att44_oorderda >= 19950101) && (att44_oorderda <= 19961231));
        }
        // -------- hash join probe (opId: 12) --------
        // -------- multiprobe multi broadcast (opId: 12) --------
        int matchEnd12 = 0;
        int matchEndBuf12 = 0;
        int matchOffset12 = 0;
        int matchOffsetBuf12 = 0;
        int probeActive12 = active;
        int att40_oorderke_bcbuf12;
        int att41_ocustkey_bcbuf12;
        unsigned att44_oorderda_bcbuf12;
        uint64_t hash12 = 0;
        if(probeActive12) {
            hash12 = 0;
            if(active) {
                hash12 = hash ( (hash12 + ((uint64_t)att40_oorderke)));
            }
            probeActive12 = hashProbeMulti ( jht12, 120024, hash12, matchOffsetBuf12, matchEndBuf12);
        }
        unsigned activeProbes12 = __ballot_sync(ALL_LANES,probeActive12);
        int num12 = 0;
        num12 = (matchEndBuf12 - matchOffsetBuf12);
        unsigned wideProbes12 = __ballot_sync(ALL_LANES,(num12 >= 32));
        att40_oorderke_bcbuf12 = att40_oorderke;
        att41_ocustkey_bcbuf12 = att41_ocustkey;
        att44_oorderda_bcbuf12 = att44_oorderda;
        while((activeProbes12 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes12)) {
                if((wideProbes12 > 0)) {
                    tupleLane = (__ffs(wideProbes12) - 1);
                    wideProbes12 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes12) - 1);
                }
                num = __shfl_sync(ALL_LANES,num12,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset12 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes12 -= (1 << tupleLane);
            }
            matchOffset12 += __shfl_sync(ALL_LANES,matchOffsetBuf12,broadcastLane);
            matchEnd12 = __shfl_sync(ALL_LANES,matchEndBuf12,broadcastLane);
            att40_oorderke = __shfl_sync(ALL_LANES,att40_oorderke_bcbuf12,broadcastLane);
            att41_ocustkey = __shfl_sync(ALL_LANES,att41_ocustkey_bcbuf12,broadcastLane);
            att44_oorderda = __shfl_sync(ALL_LANES,att44_oorderda_bcbuf12,broadcastLane);
            probeActive12 = (matchOffset12 < matchEnd12);
            while(__any_sync(ALL_LANES,probeActive12)) {
                active = probeActive12;
                active = 0;
                jpayl12 payl;
                if(probeActive12) {
                    payl = jht12_payload[matchOffset12];
                    att24_lorderke = payl.att24_lorderke;
                    att26_lsuppkey = payl.att26_lsuppkey;
                    att29_lextende = payl.att29_lextende;
                    att30_ldiscoun = payl.att30_ldiscoun;
                    active = 1;
                    active &= ((att24_lorderke == att40_oorderke));
                    matchOffset12 += 32;
                    probeActive12 &= ((matchOffset12 < matchEnd12));
                }
                // -------- hash join build (opId: 14) --------
                if(active) {
                    uint64_t hash14 = 0;
                    if(active) {
                        hash14 = 0;
                        if(active) {
                            hash14 = hash ( (hash14 + ((uint64_t)att41_ocustkey)));
                        }
                    }
                    hashCountMulti ( jht14, 300000, hash14);
                }
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_orders10_ins(
    int* iatt40_oorderke, int* iatt41_ocustkey, unsigned* iatt44_oorderda, multi_ht* jht12, jpayl12* jht12_payload, multi_ht* jht14, jpayl14* jht14_payload, int* offs14) {
    int att40_oorderke;
    int att41_ocustkey;
    unsigned att44_oorderda;
    unsigned warplane = (threadIdx.x % 32);
    int att24_lorderke;
    int att26_lsuppkey;
    float att29_lextende;
    float att30_ldiscoun;

    int tid_orders1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_orders1 = loopVar;
        active = (loopVar < 1500000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att40_oorderke = iatt40_oorderke[tid_orders1];
            att41_ocustkey = iatt41_ocustkey[tid_orders1];
            att44_oorderda = iatt44_oorderda[tid_orders1];
        }
        // -------- selection (opId: 11) --------
        if(active) {
            active = ((att44_oorderda >= 19950101) && (att44_oorderda <= 19961231));
        }
        // -------- hash join probe (opId: 12) --------
        // -------- multiprobe multi broadcast (opId: 12) --------
        int matchEnd12 = 0;
        int matchEndBuf12 = 0;
        int matchOffset12 = 0;
        int matchOffsetBuf12 = 0;
        int probeActive12 = active;
        int att40_oorderke_bcbuf12;
        int att41_ocustkey_bcbuf12;
        unsigned att44_oorderda_bcbuf12;
        uint64_t hash12 = 0;
        if(probeActive12) {
            hash12 = 0;
            if(active) {
                hash12 = hash ( (hash12 + ((uint64_t)att40_oorderke)));
            }
            probeActive12 = hashProbeMulti ( jht12, 120024, hash12, matchOffsetBuf12, matchEndBuf12);
        }
        unsigned activeProbes12 = __ballot_sync(ALL_LANES,probeActive12);
        int num12 = 0;
        num12 = (matchEndBuf12 - matchOffsetBuf12);
        unsigned wideProbes12 = __ballot_sync(ALL_LANES,(num12 >= 32));
        att40_oorderke_bcbuf12 = att40_oorderke;
        att41_ocustkey_bcbuf12 = att41_ocustkey;
        att44_oorderda_bcbuf12 = att44_oorderda;
        while((activeProbes12 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes12)) {
                if((wideProbes12 > 0)) {
                    tupleLane = (__ffs(wideProbes12) - 1);
                    wideProbes12 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes12) - 1);
                }
                num = __shfl_sync(ALL_LANES,num12,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset12 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes12 -= (1 << tupleLane);
            }
            matchOffset12 += __shfl_sync(ALL_LANES,matchOffsetBuf12,broadcastLane);
            matchEnd12 = __shfl_sync(ALL_LANES,matchEndBuf12,broadcastLane);
            att40_oorderke = __shfl_sync(ALL_LANES,att40_oorderke_bcbuf12,broadcastLane);
            att41_ocustkey = __shfl_sync(ALL_LANES,att41_ocustkey_bcbuf12,broadcastLane);
            att44_oorderda = __shfl_sync(ALL_LANES,att44_oorderda_bcbuf12,broadcastLane);
            probeActive12 = (matchOffset12 < matchEnd12);
            while(__any_sync(ALL_LANES,probeActive12)) {
                active = probeActive12;
                active = 0;
                jpayl12 payl;
                if(probeActive12) {
                    payl = jht12_payload[matchOffset12];
                    att24_lorderke = payl.att24_lorderke;
                    att26_lsuppkey = payl.att26_lsuppkey;
                    att29_lextende = payl.att29_lextende;
                    att30_ldiscoun = payl.att30_ldiscoun;
                    active = 1;
                    active &= ((att24_lorderke == att40_oorderke));
                    matchOffset12 += 32;
                    probeActive12 &= ((matchOffset12 < matchEnd12));
                }
                // -------- hash join build (opId: 14) --------
                if(active) {
                    uint64_t hash14 = 0;
                    if(active) {
                        hash14 = 0;
                        if(active) {
                            hash14 = hash ( (hash14 + ((uint64_t)att41_ocustkey)));
                        }
                    }
                    jpayl14 payl;
                    payl.att26_lsuppkey = att26_lsuppkey;
                    payl.att29_lextende = att29_lextende;
                    payl.att30_ldiscoun = att30_ldiscoun;
                    payl.att41_ocustkey = att41_ocustkey;
                    payl.att44_oorderda = att44_oorderda;
                    hashInsertMulti ( jht14, jht14_payload, offs14, 300000, hash14, &(payl));
                }
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_customer13(
    int* iatt49_ccustkey, int* iatt52_cnationk, multi_ht* jht14, jpayl14* jht14_payload, unique_ht<jpayl15>* jht15, multi_ht* jht17, jpayl17* jht17_payload) {
    int att49_ccustkey;
    int att52_cnationk;
    unsigned warplane = (threadIdx.x % 32);
    int att26_lsuppkey;
    float att29_lextende;
    float att30_ldiscoun;
    int att41_ocustkey;
    unsigned att44_oorderda;
    int att11_nnationk;

    int tid_customer1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_customer1 = loopVar;
        active = (loopVar < 150000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att49_ccustkey = iatt49_ccustkey[tid_customer1];
            att52_cnationk = iatt52_cnationk[tid_customer1];
        }
        // -------- hash join probe (opId: 14) --------
        // -------- multiprobe multi broadcast (opId: 14) --------
        int matchEnd14 = 0;
        int matchEndBuf14 = 0;
        int matchOffset14 = 0;
        int matchOffsetBuf14 = 0;
        int probeActive14 = active;
        int att49_ccustkey_bcbuf14;
        int att52_cnationk_bcbuf14;
        uint64_t hash14 = 0;
        if(probeActive14) {
            hash14 = 0;
            if(active) {
                hash14 = hash ( (hash14 + ((uint64_t)att49_ccustkey)));
            }
            probeActive14 = hashProbeMulti ( jht14, 300000, hash14, matchOffsetBuf14, matchEndBuf14);
        }
        unsigned activeProbes14 = __ballot_sync(ALL_LANES,probeActive14);
        int num14 = 0;
        num14 = (matchEndBuf14 - matchOffsetBuf14);
        unsigned wideProbes14 = __ballot_sync(ALL_LANES,(num14 >= 32));
        att49_ccustkey_bcbuf14 = att49_ccustkey;
        att52_cnationk_bcbuf14 = att52_cnationk;
        while((activeProbes14 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes14)) {
                if((wideProbes14 > 0)) {
                    tupleLane = (__ffs(wideProbes14) - 1);
                    wideProbes14 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes14) - 1);
                }
                num = __shfl_sync(ALL_LANES,num14,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset14 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes14 -= (1 << tupleLane);
            }
            matchOffset14 += __shfl_sync(ALL_LANES,matchOffsetBuf14,broadcastLane);
            matchEnd14 = __shfl_sync(ALL_LANES,matchEndBuf14,broadcastLane);
            att49_ccustkey = __shfl_sync(ALL_LANES,att49_ccustkey_bcbuf14,broadcastLane);
            att52_cnationk = __shfl_sync(ALL_LANES,att52_cnationk_bcbuf14,broadcastLane);
            probeActive14 = (matchOffset14 < matchEnd14);
            while(__any_sync(ALL_LANES,probeActive14)) {
                active = probeActive14;
                active = 0;
                jpayl14 payl;
                if(probeActive14) {
                    payl = jht14_payload[matchOffset14];
                    att26_lsuppkey = payl.att26_lsuppkey;
                    att29_lextende = payl.att29_lextende;
                    att30_ldiscoun = payl.att30_ldiscoun;
                    att41_ocustkey = payl.att41_ocustkey;
                    att44_oorderda = payl.att44_oorderda;
                    active = 1;
                    active &= ((att41_ocustkey == att49_ccustkey));
                    matchOffset14 += 32;
                    probeActive14 &= ((matchOffset14 < matchEnd14));
                }
                // -------- hash join probe (opId: 15) --------
                uint64_t hash15 = 0;
                if(active) {
                    hash15 = 0;
                    if(active) {
                        hash15 = hash ( (hash15 + ((uint64_t)att52_cnationk)));
                    }
                }
                jpayl15* probepayl15;
                int numLookups15 = 0;
                if(active) {
                    active = hashProbeUnique ( jht15, 50, hash15, numLookups15, &(probepayl15));
                }
                int bucketFound15 = 0;
                int probeActive15 = active;
                while((probeActive15 && !(bucketFound15))) {
                    jpayl15 jprobepayl15 = *(probepayl15);
                    att11_nnationk = jprobepayl15.att11_nnationk;
                    bucketFound15 = 1;
                    bucketFound15 &= ((att11_nnationk == att52_cnationk));
                    if(!(bucketFound15)) {
                        probeActive15 = hashProbeUnique ( jht15, 50, hash15, numLookups15, &(probepayl15));
                    }
                }
                active = bucketFound15;
                // -------- hash join build (opId: 17) --------
                if(active) {
                    uint64_t hash17 = 0;
                    if(active) {
                        hash17 = 0;
                        if(active) {
                            hash17 = hash ( (hash17 + ((uint64_t)att26_lsuppkey)));
                        }
                    }
                    hashCountMulti ( jht17, 75000, hash17);
                }
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_customer13_ins(
    int* iatt49_ccustkey, int* iatt52_cnationk, multi_ht* jht14, jpayl14* jht14_payload, unique_ht<jpayl15>* jht15, multi_ht* jht17, jpayl17* jht17_payload, int* offs17) {
    int att49_ccustkey;
    int att52_cnationk;
    unsigned warplane = (threadIdx.x % 32);
    int att26_lsuppkey;
    float att29_lextende;
    float att30_ldiscoun;
    int att41_ocustkey;
    unsigned att44_oorderda;
    int att11_nnationk;

    int tid_customer1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_customer1 = loopVar;
        active = (loopVar < 150000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att49_ccustkey = iatt49_ccustkey[tid_customer1];
            att52_cnationk = iatt52_cnationk[tid_customer1];
        }
        // -------- hash join probe (opId: 14) --------
        // -------- multiprobe multi broadcast (opId: 14) --------
        int matchEnd14 = 0;
        int matchEndBuf14 = 0;
        int matchOffset14 = 0;
        int matchOffsetBuf14 = 0;
        int probeActive14 = active;
        int att49_ccustkey_bcbuf14;
        int att52_cnationk_bcbuf14;
        uint64_t hash14 = 0;
        if(probeActive14) {
            hash14 = 0;
            if(active) {
                hash14 = hash ( (hash14 + ((uint64_t)att49_ccustkey)));
            }
            probeActive14 = hashProbeMulti ( jht14, 300000, hash14, matchOffsetBuf14, matchEndBuf14);
        }
        unsigned activeProbes14 = __ballot_sync(ALL_LANES,probeActive14);
        int num14 = 0;
        num14 = (matchEndBuf14 - matchOffsetBuf14);
        unsigned wideProbes14 = __ballot_sync(ALL_LANES,(num14 >= 32));
        att49_ccustkey_bcbuf14 = att49_ccustkey;
        att52_cnationk_bcbuf14 = att52_cnationk;
        while((activeProbes14 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes14)) {
                if((wideProbes14 > 0)) {
                    tupleLane = (__ffs(wideProbes14) - 1);
                    wideProbes14 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes14) - 1);
                }
                num = __shfl_sync(ALL_LANES,num14,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset14 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes14 -= (1 << tupleLane);
            }
            matchOffset14 += __shfl_sync(ALL_LANES,matchOffsetBuf14,broadcastLane);
            matchEnd14 = __shfl_sync(ALL_LANES,matchEndBuf14,broadcastLane);
            att49_ccustkey = __shfl_sync(ALL_LANES,att49_ccustkey_bcbuf14,broadcastLane);
            att52_cnationk = __shfl_sync(ALL_LANES,att52_cnationk_bcbuf14,broadcastLane);
            probeActive14 = (matchOffset14 < matchEnd14);
            while(__any_sync(ALL_LANES,probeActive14)) {
                active = probeActive14;
                active = 0;
                jpayl14 payl;
                if(probeActive14) {
                    payl = jht14_payload[matchOffset14];
                    att26_lsuppkey = payl.att26_lsuppkey;
                    att29_lextende = payl.att29_lextende;
                    att30_ldiscoun = payl.att30_ldiscoun;
                    att41_ocustkey = payl.att41_ocustkey;
                    att44_oorderda = payl.att44_oorderda;
                    active = 1;
                    active &= ((att41_ocustkey == att49_ccustkey));
                    matchOffset14 += 32;
                    probeActive14 &= ((matchOffset14 < matchEnd14));
                }
                // -------- hash join probe (opId: 15) --------
                uint64_t hash15 = 0;
                if(active) {
                    hash15 = 0;
                    if(active) {
                        hash15 = hash ( (hash15 + ((uint64_t)att52_cnationk)));
                    }
                }
                jpayl15* probepayl15;
                int numLookups15 = 0;
                if(active) {
                    active = hashProbeUnique ( jht15, 50, hash15, numLookups15, &(probepayl15));
                }
                int bucketFound15 = 0;
                int probeActive15 = active;
                while((probeActive15 && !(bucketFound15))) {
                    jpayl15 jprobepayl15 = *(probepayl15);
                    att11_nnationk = jprobepayl15.att11_nnationk;
                    bucketFound15 = 1;
                    bucketFound15 &= ((att11_nnationk == att52_cnationk));
                    if(!(bucketFound15)) {
                        probeActive15 = hashProbeUnique ( jht15, 50, hash15, numLookups15, &(probepayl15));
                    }
                }
                active = bucketFound15;
                // -------- hash join build (opId: 17) --------
                if(active) {
                    uint64_t hash17 = 0;
                    if(active) {
                        hash17 = 0;
                        if(active) {
                            hash17 = hash ( (hash17 + ((uint64_t)att26_lsuppkey)));
                        }
                    }
                    jpayl17 payl;
                    payl.att26_lsuppkey = att26_lsuppkey;
                    payl.att29_lextende = att29_lextende;
                    payl.att30_ldiscoun = att30_ldiscoun;
                    payl.att44_oorderda = att44_oorderda;
                    hashInsertMulti ( jht17, jht17_payload, offs17, 75000, hash17, &(payl));
                }
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_supplier16(
    int* iatt57_ssuppkey, int* iatt60_snationk, multi_ht* jht17, jpayl17* jht17_payload, multi_ht* jht18, jpayl18* jht18_payload, agg_ht<apayl22>* aht22, float* agg1, float* agg2, int* agg3) {
    int att57_ssuppkey;
    int att60_snationk;
    unsigned warplane = (threadIdx.x % 32);
    int att26_lsuppkey;
    float att29_lextende;
    float att30_ldiscoun;
    unsigned att44_oorderda;
    int att4_nnationk;
    str_t att5_nname;
    unsigned att64_oyear;
    float att65_volume;
    float att66_casevolu;
    str_t c3 = stringConstant ( "BRAZIL", 6);

    int tid_supplier1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_supplier1 = loopVar;
        active = (loopVar < 10000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att57_ssuppkey = iatt57_ssuppkey[tid_supplier1];
            att60_snationk = iatt60_snationk[tid_supplier1];
        }
        // -------- hash join probe (opId: 17) --------
        // -------- multiprobe multi broadcast (opId: 17) --------
        int matchEnd17 = 0;
        int matchEndBuf17 = 0;
        int matchOffset17 = 0;
        int matchOffsetBuf17 = 0;
        int probeActive17 = active;
        int att57_ssuppkey_bcbuf17;
        int att60_snationk_bcbuf17;
        uint64_t hash17 = 0;
        if(probeActive17) {
            hash17 = 0;
            if(active) {
                hash17 = hash ( (hash17 + ((uint64_t)att57_ssuppkey)));
            }
            probeActive17 = hashProbeMulti ( jht17, 75000, hash17, matchOffsetBuf17, matchEndBuf17);
        }
        unsigned activeProbes17 = __ballot_sync(ALL_LANES,probeActive17);
        int num17 = 0;
        num17 = (matchEndBuf17 - matchOffsetBuf17);
        unsigned wideProbes17 = __ballot_sync(ALL_LANES,(num17 >= 32));
        att57_ssuppkey_bcbuf17 = att57_ssuppkey;
        att60_snationk_bcbuf17 = att60_snationk;
        while((activeProbes17 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes17)) {
                if((wideProbes17 > 0)) {
                    tupleLane = (__ffs(wideProbes17) - 1);
                    wideProbes17 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes17) - 1);
                }
                num = __shfl_sync(ALL_LANES,num17,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset17 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes17 -= (1 << tupleLane);
            }
            matchOffset17 += __shfl_sync(ALL_LANES,matchOffsetBuf17,broadcastLane);
            matchEnd17 = __shfl_sync(ALL_LANES,matchEndBuf17,broadcastLane);
            att57_ssuppkey = __shfl_sync(ALL_LANES,att57_ssuppkey_bcbuf17,broadcastLane);
            att60_snationk = __shfl_sync(ALL_LANES,att60_snationk_bcbuf17,broadcastLane);
            probeActive17 = (matchOffset17 < matchEnd17);
            while(__any_sync(ALL_LANES,probeActive17)) {
                active = probeActive17;
                active = 0;
                jpayl17 payl;
                if(probeActive17) {
                    payl = jht17_payload[matchOffset17];
                    att26_lsuppkey = payl.att26_lsuppkey;
                    att29_lextende = payl.att29_lextende;
                    att30_ldiscoun = payl.att30_ldiscoun;
                    att44_oorderda = payl.att44_oorderda;
                    active = 1;
                    active &= ((att26_lsuppkey == att57_ssuppkey));
                    matchOffset17 += 32;
                    probeActive17 &= ((matchOffset17 < matchEnd17));
                }
                // -------- hash join probe (opId: 18) --------
                // -------- multiprobe multi broadcast (opId: 18) --------
                int matchEnd18 = 0;
                int matchEndBuf18 = 0;
                int matchOffset18 = 0;
                int matchOffsetBuf18 = 0;
                int probeActive18 = active;
                float att29_lextende_bcbuf18;
                float att30_ldiscoun_bcbuf18;
                unsigned att44_oorderda_bcbuf18;
                int att60_snationk_bcbuf18;
                uint64_t hash18 = 0;
                if(probeActive18) {
                    hash18 = 0;
                    if(active) {
                        hash18 = hash ( (hash18 + ((uint64_t)att60_snationk)));
                    }
                    probeActive18 = hashProbeMulti ( jht18, 50, hash18, matchOffsetBuf18, matchEndBuf18);
                }
                unsigned activeProbes18 = __ballot_sync(ALL_LANES,probeActive18);
                int num18 = 0;
                num18 = (matchEndBuf18 - matchOffsetBuf18);
                unsigned wideProbes18 = __ballot_sync(ALL_LANES,(num18 >= 32));
                att29_lextende_bcbuf18 = att29_lextende;
                att30_ldiscoun_bcbuf18 = att30_ldiscoun;
                att44_oorderda_bcbuf18 = att44_oorderda;
                att60_snationk_bcbuf18 = att60_snationk;
                while((activeProbes18 > 0)) {
                    unsigned tupleLane;
                    unsigned broadcastLane;
                    int numFilled = 0;
                    int num = 0;
                    while(((numFilled < 32) && activeProbes18)) {
                        if((wideProbes18 > 0)) {
                            tupleLane = (__ffs(wideProbes18) - 1);
                            wideProbes18 -= (1 << tupleLane);
                        }
                        else {
                            tupleLane = (__ffs(activeProbes18) - 1);
                        }
                        num = __shfl_sync(ALL_LANES,num18,tupleLane);
                        if((numFilled && ((numFilled + num) > 32))) {
                            break;
                        }
                        if((warplane >= numFilled)) {
                            broadcastLane = tupleLane;
                            matchOffset18 = (warplane - numFilled);
                        }
                        numFilled += num;
                        activeProbes18 -= (1 << tupleLane);
                    }
                    matchOffset18 += __shfl_sync(ALL_LANES,matchOffsetBuf18,broadcastLane);
                    matchEnd18 = __shfl_sync(ALL_LANES,matchEndBuf18,broadcastLane);
                    att29_lextende = __shfl_sync(ALL_LANES,att29_lextende_bcbuf18,broadcastLane);
                    att30_ldiscoun = __shfl_sync(ALL_LANES,att30_ldiscoun_bcbuf18,broadcastLane);
                    att44_oorderda = __shfl_sync(ALL_LANES,att44_oorderda_bcbuf18,broadcastLane);
                    att60_snationk = __shfl_sync(ALL_LANES,att60_snationk_bcbuf18,broadcastLane);
                    probeActive18 = (matchOffset18 < matchEnd18);
                    while(__any_sync(ALL_LANES,probeActive18)) {
                        active = probeActive18;
                        active = 0;
                        jpayl18 payl;
                        if(probeActive18) {
                            payl = jht18_payload[matchOffset18];
                            att4_nnationk = payl.att4_nnationk;
                            att5_nname = payl.att5_nname;
                            active = 1;
                            active &= ((att4_nnationk == att60_snationk));
                            matchOffset18 += 32;
                            probeActive18 &= ((matchOffset18 < matchEnd18));
                        }
                        // -------- map (opId: 19) --------
                        if(active) {
                            att64_oyear = (att44_oorderda / 10000);
                        }
                        // -------- map (opId: 20) --------
                        if(active) {
                            att65_volume = (att29_lextende * ((float)1.0f - att30_ldiscoun));
                        }
                        // -------- map (opId: 21) --------
                        if(active) {
                            float casevar1138;
                            if(stringEquals ( att5_nname, c3)) {
                                casevar1138 = att65_volume;
                            }
                            else {
                                casevar1138 = (float)0;
                            }
                            att66_casevolu = casevar1138;
                        }
                        // -------- aggregation (opId: 22) --------
                        int bucket = 0;
                        if(active) {
                            uint64_t hash22 = 0;
                            hash22 = 0;
                            if(active) {
                                hash22 = hash ( (hash22 + ((uint64_t)att64_oyear)));
                            }
                            apayl22 payl;
                            payl.att64_oyear = att64_oyear;
                            int bucketFound = 0;
                            int numLookups = 0;
                            while(!(bucketFound)) {
                                bucket = hashAggregateGetBucket ( aht22, 75000, hash22, numLookups, &(payl));
                                apayl22 probepayl = aht22[bucket].payload;
                                bucketFound = 1;
                                bucketFound &= ((payl.att64_oyear == probepayl.att64_oyear));
                            }
                        }
                        if(active) {
                            atomicAdd(&(agg1[bucket]), ((float)att66_casevolu));
                            atomicAdd(&(agg2[bucket]), ((float)att65_volume));
                            atomicAdd(&(agg3[bucket]), ((int)1));
                        }
                    }
                }
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation22(
    agg_ht<apayl22>* aht22, float* agg1, float* agg2, int* agg3, int* nout_result, unsigned* oatt64_oyear, float* oatt67_mktshare, int* oatt3_salesnum) {
    unsigned att64_oyear;
    float att1_sumvolum;
    float att2_sumvolum;
    int att3_salesnum;
    float att67_mktshare;
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_aggregation22 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation22 = loopVar;
        active = (loopVar < 75000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 22) --------
        if(active) {
            active &= ((aht22[tid_aggregation22].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl22 payl = aht22[tid_aggregation22].payload;
            att64_oyear = payl.att64_oyear;
        }
        if(active) {
            att1_sumvolum = agg1[tid_aggregation22];
            att2_sumvolum = agg2[tid_aggregation22];
            att3_salesnum = agg3[tid_aggregation22];
        }
        // -------- map (opId: 23) --------
        if(active) {
            att67_mktshare = (att1_sumvolum / att2_sumvolum);
        }
        // -------- projection (no code) (opId: 24) --------
        // -------- materialize (opId: 25) --------
        int wp;
        int writeMask;
        int numProj;
        writeMask = __ballot_sync(ALL_LANES,active);
        numProj = __popc(writeMask);
        if((warplane == 0)) {
            wp = atomicAdd(nout_result, numProj);
        }
        wp = __shfl_sync(ALL_LANES,wp,0);
        wp = (wp + __popc((writeMask & prefixlanes)));
        if(active) {
            oatt64_oyear[wp] = att64_oyear;
            oatt67_mktshare[wp] = att67_mktshare;
            oatt3_salesnum[wp] = att3_salesnum;
        }
        loopVar += step;
    }

}

int main() {
    int* iatt4_nnationk;
    iatt4_nnationk = ( int*) map_memory_file ( "mmdb/nation_n_nationkey" );
    size_t* iatt5_nname_offset;
    iatt5_nname_offset = ( size_t*) map_memory_file ( "mmdb/nation_n_name_offset" );
    char* iatt5_nname_char;
    iatt5_nname_char = ( char*) map_memory_file ( "mmdb/nation_n_name_char" );
    int* iatt8_rregionk;
    iatt8_rregionk = ( int*) map_memory_file ( "mmdb/region_r_regionkey" );
    size_t* iatt9_rname_offset;
    iatt9_rname_offset = ( size_t*) map_memory_file ( "mmdb/region_r_name_offset" );
    char* iatt9_rname_char;
    iatt9_rname_char = ( char*) map_memory_file ( "mmdb/region_r_name_char" );
    int* iatt11_nnationk;
    iatt11_nnationk = ( int*) map_memory_file ( "mmdb/nation_n_nationkey" );
    int* iatt13_nregionk;
    iatt13_nregionk = ( int*) map_memory_file ( "mmdb/nation_n_regionkey" );
    int* iatt15_ppartkey;
    iatt15_ppartkey = ( int*) map_memory_file ( "mmdb/part_p_partkey" );
    size_t* iatt19_ptype_offset;
    iatt19_ptype_offset = ( size_t*) map_memory_file ( "mmdb/part_p_type_offset" );
    char* iatt19_ptype_char;
    iatt19_ptype_char = ( char*) map_memory_file ( "mmdb/part_p_type_char" );
    int* iatt24_lorderke;
    iatt24_lorderke = ( int*) map_memory_file ( "mmdb/lineitem_l_orderkey" );
    int* iatt25_lpartkey;
    iatt25_lpartkey = ( int*) map_memory_file ( "mmdb/lineitem_l_partkey" );
    int* iatt26_lsuppkey;
    iatt26_lsuppkey = ( int*) map_memory_file ( "mmdb/lineitem_l_suppkey" );
    float* iatt29_lextende;
    iatt29_lextende = ( float*) map_memory_file ( "mmdb/lineitem_l_extendedprice" );
    float* iatt30_ldiscoun;
    iatt30_ldiscoun = ( float*) map_memory_file ( "mmdb/lineitem_l_discount" );
    int* iatt40_oorderke;
    iatt40_oorderke = ( int*) map_memory_file ( "mmdb/orders_o_orderkey" );
    int* iatt41_ocustkey;
    iatt41_ocustkey = ( int*) map_memory_file ( "mmdb/orders_o_custkey" );
    unsigned* iatt44_oorderda;
    iatt44_oorderda = ( unsigned*) map_memory_file ( "mmdb/orders_o_orderdate" );
    int* iatt49_ccustkey;
    iatt49_ccustkey = ( int*) map_memory_file ( "mmdb/customer_c_custkey" );
    int* iatt52_cnationk;
    iatt52_cnationk = ( int*) map_memory_file ( "mmdb/customer_c_nationkey" );
    int* iatt57_ssuppkey;
    iatt57_ssuppkey = ( int*) map_memory_file ( "mmdb/supplier_s_suppkey" );
    int* iatt60_snationk;
    iatt60_snationk = ( int*) map_memory_file ( "mmdb/supplier_s_nationkey" );

    int nout_result;
    std::vector < unsigned > oatt64_oyear(37500);
    std::vector < float > oatt67_mktshare(37500);
    std::vector < int > oatt3_salesnum(37500);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt4_nnationk;
    hipMalloc((void**) &d_iatt4_nnationk, 25* sizeof(int) );
    size_t* d_iatt5_nname_offset;
    hipMalloc((void**) &d_iatt5_nname_offset, (25 + 1)* sizeof(size_t) );
    char* d_iatt5_nname_char;
    hipMalloc((void**) &d_iatt5_nname_char, 186* sizeof(char) );
    int* d_iatt8_rregionk;
    hipMalloc((void**) &d_iatt8_rregionk, 5* sizeof(int) );
    size_t* d_iatt9_rname_offset;
    hipMalloc((void**) &d_iatt9_rname_offset, (5 + 1)* sizeof(size_t) );
    char* d_iatt9_rname_char;
    hipMalloc((void**) &d_iatt9_rname_char, 43* sizeof(char) );
    int* d_iatt11_nnationk;
    d_iatt11_nnationk = d_iatt4_nnationk;
    int* d_iatt13_nregionk;
    hipMalloc((void**) &d_iatt13_nregionk, 25* sizeof(int) );
    int* d_iatt15_ppartkey;
    hipMalloc((void**) &d_iatt15_ppartkey, 200000* sizeof(int) );
    size_t* d_iatt19_ptype_offset;
    hipMalloc((void**) &d_iatt19_ptype_offset, (200000 + 1)* sizeof(size_t) );
    char* d_iatt19_ptype_char;
    hipMalloc((void**) &d_iatt19_ptype_char, 4119955* sizeof(char) );
    int* d_iatt24_lorderke;
    hipMalloc((void**) &d_iatt24_lorderke, 6001215* sizeof(int) );
    int* d_iatt25_lpartkey;
    hipMalloc((void**) &d_iatt25_lpartkey, 6001215* sizeof(int) );
    int* d_iatt26_lsuppkey;
    hipMalloc((void**) &d_iatt26_lsuppkey, 6001215* sizeof(int) );
    float* d_iatt29_lextende;
    hipMalloc((void**) &d_iatt29_lextende, 6001215* sizeof(float) );
    float* d_iatt30_ldiscoun;
    hipMalloc((void**) &d_iatt30_ldiscoun, 6001215* sizeof(float) );
    int* d_iatt40_oorderke;
    hipMalloc((void**) &d_iatt40_oorderke, 1500000* sizeof(int) );
    int* d_iatt41_ocustkey;
    hipMalloc((void**) &d_iatt41_ocustkey, 1500000* sizeof(int) );
    unsigned* d_iatt44_oorderda;
    hipMalloc((void**) &d_iatt44_oorderda, 1500000* sizeof(unsigned) );
    int* d_iatt49_ccustkey;
    hipMalloc((void**) &d_iatt49_ccustkey, 150000* sizeof(int) );
    int* d_iatt52_cnationk;
    hipMalloc((void**) &d_iatt52_cnationk, 150000* sizeof(int) );
    int* d_iatt57_ssuppkey;
    hipMalloc((void**) &d_iatt57_ssuppkey, 10000* sizeof(int) );
    int* d_iatt60_snationk;
    hipMalloc((void**) &d_iatt60_snationk, 10000* sizeof(int) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    unsigned* d_oatt64_oyear;
    hipMalloc((void**) &d_oatt64_oyear, 37500* sizeof(unsigned) );
    float* d_oatt67_mktshare;
    hipMalloc((void**) &d_oatt67_mktshare, 37500* sizeof(float) );
    int* d_oatt3_salesnum;
    hipMalloc((void**) &d_oatt3_salesnum, 37500* sizeof(int) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    multi_ht* d_jht18;
    hipMalloc((void**) &d_jht18, 50* sizeof(multi_ht) );
    jpayl18* d_jht18_payload;
    hipMalloc((void**) &d_jht18_payload, 50* sizeof(jpayl18) );
    {
        int gridsize=920;
        int blocksize=128;
        initMultiHT<<<gridsize, blocksize>>>(d_jht18, 50);
    }
    int* d_offs18;
    hipMalloc((void**) &d_offs18, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs18, 0, 1);
    }
    unique_ht<jpayl5>* d_jht5;
    hipMalloc((void**) &d_jht5, 10* sizeof(unique_ht<jpayl5>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht5, 10);
    }
    unique_ht<jpayl15>* d_jht15;
    hipMalloc((void**) &d_jht15, 50* sizeof(unique_ht<jpayl15>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht15, 50);
    }
    unique_ht<jpayl9>* d_jht9;
    hipMalloc((void**) &d_jht9, 400000* sizeof(unique_ht<jpayl9>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht9, 400000);
    }
    multi_ht* d_jht12;
    hipMalloc((void**) &d_jht12, 120024* sizeof(multi_ht) );
    jpayl12* d_jht12_payload;
    hipMalloc((void**) &d_jht12_payload, 120024* sizeof(jpayl12) );
    {
        int gridsize=920;
        int blocksize=128;
        initMultiHT<<<gridsize, blocksize>>>(d_jht12, 120024);
    }
    int* d_offs12;
    hipMalloc((void**) &d_offs12, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs12, 0, 1);
    }
    multi_ht* d_jht14;
    hipMalloc((void**) &d_jht14, 300000* sizeof(multi_ht) );
    jpayl14* d_jht14_payload;
    hipMalloc((void**) &d_jht14_payload, 300000* sizeof(jpayl14) );
    {
        int gridsize=920;
        int blocksize=128;
        initMultiHT<<<gridsize, blocksize>>>(d_jht14, 300000);
    }
    int* d_offs14;
    hipMalloc((void**) &d_offs14, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs14, 0, 1);
    }
    multi_ht* d_jht17;
    hipMalloc((void**) &d_jht17, 75000* sizeof(multi_ht) );
    jpayl17* d_jht17_payload;
    hipMalloc((void**) &d_jht17_payload, 75000* sizeof(jpayl17) );
    {
        int gridsize=920;
        int blocksize=128;
        initMultiHT<<<gridsize, blocksize>>>(d_jht17, 75000);
    }
    int* d_offs17;
    hipMalloc((void**) &d_offs17, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs17, 0, 1);
    }
    agg_ht<apayl22>* d_aht22;
    hipMalloc((void**) &d_aht22, 75000* sizeof(agg_ht<apayl22>) );
    {
        int gridsize=920;
        int blocksize=128;
        initAggHT<<<gridsize, blocksize>>>(d_aht22, 75000);
    }
    float* d_agg1;
    hipMalloc((void**) &d_agg1, 75000* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg1, 0.0f, 75000);
    }
    float* d_agg2;
    hipMalloc((void**) &d_agg2, 75000* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg2, 0.0f, 75000);
    }
    int* d_agg3;
    hipMalloc((void**) &d_agg3, 75000* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg3, 0, 75000);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt4_nnationk, iatt4_nnationk, 25 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt5_nname_offset, iatt5_nname_offset, (25 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt5_nname_char, iatt5_nname_char, 186 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt8_rregionk, iatt8_rregionk, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt9_rname_offset, iatt9_rname_offset, (5 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt9_rname_char, iatt9_rname_char, 43 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt13_nregionk, iatt13_nregionk, 25 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt15_ppartkey, iatt15_ppartkey, 200000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt19_ptype_offset, iatt19_ptype_offset, (200000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt19_ptype_char, iatt19_ptype_char, 4119955 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt24_lorderke, iatt24_lorderke, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt25_lpartkey, iatt25_lpartkey, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt26_lsuppkey, iatt26_lsuppkey, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt29_lextende, iatt29_lextende, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt30_ldiscoun, iatt30_ldiscoun, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt40_oorderke, iatt40_oorderke, 1500000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt41_ocustkey, iatt41_ocustkey, 1500000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt44_oorderda, iatt44_oorderda, 1500000 * sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt49_ccustkey, iatt49_ccustkey, 150000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt52_cnationk, iatt52_cnationk, 150000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt57_ssuppkey, iatt57_ssuppkey, 10000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt60_snationk, iatt60_snationk, 10000 * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime56 = std::clock();
    std::clock_t start_krnl_nation157 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_nation1<<<gridsize, blocksize>>>(d_iatt4_nnationk, d_iatt5_nname_offset, d_iatt5_nname_char, d_jht18, d_jht18_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation157 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation1")
        }
    }

    std::clock_t start_scanMultiHT58 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht18, 50, d_offs18);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT58 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT")
        }
    }

    std::clock_t start_krnl_nation1_ins59 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_nation1_ins<<<gridsize, blocksize>>>(d_iatt4_nnationk, d_iatt5_nname_offset, d_iatt5_nname_char, d_jht18, d_jht18_payload, d_offs18);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation1_ins59 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation1_ins! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation1_ins")
        }
    }

    std::clock_t start_krnl_region260 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_region2<<<gridsize, blocksize>>>(d_iatt8_rregionk, d_iatt9_rname_offset, d_iatt9_rname_char, d_jht5);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_region260 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_region2! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_region2")
        }
    }

    std::clock_t start_krnl_nation2461 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_nation24<<<gridsize, blocksize>>>(d_iatt11_nnationk, d_iatt13_nregionk, d_jht5, d_jht15);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation2461 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation24! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation24")
        }
    }

    std::clock_t start_krnl_part662 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_part6<<<gridsize, blocksize>>>(d_iatt15_ppartkey, d_iatt19_ptype_offset, d_iatt19_ptype_char, d_jht9);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_part662 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_part6! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_part6")
        }
    }

    std::clock_t start_krnl_lineitem863 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_lineitem8<<<gridsize, blocksize>>>(d_iatt24_lorderke, d_iatt25_lpartkey, d_iatt26_lsuppkey, d_iatt29_lextende, d_iatt30_ldiscoun, d_jht9, d_jht12, d_jht12_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_lineitem863 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem8! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem8")
        }
    }

    std::clock_t start_scanMultiHT64 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht12, 120024, d_offs12);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT64 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT")
        }
    }

    std::clock_t start_krnl_lineitem8_ins65 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_lineitem8_ins<<<gridsize, blocksize>>>(d_iatt24_lorderke, d_iatt25_lpartkey, d_iatt26_lsuppkey, d_iatt29_lextende, d_iatt30_ldiscoun, d_jht9, d_jht12, d_jht12_payload, d_offs12);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_lineitem8_ins65 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem8_ins! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem8_ins")
        }
    }

    std::clock_t start_krnl_orders1066 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_orders10<<<gridsize, blocksize>>>(d_iatt40_oorderke, d_iatt41_ocustkey, d_iatt44_oorderda, d_jht12, d_jht12_payload, d_jht14, d_jht14_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_orders1066 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_orders10! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_orders10")
        }
    }

    std::clock_t start_scanMultiHT67 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht14, 300000, d_offs14);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT67 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT")
        }
    }

    std::clock_t start_krnl_orders10_ins68 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_orders10_ins<<<gridsize, blocksize>>>(d_iatt40_oorderke, d_iatt41_ocustkey, d_iatt44_oorderda, d_jht12, d_jht12_payload, d_jht14, d_jht14_payload, d_offs14);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_orders10_ins68 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_orders10_ins! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_orders10_ins")
        }
    }

    std::clock_t start_krnl_customer1369 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_customer13<<<gridsize, blocksize>>>(d_iatt49_ccustkey, d_iatt52_cnationk, d_jht14, d_jht14_payload, d_jht15, d_jht17, d_jht17_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_customer1369 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_customer13! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_customer13")
        }
    }

    std::clock_t start_scanMultiHT70 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht17, 75000, d_offs17);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT70 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT")
        }
    }

    std::clock_t start_krnl_customer13_ins71 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_customer13_ins<<<gridsize, blocksize>>>(d_iatt49_ccustkey, d_iatt52_cnationk, d_jht14, d_jht14_payload, d_jht15, d_jht17, d_jht17_payload, d_offs17);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_customer13_ins71 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_customer13_ins! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_customer13_ins")
        }
    }

    std::clock_t start_krnl_supplier1672 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_supplier16<<<gridsize, blocksize>>>(d_iatt57_ssuppkey, d_iatt60_snationk, d_jht17, d_jht17_payload, d_jht18, d_jht18_payload, d_aht22, d_agg1, d_agg2, d_agg3);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_supplier1672 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_supplier16! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_supplier16")
        }
    }

    std::clock_t start_krnl_aggregation2273 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation22<<<gridsize, blocksize>>>(d_aht22, d_agg1, d_agg2, d_agg3, d_nout_result, d_oatt64_oyear, d_oatt67_mktshare, d_oatt3_salesnum);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation2273 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation22! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation22")
        }
    }

    std::clock_t stop_totalKernelTime56 = std::clock();
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt64_oyear.data(), d_oatt64_oyear, 37500 * sizeof(unsigned), hipMemcpyDeviceToHost);
    hipMemcpy( oatt67_mktshare.data(), d_oatt67_mktshare, 37500 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( oatt3_salesnum.data(), d_oatt3_salesnum, 37500 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt4_nnationk);
    hipFree( d_iatt5_nname_offset);
    hipFree( d_iatt5_nname_char);
    hipFree( d_jht18);
    hipFree( d_jht18_payload);
    hipFree( d_offs18);
    hipFree( d_iatt8_rregionk);
    hipFree( d_iatt9_rname_offset);
    hipFree( d_iatt9_rname_char);
    hipFree( d_jht5);
    hipFree( d_iatt13_nregionk);
    hipFree( d_jht15);
    hipFree( d_iatt15_ppartkey);
    hipFree( d_iatt19_ptype_offset);
    hipFree( d_iatt19_ptype_char);
    hipFree( d_jht9);
    hipFree( d_iatt24_lorderke);
    hipFree( d_iatt25_lpartkey);
    hipFree( d_iatt26_lsuppkey);
    hipFree( d_iatt29_lextende);
    hipFree( d_iatt30_ldiscoun);
    hipFree( d_jht12);
    hipFree( d_jht12_payload);
    hipFree( d_offs12);
    hipFree( d_iatt40_oorderke);
    hipFree( d_iatt41_ocustkey);
    hipFree( d_iatt44_oorderda);
    hipFree( d_jht14);
    hipFree( d_jht14_payload);
    hipFree( d_offs14);
    hipFree( d_iatt49_ccustkey);
    hipFree( d_iatt52_cnationk);
    hipFree( d_jht17);
    hipFree( d_jht17_payload);
    hipFree( d_offs17);
    hipFree( d_iatt57_ssuppkey);
    hipFree( d_iatt60_snationk);
    hipFree( d_aht22);
    hipFree( d_agg1);
    hipFree( d_agg2);
    hipFree( d_agg3);
    hipFree( d_nout_result);
    hipFree( d_oatt64_oyear);
    hipFree( d_oatt67_mktshare);
    hipFree( d_oatt3_salesnum);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish74 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 37500)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("o_year: ");
        printf("%10i", oatt64_oyear[pv]);
        printf("  ");
        printf("mkt_share: ");
        printf("%15.2f", oatt67_mktshare[pv]);
        printf("  ");
        printf("salesnum: ");
        printf("%8i", oatt3_salesnum[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish74 = std::clock();

    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "krnl_nation1", (stop_krnl_nation157 - start_krnl_nation157) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "scanMultiHT", (stop_scanMultiHT58 - start_scanMultiHT58) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_nation1_ins", (stop_krnl_nation1_ins59 - start_krnl_nation1_ins59) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_region2", (stop_krnl_region260 - start_krnl_region260) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_nation24", (stop_krnl_nation2461 - start_krnl_nation2461) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_part6", (stop_krnl_part662 - start_krnl_part662) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_lineitem8", (stop_krnl_lineitem863 - start_krnl_lineitem863) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "scanMultiHT", (stop_scanMultiHT64 - start_scanMultiHT64) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_lineitem8_ins", (stop_krnl_lineitem8_ins65 - start_krnl_lineitem8_ins65) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_orders10", (stop_krnl_orders1066 - start_krnl_orders1066) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "scanMultiHT", (stop_scanMultiHT67 - start_scanMultiHT67) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_orders10_ins", (stop_krnl_orders10_ins68 - start_krnl_orders10_ins68) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_customer13", (stop_krnl_customer1369 - start_krnl_customer1369) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "scanMultiHT", (stop_scanMultiHT70 - start_scanMultiHT70) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_customer13_ins", (stop_krnl_customer13_ins71 - start_krnl_customer13_ins71) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_supplier16", (stop_krnl_supplier1672 - start_krnl_supplier1672) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation22", (stop_krnl_aggregation2273 - start_krnl_aggregation2273) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish74 - start_finish74) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime56 - start_totalKernelTime56) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
