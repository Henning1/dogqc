#include "hip/hip_runtime.h"
#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct jpayl4 {
    int att2_ppartkey;
};
struct jpayl8 {
    int att11_pspartke;
    int att12_pssuppke;
    int att13_psavailq;
};
struct apayl7 {
    int att17_lpartkey;
    int att18_lsuppkey;
};
struct jpayl14 {
    int att12_pssuppke;
};
struct jpayl13 {
    int att32_nnationk;
};

__global__ void krnl_part1(
    int* iatt2_ppartkey, size_t* iatt3_pname_offset, char* iatt3_pname_char, agg_ht<jpayl4>* jht4) {
    int att2_ppartkey;
    str_t att3_pname;
    str_t c1 = stringConstant ( "forest%", 7);

    int tid_part1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_part1 = loopVar;
        active = (loopVar < 200000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att2_ppartkey = iatt2_ppartkey[tid_part1];
            att3_pname = stringScan ( iatt3_pname_offset, iatt3_pname_char, tid_part1);
        }
        // -------- selection (opId: 2) --------
        if(active) {
            active = stringLikeCheck ( att3_pname, c1);
        }
        // -------- hash join build (opId: 4) --------
        if(active) {
            uint64_t hash4;
            hash4 = 0;
            if(active) {
                hash4 = hash ( (hash4 + ((uint64_t)att2_ppartkey)));
            }
            int bucket = 0;
            jpayl4 payl4;
            payl4.att2_ppartkey = att2_ppartkey;
            int bucketFound = 0;
            int numLookups = 0;
            while(!(bucketFound)) {
                bucket = hashAggregateGetBucket ( jht4, 40000, hash4, numLookups, &(payl4));
                jpayl4 probepayl = jht4[bucket].payload;
                bucketFound = 1;
                bucketFound &= ((payl4.att2_ppartkey == probepayl.att2_ppartkey));
            }
        }
        loopVar += step;
    }

}

__global__ void krnl_partsupp3(
    int* iatt11_pspartke, int* iatt12_pssuppke, int* iatt13_psavailq, agg_ht<jpayl4>* jht4, unique_ht<jpayl8>* jht8) {
    int att11_pspartke;
    int att12_pssuppke;
    int att13_psavailq;
    int att2_ppartkey;

    int tid_partsupp1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_partsupp1 = loopVar;
        active = (loopVar < 800000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att11_pspartke = iatt11_pspartke[tid_partsupp1];
            att12_pssuppke = iatt12_pssuppke[tid_partsupp1];
            att13_psavailq = iatt13_psavailq[tid_partsupp1];
        }
        // -------- hash join probe (opId: 4) --------
        if(active) {
            uint64_t hash4 = 0;
            hash4 = 0;
            if(active) {
                hash4 = hash ( (hash4 + ((uint64_t)att11_pspartke)));
            }
            int numLookups4 = 0;
            int location4 = 0;
            int filterMatch4 = 0;
            int activeProbe4 = 1;
            while((!(filterMatch4) && activeProbe4)) {
                activeProbe4 = hashAggregateFindBucket ( jht4, 40000, hash4, numLookups4, location4);
                if(activeProbe4) {
                    jpayl4 probepayl = jht4[location4].payload;
                    att2_ppartkey = probepayl.att2_ppartkey;
                    filterMatch4 = 1;
                    filterMatch4 &= ((att2_ppartkey == att11_pspartke));
                }
            }
            active &= (filterMatch4);
        }
        // -------- hash join build (opId: 8) --------
        if(active) {
            jpayl8 payl8;
            payl8.att11_pspartke = att11_pspartke;
            payl8.att12_pssuppke = att12_pssuppke;
            payl8.att13_psavailq = att13_psavailq;
            uint64_t hash8;
            hash8 = 0;
            if(active) {
                hash8 = hash ( (hash8 + ((uint64_t)att11_pspartke)));
            }
            if(active) {
                hash8 = hash ( (hash8 + ((uint64_t)att12_pssuppke)));
            }
            hashBuildUnique ( jht8, 160000, hash8, &(payl8));
        }
        loopVar += step;
    }

}

__global__ void krnl_lineitem5(
    int* iatt17_lpartkey, int* iatt18_lsuppkey, int* iatt20_lquantit, unsigned* iatt26_lshipdat, agg_ht<apayl7>* aht7, float* agg1) {
    int att17_lpartkey;
    int att18_lsuppkey;
    int att20_lquantit;
    unsigned att26_lshipdat;

    int tid_lineitem1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_lineitem1 = loopVar;
        active = (loopVar < 6001215);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att17_lpartkey = iatt17_lpartkey[tid_lineitem1];
            att18_lsuppkey = iatt18_lsuppkey[tid_lineitem1];
            att20_lquantit = iatt20_lquantit[tid_lineitem1];
            att26_lshipdat = iatt26_lshipdat[tid_lineitem1];
        }
        // -------- selection (opId: 6) --------
        if(active) {
            active = ((att26_lshipdat >= 19940101) && (att26_lshipdat < 19950101));
        }
        // -------- aggregation (opId: 7) --------
        int bucket = 0;
        if(active) {
            uint64_t hash7 = 0;
            hash7 = 0;
            if(active) {
                hash7 = hash ( (hash7 + ((uint64_t)att17_lpartkey)));
            }
            if(active) {
                hash7 = hash ( (hash7 + ((uint64_t)att18_lsuppkey)));
            }
            apayl7 payl;
            payl.att17_lpartkey = att17_lpartkey;
            payl.att18_lsuppkey = att18_lsuppkey;
            int bucketFound = 0;
            int numLookups = 0;
            while(!(bucketFound)) {
                bucket = hashAggregateGetBucket ( aht7, 2400486, hash7, numLookups, &(payl));
                apayl7 probepayl = aht7[bucket].payload;
                bucketFound = 1;
                bucketFound &= ((payl.att17_lpartkey == probepayl.att17_lpartkey));
                bucketFound &= ((payl.att18_lsuppkey == probepayl.att18_lsuppkey));
            }
        }
        if(active) {
            atomicAdd(&(agg1[bucket]), ((float)att20_lquantit));
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation7(
    agg_ht<apayl7>* aht7, float* agg1, unique_ht<jpayl8>* jht8, multi_ht* jht14, jpayl14* jht14_payload) {
    int att17_lpartkey;
    int att18_lsuppkey;
    float att1_sumqty;
    int att11_pspartke;
    int att12_pssuppke;
    int att13_psavailq;

    int tid_aggregation7 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation7 = loopVar;
        active = (loopVar < 2400486);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 7) --------
        if(active) {
            active &= ((aht7[tid_aggregation7].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl7 payl = aht7[tid_aggregation7].payload;
            att17_lpartkey = payl.att17_lpartkey;
            att18_lsuppkey = payl.att18_lsuppkey;
        }
        if(active) {
            att1_sumqty = agg1[tid_aggregation7];
        }
        // -------- hash join probe (opId: 8) --------
        uint64_t hash8 = 0;
        if(active) {
            hash8 = 0;
            if(active) {
                hash8 = hash ( (hash8 + ((uint64_t)att17_lpartkey)));
            }
            if(active) {
                hash8 = hash ( (hash8 + ((uint64_t)att18_lsuppkey)));
            }
        }
        jpayl8* probepayl8;
        int numLookups8 = 0;
        if(active) {
            active = hashProbeUnique ( jht8, 160000, hash8, numLookups8, &(probepayl8));
        }
        int bucketFound8 = 0;
        int probeActive8 = active;
        while((probeActive8 && !(bucketFound8))) {
            jpayl8 jprobepayl8 = *(probepayl8);
            att11_pspartke = jprobepayl8.att11_pspartke;
            att12_pssuppke = jprobepayl8.att12_pssuppke;
            att13_psavailq = jprobepayl8.att13_psavailq;
            bucketFound8 = 1;
            bucketFound8 &= ((att11_pspartke == att17_lpartkey));
            bucketFound8 &= ((att12_pssuppke == att18_lsuppkey));
            if(!(bucketFound8)) {
                probeActive8 = hashProbeUnique ( jht8, 160000, hash8, numLookups8, &(probepayl8));
            }
        }
        active = bucketFound8;
        // -------- selection (opId: 9) --------
        if(active) {
            active = (att13_psavailq > (0.5f * att1_sumqty));
        }
        // -------- hash join build (opId: 14) --------
        if(active) {
            uint64_t hash14 = 0;
            if(active) {
                hash14 = 0;
                if(active) {
                    hash14 = hash ( (hash14 + ((uint64_t)att12_pssuppke)));
                }
            }
            hashCountMulti ( jht14, 192038, hash14);
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation7_ins(
    agg_ht<apayl7>* aht7, float* agg1, unique_ht<jpayl8>* jht8, multi_ht* jht14, jpayl14* jht14_payload, int* offs14) {
    int att17_lpartkey;
    int att18_lsuppkey;
    float att1_sumqty;
    int att11_pspartke;
    int att12_pssuppke;
    int att13_psavailq;

    int tid_aggregation7 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation7 = loopVar;
        active = (loopVar < 2400486);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 7) --------
        if(active) {
            active &= ((aht7[tid_aggregation7].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl7 payl = aht7[tid_aggregation7].payload;
            att17_lpartkey = payl.att17_lpartkey;
            att18_lsuppkey = payl.att18_lsuppkey;
        }
        if(active) {
            att1_sumqty = agg1[tid_aggregation7];
        }
        // -------- hash join probe (opId: 8) --------
        uint64_t hash8 = 0;
        if(active) {
            hash8 = 0;
            if(active) {
                hash8 = hash ( (hash8 + ((uint64_t)att17_lpartkey)));
            }
            if(active) {
                hash8 = hash ( (hash8 + ((uint64_t)att18_lsuppkey)));
            }
        }
        jpayl8* probepayl8;
        int numLookups8 = 0;
        if(active) {
            active = hashProbeUnique ( jht8, 160000, hash8, numLookups8, &(probepayl8));
        }
        int bucketFound8 = 0;
        int probeActive8 = active;
        while((probeActive8 && !(bucketFound8))) {
            jpayl8 jprobepayl8 = *(probepayl8);
            att11_pspartke = jprobepayl8.att11_pspartke;
            att12_pssuppke = jprobepayl8.att12_pssuppke;
            att13_psavailq = jprobepayl8.att13_psavailq;
            bucketFound8 = 1;
            bucketFound8 &= ((att11_pspartke == att17_lpartkey));
            bucketFound8 &= ((att12_pssuppke == att18_lsuppkey));
            if(!(bucketFound8)) {
                probeActive8 = hashProbeUnique ( jht8, 160000, hash8, numLookups8, &(probepayl8));
            }
        }
        active = bucketFound8;
        // -------- selection (opId: 9) --------
        if(active) {
            active = (att13_psavailq > (0.5f * att1_sumqty));
        }
        // -------- hash join build (opId: 14) --------
        if(active) {
            uint64_t hash14 = 0;
            if(active) {
                hash14 = 0;
                if(active) {
                    hash14 = hash ( (hash14 + ((uint64_t)att12_pssuppke)));
                }
            }
            jpayl14 payl;
            payl.att12_pssuppke = att12_pssuppke;
            hashInsertMulti ( jht14, jht14_payload, offs14, 192038, hash14, &(payl));
        }
        loopVar += step;
    }

}

__global__ void krnl_nation10(
    int* iatt32_nnationk, size_t* iatt33_nname_offset, char* iatt33_nname_char, multi_ht* jht13, jpayl13* jht13_payload) {
    int att32_nnationk;
    str_t att33_nname;
    str_t c2 = stringConstant ( "CANADA", 6);

    int tid_nation1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation1 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att32_nnationk = iatt32_nnationk[tid_nation1];
            att33_nname = stringScan ( iatt33_nname_offset, iatt33_nname_char, tid_nation1);
        }
        // -------- selection (opId: 11) --------
        if(active) {
            active = stringEquals ( att33_nname, c2);
        }
        // -------- hash join build (opId: 13) --------
        if(active) {
            uint64_t hash13 = 0;
            if(active) {
                hash13 = 0;
                if(active) {
                    hash13 = hash ( (hash13 + ((uint64_t)att32_nnationk)));
                }
            }
            hashCountMulti ( jht13, 50, hash13);
        }
        loopVar += step;
    }

}

__global__ void krnl_nation10_ins(
    int* iatt32_nnationk, size_t* iatt33_nname_offset, char* iatt33_nname_char, multi_ht* jht13, jpayl13* jht13_payload, int* offs13) {
    int att32_nnationk;
    str_t att33_nname;
    str_t c2 = stringConstant ( "CANADA", 6);

    int tid_nation1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation1 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att32_nnationk = iatt32_nnationk[tid_nation1];
            att33_nname = stringScan ( iatt33_nname_offset, iatt33_nname_char, tid_nation1);
        }
        // -------- selection (opId: 11) --------
        if(active) {
            active = stringEquals ( att33_nname, c2);
        }
        // -------- hash join build (opId: 13) --------
        if(active) {
            uint64_t hash13 = 0;
            if(active) {
                hash13 = 0;
                if(active) {
                    hash13 = hash ( (hash13 + ((uint64_t)att32_nnationk)));
                }
            }
            jpayl13 payl;
            payl.att32_nnationk = att32_nnationk;
            hashInsertMulti ( jht13, jht13_payload, offs13, 50, hash13, &(payl));
        }
        loopVar += step;
    }

}

__global__ void krnl_supplier12(
    int* iatt36_ssuppkey, size_t* iatt37_sname_offset, char* iatt37_sname_char, size_t* iatt38_saddress_offset, char* iatt38_saddress_char, int* iatt39_snationk, multi_ht* jht13, jpayl13* jht13_payload, multi_ht* jht14, jpayl14* jht14_payload, int* nout_result, str_offs* oatt37_sname_offset, str_offs* oatt38_saddress_offset) {
    int att36_ssuppkey;
    str_t att37_sname;
    str_t att38_saddress;
    int att39_snationk;
    unsigned warplane = (threadIdx.x % 32);
    int att32_nnationk;
    int att12_pssuppke;
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_supplier1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_supplier1 = loopVar;
        active = (loopVar < 10000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att36_ssuppkey = iatt36_ssuppkey[tid_supplier1];
            att37_sname = stringScan ( iatt37_sname_offset, iatt37_sname_char, tid_supplier1);
            att38_saddress = stringScan ( iatt38_saddress_offset, iatt38_saddress_char, tid_supplier1);
            att39_snationk = iatt39_snationk[tid_supplier1];
        }
        // -------- hash join probe (opId: 13) --------
        // -------- multiprobe multi broadcast (opId: 13) --------
        int matchEnd13 = 0;
        int matchEndBuf13 = 0;
        int matchOffset13 = 0;
        int matchOffsetBuf13 = 0;
        int probeActive13 = active;
        int att36_ssuppkey_bcbuf13;
        str_t att37_sname_bcbuf13;
        str_t att38_saddress_bcbuf13;
        int att39_snationk_bcbuf13;
        uint64_t hash13 = 0;
        if(probeActive13) {
            hash13 = 0;
            if(active) {
                hash13 = hash ( (hash13 + ((uint64_t)att39_snationk)));
            }
            probeActive13 = hashProbeMulti ( jht13, 50, hash13, matchOffsetBuf13, matchEndBuf13);
        }
        unsigned activeProbes13 = __ballot_sync(ALL_LANES,probeActive13);
        int num13 = 0;
        num13 = (matchEndBuf13 - matchOffsetBuf13);
        unsigned wideProbes13 = __ballot_sync(ALL_LANES,(num13 >= 32));
        att36_ssuppkey_bcbuf13 = att36_ssuppkey;
        att37_sname_bcbuf13 = att37_sname;
        att38_saddress_bcbuf13 = att38_saddress;
        att39_snationk_bcbuf13 = att39_snationk;
        while((activeProbes13 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes13)) {
                if((wideProbes13 > 0)) {
                    tupleLane = (__ffs(wideProbes13) - 1);
                    wideProbes13 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes13) - 1);
                }
                num = __shfl_sync(ALL_LANES,num13,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset13 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes13 -= (1 << tupleLane);
            }
            matchOffset13 += __shfl_sync(ALL_LANES,matchOffsetBuf13,broadcastLane);
            matchEnd13 = __shfl_sync(ALL_LANES,matchEndBuf13,broadcastLane);
            att36_ssuppkey = __shfl_sync(ALL_LANES,att36_ssuppkey_bcbuf13,broadcastLane);
            att37_sname = __shfl_sync(ALL_LANES,att37_sname_bcbuf13,broadcastLane);
            att38_saddress = __shfl_sync(ALL_LANES,att38_saddress_bcbuf13,broadcastLane);
            att39_snationk = __shfl_sync(ALL_LANES,att39_snationk_bcbuf13,broadcastLane);
            probeActive13 = (matchOffset13 < matchEnd13);
            while(__any_sync(ALL_LANES,probeActive13)) {
                active = probeActive13;
                active = 0;
                jpayl13 payl;
                if(probeActive13) {
                    payl = jht13_payload[matchOffset13];
                    att32_nnationk = payl.att32_nnationk;
                    active = 1;
                    active &= ((att32_nnationk == att39_snationk));
                    matchOffset13 += 32;
                    probeActive13 &= ((matchOffset13 < matchEnd13));
                }
                // -------- hash join probe (opId: 14) --------
                int matchEnd14 = 0;
                int matchOffset14 = 0;
                int matchStep14 = 1;
                int filterMatch14 = 0;
                int probeActive14 = active;
                uint64_t hash14 = 0;
                if(probeActive14) {
                    hash14 = 0;
                    if(active) {
                        hash14 = hash ( (hash14 + ((uint64_t)att36_ssuppkey)));
                    }
                    probeActive14 = hashProbeMulti ( jht14, 192038, hash14, matchOffset14, matchEnd14);
                }
                while(probeActive14) {
                    jpayl14 payl;
                    payl = jht14_payload[matchOffset14];
                    att12_pssuppke = payl.att12_pssuppke;
                    filterMatch14 = 1;
                    filterMatch14 &= ((att12_pssuppke == att36_ssuppkey));
                    matchOffset14 += matchStep14;
                    probeActive14 &= (!(filterMatch14));
                    probeActive14 &= ((matchOffset14 < matchEnd14));
                }
                active &= (filterMatch14);
                // -------- projection (no code) (opId: 15) --------
                // -------- materialize (opId: 16) --------
                int wp;
                int writeMask;
                int numProj;
                writeMask = __ballot_sync(ALL_LANES,active);
                numProj = __popc(writeMask);
                if((warplane == 0)) {
                    wp = atomicAdd(nout_result, numProj);
                }
                wp = __shfl_sync(ALL_LANES,wp,0);
                wp = (wp + __popc((writeMask & prefixlanes)));
                if(active) {
                    oatt37_sname_offset[wp] = toStringOffset ( iatt37_sname_char, att37_sname);
                    oatt38_saddress_offset[wp] = toStringOffset ( iatt38_saddress_char, att38_saddress);
                }
            }
        }
        loopVar += step;
    }

}

int main() {
    int* iatt2_ppartkey;
    iatt2_ppartkey = ( int*) map_memory_file ( "mmdb/part_p_partkey" );
    size_t* iatt3_pname_offset;
    iatt3_pname_offset = ( size_t*) map_memory_file ( "mmdb/part_p_name_offset" );
    char* iatt3_pname_char;
    iatt3_pname_char = ( char*) map_memory_file ( "mmdb/part_p_name_char" );
    int* iatt11_pspartke;
    iatt11_pspartke = ( int*) map_memory_file ( "mmdb/partsupp_ps_partkey" );
    int* iatt12_pssuppke;
    iatt12_pssuppke = ( int*) map_memory_file ( "mmdb/partsupp_ps_suppkey" );
    int* iatt13_psavailq;
    iatt13_psavailq = ( int*) map_memory_file ( "mmdb/partsupp_ps_availqty" );
    int* iatt17_lpartkey;
    iatt17_lpartkey = ( int*) map_memory_file ( "mmdb/lineitem_l_partkey" );
    int* iatt18_lsuppkey;
    iatt18_lsuppkey = ( int*) map_memory_file ( "mmdb/lineitem_l_suppkey" );
    int* iatt20_lquantit;
    iatt20_lquantit = ( int*) map_memory_file ( "mmdb/lineitem_l_quantity" );
    unsigned* iatt26_lshipdat;
    iatt26_lshipdat = ( unsigned*) map_memory_file ( "mmdb/lineitem_l_shipdate" );
    int* iatt32_nnationk;
    iatt32_nnationk = ( int*) map_memory_file ( "mmdb/nation_n_nationkey" );
    size_t* iatt33_nname_offset;
    iatt33_nname_offset = ( size_t*) map_memory_file ( "mmdb/nation_n_name_offset" );
    char* iatt33_nname_char;
    iatt33_nname_char = ( char*) map_memory_file ( "mmdb/nation_n_name_char" );
    int* iatt36_ssuppkey;
    iatt36_ssuppkey = ( int*) map_memory_file ( "mmdb/supplier_s_suppkey" );
    size_t* iatt37_sname_offset;
    iatt37_sname_offset = ( size_t*) map_memory_file ( "mmdb/supplier_s_name_offset" );
    char* iatt37_sname_char;
    iatt37_sname_char = ( char*) map_memory_file ( "mmdb/supplier_s_name_char" );
    size_t* iatt38_saddress_offset;
    iatt38_saddress_offset = ( size_t*) map_memory_file ( "mmdb/supplier_s_address_offset" );
    char* iatt38_saddress_char;
    iatt38_saddress_char = ( char*) map_memory_file ( "mmdb/supplier_s_address_char" );
    int* iatt39_snationk;
    iatt39_snationk = ( int*) map_memory_file ( "mmdb/supplier_s_nationkey" );

    int nout_result;
    std::vector < str_offs > oatt37_sname_offset(3840);
    std::vector < str_offs > oatt38_saddress_offset(3840);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt2_ppartkey;
    hipMalloc((void**) &d_iatt2_ppartkey, 200000* sizeof(int) );
    size_t* d_iatt3_pname_offset;
    hipMalloc((void**) &d_iatt3_pname_offset, (200000 + 1)* sizeof(size_t) );
    char* d_iatt3_pname_char;
    hipMalloc((void**) &d_iatt3_pname_char, 6550230* sizeof(char) );
    int* d_iatt11_pspartke;
    hipMalloc((void**) &d_iatt11_pspartke, 800000* sizeof(int) );
    int* d_iatt12_pssuppke;
    hipMalloc((void**) &d_iatt12_pssuppke, 800000* sizeof(int) );
    int* d_iatt13_psavailq;
    hipMalloc((void**) &d_iatt13_psavailq, 800000* sizeof(int) );
    int* d_iatt17_lpartkey;
    hipMalloc((void**) &d_iatt17_lpartkey, 6001215* sizeof(int) );
    int* d_iatt18_lsuppkey;
    hipMalloc((void**) &d_iatt18_lsuppkey, 6001215* sizeof(int) );
    int* d_iatt20_lquantit;
    hipMalloc((void**) &d_iatt20_lquantit, 6001215* sizeof(int) );
    unsigned* d_iatt26_lshipdat;
    hipMalloc((void**) &d_iatt26_lshipdat, 6001215* sizeof(unsigned) );
    int* d_iatt32_nnationk;
    hipMalloc((void**) &d_iatt32_nnationk, 25* sizeof(int) );
    size_t* d_iatt33_nname_offset;
    hipMalloc((void**) &d_iatt33_nname_offset, (25 + 1)* sizeof(size_t) );
    char* d_iatt33_nname_char;
    hipMalloc((void**) &d_iatt33_nname_char, 186* sizeof(char) );
    int* d_iatt36_ssuppkey;
    hipMalloc((void**) &d_iatt36_ssuppkey, 10000* sizeof(int) );
    size_t* d_iatt37_sname_offset;
    hipMalloc((void**) &d_iatt37_sname_offset, (10000 + 1)* sizeof(size_t) );
    char* d_iatt37_sname_char;
    hipMalloc((void**) &d_iatt37_sname_char, 180009* sizeof(char) );
    size_t* d_iatt38_saddress_offset;
    hipMalloc((void**) &d_iatt38_saddress_offset, (10000 + 1)* sizeof(size_t) );
    char* d_iatt38_saddress_char;
    hipMalloc((void**) &d_iatt38_saddress_char, 249461* sizeof(char) );
    int* d_iatt39_snationk;
    hipMalloc((void**) &d_iatt39_snationk, 10000* sizeof(int) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    str_offs* d_oatt37_sname_offset;
    hipMalloc((void**) &d_oatt37_sname_offset, 3840* sizeof(str_offs) );
    str_offs* d_oatt38_saddress_offset;
    hipMalloc((void**) &d_oatt38_saddress_offset, 3840* sizeof(str_offs) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    agg_ht<jpayl4>* d_jht4;
    hipMalloc((void**) &d_jht4, 40000* sizeof(agg_ht<jpayl4>) );
    {
        int gridsize=920;
        int blocksize=128;
        initAggHT<<<gridsize, blocksize>>>(d_jht4, 40000);
    }
    unique_ht<jpayl8>* d_jht8;
    hipMalloc((void**) &d_jht8, 160000* sizeof(unique_ht<jpayl8>) );
    {
        int gridsize=920;
        int blocksize=128;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht8, 160000);
    }
    agg_ht<apayl7>* d_aht7;
    hipMalloc((void**) &d_aht7, 2400486* sizeof(agg_ht<apayl7>) );
    {
        int gridsize=920;
        int blocksize=128;
        initAggHT<<<gridsize, blocksize>>>(d_aht7, 2400486);
    }
    float* d_agg1;
    hipMalloc((void**) &d_agg1, 2400486* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg1, 0.0f, 2400486);
    }
    multi_ht* d_jht14;
    hipMalloc((void**) &d_jht14, 192038* sizeof(multi_ht) );
    jpayl14* d_jht14_payload;
    hipMalloc((void**) &d_jht14_payload, 192038* sizeof(jpayl14) );
    {
        int gridsize=920;
        int blocksize=128;
        initMultiHT<<<gridsize, blocksize>>>(d_jht14, 192038);
    }
    int* d_offs14;
    hipMalloc((void**) &d_offs14, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs14, 0, 1);
    }
    multi_ht* d_jht13;
    hipMalloc((void**) &d_jht13, 50* sizeof(multi_ht) );
    jpayl13* d_jht13_payload;
    hipMalloc((void**) &d_jht13_payload, 50* sizeof(jpayl13) );
    {
        int gridsize=920;
        int blocksize=128;
        initMultiHT<<<gridsize, blocksize>>>(d_jht13, 50);
    }
    int* d_offs13;
    hipMalloc((void**) &d_offs13, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs13, 0, 1);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt2_ppartkey, iatt2_ppartkey, 200000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt3_pname_offset, iatt3_pname_offset, (200000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt3_pname_char, iatt3_pname_char, 6550230 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt11_pspartke, iatt11_pspartke, 800000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt12_pssuppke, iatt12_pssuppke, 800000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt13_psavailq, iatt13_psavailq, 800000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt17_lpartkey, iatt17_lpartkey, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt18_lsuppkey, iatt18_lsuppkey, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt20_lquantit, iatt20_lquantit, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt26_lshipdat, iatt26_lshipdat, 6001215 * sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt32_nnationk, iatt32_nnationk, 25 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt33_nname_offset, iatt33_nname_offset, (25 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt33_nname_char, iatt33_nname_char, 186 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt36_ssuppkey, iatt36_ssuppkey, 10000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt37_sname_offset, iatt37_sname_offset, (10000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt37_sname_char, iatt37_sname_char, 180009 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt38_saddress_offset, iatt38_saddress_offset, (10000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt38_saddress_char, iatt38_saddress_char, 249461 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt39_snationk, iatt39_snationk, 10000 * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime166 = std::clock();
    std::clock_t start_krnl_part1167 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_part1<<<gridsize, blocksize>>>(d_iatt2_ppartkey, d_iatt3_pname_offset, d_iatt3_pname_char, d_jht4);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_part1167 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_part1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_part1")
        }
    }

    std::clock_t start_krnl_partsupp3168 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_partsupp3<<<gridsize, blocksize>>>(d_iatt11_pspartke, d_iatt12_pssuppke, d_iatt13_psavailq, d_jht4, d_jht8);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_partsupp3168 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_partsupp3! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_partsupp3")
        }
    }

    std::clock_t start_krnl_lineitem5169 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_lineitem5<<<gridsize, blocksize>>>(d_iatt17_lpartkey, d_iatt18_lsuppkey, d_iatt20_lquantit, d_iatt26_lshipdat, d_aht7, d_agg1);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_lineitem5169 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem5! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem5")
        }
    }

    std::clock_t start_krnl_aggregation7170 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation7<<<gridsize, blocksize>>>(d_aht7, d_agg1, d_jht8, d_jht14, d_jht14_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation7170 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation7! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation7")
        }
    }

    std::clock_t start_scanMultiHT171 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht14, 192038, d_offs14);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT171 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT")
        }
    }

    std::clock_t start_krnl_aggregation7_ins172 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation7_ins<<<gridsize, blocksize>>>(d_aht7, d_agg1, d_jht8, d_jht14, d_jht14_payload, d_offs14);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation7_ins172 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation7_ins! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation7_ins")
        }
    }

    std::clock_t start_krnl_nation10173 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_nation10<<<gridsize, blocksize>>>(d_iatt32_nnationk, d_iatt33_nname_offset, d_iatt33_nname_char, d_jht13, d_jht13_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation10173 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation10! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation10")
        }
    }

    std::clock_t start_scanMultiHT174 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht13, 50, d_offs13);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT174 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT")
        }
    }

    std::clock_t start_krnl_nation10_ins175 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_nation10_ins<<<gridsize, blocksize>>>(d_iatt32_nnationk, d_iatt33_nname_offset, d_iatt33_nname_char, d_jht13, d_jht13_payload, d_offs13);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation10_ins175 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation10_ins! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation10_ins")
        }
    }

    std::clock_t start_krnl_supplier12176 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_supplier12<<<gridsize, blocksize>>>(d_iatt36_ssuppkey, d_iatt37_sname_offset, d_iatt37_sname_char, d_iatt38_saddress_offset, d_iatt38_saddress_char, d_iatt39_snationk, d_jht13, d_jht13_payload, d_jht14, d_jht14_payload, d_nout_result, d_oatt37_sname_offset, d_oatt38_saddress_offset);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_supplier12176 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_supplier12! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_supplier12")
        }
    }

    std::clock_t stop_totalKernelTime166 = std::clock();
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt37_sname_offset.data(), d_oatt37_sname_offset, 3840 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt38_saddress_offset.data(), d_oatt38_saddress_offset, 3840 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt2_ppartkey);
    hipFree( d_iatt3_pname_offset);
    hipFree( d_iatt3_pname_char);
    hipFree( d_jht4);
    hipFree( d_iatt11_pspartke);
    hipFree( d_iatt12_pssuppke);
    hipFree( d_iatt13_psavailq);
    hipFree( d_jht8);
    hipFree( d_iatt17_lpartkey);
    hipFree( d_iatt18_lsuppkey);
    hipFree( d_iatt20_lquantit);
    hipFree( d_iatt26_lshipdat);
    hipFree( d_aht7);
    hipFree( d_agg1);
    hipFree( d_jht14);
    hipFree( d_jht14_payload);
    hipFree( d_offs14);
    hipFree( d_iatt32_nnationk);
    hipFree( d_iatt33_nname_offset);
    hipFree( d_iatt33_nname_char);
    hipFree( d_jht13);
    hipFree( d_jht13_payload);
    hipFree( d_offs13);
    hipFree( d_iatt36_ssuppkey);
    hipFree( d_iatt37_sname_offset);
    hipFree( d_iatt37_sname_char);
    hipFree( d_iatt38_saddress_offset);
    hipFree( d_iatt38_saddress_char);
    hipFree( d_iatt39_snationk);
    hipFree( d_nout_result);
    hipFree( d_oatt37_sname_offset);
    hipFree( d_oatt38_saddress_offset);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish177 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 3840)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("s_name: ");
        stringPrint ( iatt37_sname_char, oatt37_sname_offset[pv]);
        printf("  ");
        printf("s_address: ");
        stringPrint ( iatt38_saddress_char, oatt38_saddress_offset[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish177 = std::clock();

    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "krnl_part1", (stop_krnl_part1167 - start_krnl_part1167) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_partsupp3", (stop_krnl_partsupp3168 - start_krnl_partsupp3168) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_lineitem5", (stop_krnl_lineitem5169 - start_krnl_lineitem5169) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation7", (stop_krnl_aggregation7170 - start_krnl_aggregation7170) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "scanMultiHT", (stop_scanMultiHT171 - start_scanMultiHT171) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation7_ins", (stop_krnl_aggregation7_ins172 - start_krnl_aggregation7_ins172) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_nation10", (stop_krnl_nation10173 - start_krnl_nation10173) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "scanMultiHT", (stop_scanMultiHT174 - start_scanMultiHT174) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_nation10_ins", (stop_krnl_nation10_ins175 - start_krnl_nation10_ins175) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_supplier12", (stop_krnl_supplier12176 - start_krnl_supplier12176) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish177 - start_finish177) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime166 - start_totalKernelTime166) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
