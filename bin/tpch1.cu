#include "hip/hip_runtime.h"
#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct apayl5 {
    char att17_lreturnf;
    char att18_llinesta;
};

__global__ void krnl_lineitem1(
    int* iatt13_lquantit, float* iatt14_lextende, float* iatt15_ldiscoun, float* iatt16_ltax, char* iatt17_lreturnf, char* iatt18_llinesta, unsigned* iatt19_lshipdat, agg_ht<apayl5>* aht5, float* agg1, float* agg2, float* agg3, float* agg4, float* agg5, float* agg6, float* agg7, int* agg8) {
    int att13_lquantit;
    float att14_lextende;
    float att15_ldiscoun;
    float att16_ltax;
    char att17_lreturnf;
    char att18_llinesta;
    unsigned att19_lshipdat;
    float att25_charge;
    float att26_discpric;

    int tid_lineitem1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_lineitem1 = loopVar;
        active = (loopVar < 6001215);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att13_lquantit = iatt13_lquantit[tid_lineitem1];
            att14_lextende = iatt14_lextende[tid_lineitem1];
            att15_ldiscoun = iatt15_ldiscoun[tid_lineitem1];
            att16_ltax = iatt16_ltax[tid_lineitem1];
            att17_lreturnf = iatt17_lreturnf[tid_lineitem1];
            att18_llinesta = iatt18_llinesta[tid_lineitem1];
            att19_lshipdat = iatt19_lshipdat[tid_lineitem1];
        }
        // -------- selection (opId: 2) --------
        if(active) {
            active = (att19_lshipdat <= 19980902);
        }
        // -------- map (opId: 3) --------
        if(active) {
            att25_charge = ((att14_lextende * ((float)1.0f - att15_ldiscoun)) * ((float)1.0f + att16_ltax));
        }
        // -------- map (opId: 4) --------
        if(active) {
            att26_discpric = (att14_lextende * ((float)1.0f - att15_ldiscoun));
        }
        // -------- aggregation (opId: 5) --------
        int bucket = 0;
        if(active) {
            uint64_t hash5 = 0;
            hash5 = 0;
            if(active) {
                hash5 = hash ( (hash5 + ((uint64_t)att17_lreturnf)));
            }
            if(active) {
                hash5 = hash ( (hash5 + ((uint64_t)att18_llinesta)));
            }
            apayl5 payl;
            payl.att17_lreturnf = att17_lreturnf;
            payl.att18_llinesta = att18_llinesta;
            int bucketFound = 0;
            int numLookups = 0;
            while(!(bucketFound)) {
                bucket = hashAggregateGetBucket ( aht5, 200, hash5, numLookups, &(payl));
                apayl5 probepayl = aht5[bucket].payload;
                bucketFound = 1;
                bucketFound &= ((payl.att17_lreturnf == probepayl.att17_lreturnf));
                bucketFound &= ((payl.att18_llinesta == probepayl.att18_llinesta));
            }
        }
        if(active) {
            atomicAdd(&(agg1[bucket]), ((float)att13_lquantit));
            atomicAdd(&(agg2[bucket]), ((float)att14_lextende));
            atomicAdd(&(agg3[bucket]), ((float)att26_discpric));
            atomicAdd(&(agg4[bucket]), ((float)att25_charge));
            atomicAdd(&(agg5[bucket]), ((float)att13_lquantit));
            atomicAdd(&(agg6[bucket]), ((float)att14_lextende));
            atomicAdd(&(agg7[bucket]), ((float)att15_ldiscoun));
            atomicAdd(&(agg8[bucket]), ((int)1));
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation5(
    agg_ht<apayl5>* aht5, float* agg1, float* agg2, float* agg3, float* agg4, float* agg5, float* agg6, float* agg7, int* agg8, int* nout_result, char* oatt17_lreturnf, char* oatt18_llinesta, float* oatt1_sumqty, float* oatt2_sumbasep, float* oatt3_sumdiscp, float* oatt5_avgqty, float* oatt6_avgprice, float* oatt7_avgdisc, int* oatt8_countord) {
    char att17_lreturnf;
    char att18_llinesta;
    float att1_sumqty;
    float att2_sumbasep;
    float att3_sumdiscp;
    float att4_sumcharg;
    float att5_avgqty;
    float att6_avgprice;
    float att7_avgdisc;
    int att8_countord;
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_aggregation5 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation5 = loopVar;
        active = (loopVar < 200);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 5) --------
        if(active) {
            active &= ((aht5[tid_aggregation5].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl5 payl = aht5[tid_aggregation5].payload;
            att17_lreturnf = payl.att17_lreturnf;
            att18_llinesta = payl.att18_llinesta;
        }
        if(active) {
            att1_sumqty = agg1[tid_aggregation5];
            att2_sumbasep = agg2[tid_aggregation5];
            att3_sumdiscp = agg3[tid_aggregation5];
            att4_sumcharg = agg4[tid_aggregation5];
            att5_avgqty = agg5[tid_aggregation5];
            att6_avgprice = agg6[tid_aggregation5];
            att7_avgdisc = agg7[tid_aggregation5];
            att8_countord = agg8[tid_aggregation5];
            att5_avgqty = (att5_avgqty / ((float)att8_countord));
            att6_avgprice = (att6_avgprice / ((float)att8_countord));
            att7_avgdisc = (att7_avgdisc / ((float)att8_countord));
        }
        // -------- projection (no code) (opId: 6) --------
        // -------- materialize (opId: 7) --------
        int wp;
        int writeMask;
        int numProj;
        writeMask = __ballot_sync(ALL_LANES,active);
        numProj = __popc(writeMask);
        if((warplane == 0)) {
            wp = atomicAdd(nout_result, numProj);
        }
        wp = __shfl_sync(ALL_LANES,wp,0);
        wp = (wp + __popc((writeMask & prefixlanes)));
        if(active) {
            oatt17_lreturnf[wp] = att17_lreturnf;
            oatt18_llinesta[wp] = att18_llinesta;
            oatt1_sumqty[wp] = att1_sumqty;
            oatt2_sumbasep[wp] = att2_sumbasep;
            oatt3_sumdiscp[wp] = att3_sumdiscp;
            oatt5_avgqty[wp] = att5_avgqty;
            oatt6_avgprice[wp] = att6_avgprice;
            oatt7_avgdisc[wp] = att7_avgdisc;
            oatt8_countord[wp] = att8_countord;
        }
        loopVar += step;
    }

}

int main() {
    int* iatt13_lquantit;
    iatt13_lquantit = ( int*) map_memory_file ( "mmdb/lineitem_l_quantity" );
    float* iatt14_lextende;
    iatt14_lextende = ( float*) map_memory_file ( "mmdb/lineitem_l_extendedprice" );
    float* iatt15_ldiscoun;
    iatt15_ldiscoun = ( float*) map_memory_file ( "mmdb/lineitem_l_discount" );
    float* iatt16_ltax;
    iatt16_ltax = ( float*) map_memory_file ( "mmdb/lineitem_l_tax" );
    char* iatt17_lreturnf;
    iatt17_lreturnf = ( char*) map_memory_file ( "mmdb/lineitem_l_returnflag" );
    char* iatt18_llinesta;
    iatt18_llinesta = ( char*) map_memory_file ( "mmdb/lineitem_l_linestatus" );
    unsigned* iatt19_lshipdat;
    iatt19_lshipdat = ( unsigned*) map_memory_file ( "mmdb/lineitem_l_shipdate" );

    int nout_result;
    std::vector < char > oatt17_lreturnf(100);
    std::vector < char > oatt18_llinesta(100);
    std::vector < float > oatt1_sumqty(100);
    std::vector < float > oatt2_sumbasep(100);
    std::vector < float > oatt3_sumdiscp(100);
    std::vector < float > oatt5_avgqty(100);
    std::vector < float > oatt6_avgprice(100);
    std::vector < float > oatt7_avgdisc(100);
    std::vector < int > oatt8_countord(100);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt13_lquantit;
    hipMalloc((void**) &d_iatt13_lquantit, 6001215* sizeof(int) );
    float* d_iatt14_lextende;
    hipMalloc((void**) &d_iatt14_lextende, 6001215* sizeof(float) );
    float* d_iatt15_ldiscoun;
    hipMalloc((void**) &d_iatt15_ldiscoun, 6001215* sizeof(float) );
    float* d_iatt16_ltax;
    hipMalloc((void**) &d_iatt16_ltax, 6001215* sizeof(float) );
    char* d_iatt17_lreturnf;
    hipMalloc((void**) &d_iatt17_lreturnf, 6001215* sizeof(char) );
    char* d_iatt18_llinesta;
    hipMalloc((void**) &d_iatt18_llinesta, 6001215* sizeof(char) );
    unsigned* d_iatt19_lshipdat;
    hipMalloc((void**) &d_iatt19_lshipdat, 6001215* sizeof(unsigned) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    char* d_oatt17_lreturnf;
    hipMalloc((void**) &d_oatt17_lreturnf, 100* sizeof(char) );
    char* d_oatt18_llinesta;
    hipMalloc((void**) &d_oatt18_llinesta, 100* sizeof(char) );
    float* d_oatt1_sumqty;
    hipMalloc((void**) &d_oatt1_sumqty, 100* sizeof(float) );
    float* d_oatt2_sumbasep;
    hipMalloc((void**) &d_oatt2_sumbasep, 100* sizeof(float) );
    float* d_oatt3_sumdiscp;
    hipMalloc((void**) &d_oatt3_sumdiscp, 100* sizeof(float) );
    float* d_oatt5_avgqty;
    hipMalloc((void**) &d_oatt5_avgqty, 100* sizeof(float) );
    float* d_oatt6_avgprice;
    hipMalloc((void**) &d_oatt6_avgprice, 100* sizeof(float) );
    float* d_oatt7_avgdisc;
    hipMalloc((void**) &d_oatt7_avgdisc, 100* sizeof(float) );
    int* d_oatt8_countord;
    hipMalloc((void**) &d_oatt8_countord, 100* sizeof(int) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    agg_ht<apayl5>* d_aht5;
    hipMalloc((void**) &d_aht5, 200* sizeof(agg_ht<apayl5>) );
    {
        int gridsize=920;
        int blocksize=128;
        initAggHT<<<gridsize, blocksize>>>(d_aht5, 200);
    }
    float* d_agg1;
    hipMalloc((void**) &d_agg1, 200* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg1, 0.0f, 200);
    }
    float* d_agg2;
    hipMalloc((void**) &d_agg2, 200* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg2, 0.0f, 200);
    }
    float* d_agg3;
    hipMalloc((void**) &d_agg3, 200* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg3, 0.0f, 200);
    }
    float* d_agg4;
    hipMalloc((void**) &d_agg4, 200* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg4, 0.0f, 200);
    }
    float* d_agg5;
    hipMalloc((void**) &d_agg5, 200* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg5, 0.0f, 200);
    }
    float* d_agg6;
    hipMalloc((void**) &d_agg6, 200* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg6, 0.0f, 200);
    }
    float* d_agg7;
    hipMalloc((void**) &d_agg7, 200* sizeof(float) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg7, 0.0f, 200);
    }
    int* d_agg8;
    hipMalloc((void**) &d_agg8, 200* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg8, 0, 200);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt13_lquantit, iatt13_lquantit, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt14_lextende, iatt14_lextende, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt15_ldiscoun, iatt15_ldiscoun, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt16_ltax, iatt16_ltax, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt17_lreturnf, iatt17_lreturnf, 6001215 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt18_llinesta, iatt18_llinesta, 6001215 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt19_lshipdat, iatt19_lshipdat, 6001215 * sizeof(unsigned), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime0 = std::clock();
    std::clock_t start_krnl_lineitem11 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_lineitem1<<<gridsize, blocksize>>>(d_iatt13_lquantit, d_iatt14_lextende, d_iatt15_ldiscoun, d_iatt16_ltax, d_iatt17_lreturnf, d_iatt18_llinesta, d_iatt19_lshipdat, d_aht5, d_agg1, d_agg2, d_agg3, d_agg4, d_agg5, d_agg6, d_agg7, d_agg8);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_lineitem11 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem1")
        }
    }

    std::clock_t start_krnl_aggregation52 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation5<<<gridsize, blocksize>>>(d_aht5, d_agg1, d_agg2, d_agg3, d_agg4, d_agg5, d_agg6, d_agg7, d_agg8, d_nout_result, d_oatt17_lreturnf, d_oatt18_llinesta, d_oatt1_sumqty, d_oatt2_sumbasep, d_oatt3_sumdiscp, d_oatt5_avgqty, d_oatt6_avgprice, d_oatt7_avgdisc, d_oatt8_countord);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation52 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation5! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation5")
        }
    }

    std::clock_t stop_totalKernelTime0 = std::clock();
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt17_lreturnf.data(), d_oatt17_lreturnf, 100 * sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy( oatt18_llinesta.data(), d_oatt18_llinesta, 100 * sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy( oatt1_sumqty.data(), d_oatt1_sumqty, 100 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( oatt2_sumbasep.data(), d_oatt2_sumbasep, 100 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( oatt3_sumdiscp.data(), d_oatt3_sumdiscp, 100 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( oatt5_avgqty.data(), d_oatt5_avgqty, 100 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( oatt6_avgprice.data(), d_oatt6_avgprice, 100 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( oatt7_avgdisc.data(), d_oatt7_avgdisc, 100 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( oatt8_countord.data(), d_oatt8_countord, 100 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt13_lquantit);
    hipFree( d_iatt14_lextende);
    hipFree( d_iatt15_ldiscoun);
    hipFree( d_iatt16_ltax);
    hipFree( d_iatt17_lreturnf);
    hipFree( d_iatt18_llinesta);
    hipFree( d_iatt19_lshipdat);
    hipFree( d_aht5);
    hipFree( d_agg1);
    hipFree( d_agg2);
    hipFree( d_agg3);
    hipFree( d_agg4);
    hipFree( d_agg5);
    hipFree( d_agg6);
    hipFree( d_agg7);
    hipFree( d_agg8);
    hipFree( d_nout_result);
    hipFree( d_oatt17_lreturnf);
    hipFree( d_oatt18_llinesta);
    hipFree( d_oatt1_sumqty);
    hipFree( d_oatt2_sumbasep);
    hipFree( d_oatt3_sumdiscp);
    hipFree( d_oatt5_avgqty);
    hipFree( d_oatt6_avgprice);
    hipFree( d_oatt7_avgdisc);
    hipFree( d_oatt8_countord);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish3 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 100)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("l_returnflag: ");
        printf("%c", oatt17_lreturnf[pv]);
        printf("  ");
        printf("l_linestatus: ");
        printf("%c", oatt18_llinesta[pv]);
        printf("  ");
        printf("sum_qty: ");
        printf("%15.2f", oatt1_sumqty[pv]);
        printf("  ");
        printf("sum_base_price: ");
        printf("%15.2f", oatt2_sumbasep[pv]);
        printf("  ");
        printf("sum_disc_price: ");
        printf("%15.2f", oatt3_sumdiscp[pv]);
        printf("  ");
        printf("avg_qty: ");
        printf("%15.2f", oatt5_avgqty[pv]);
        printf("  ");
        printf("avg_price: ");
        printf("%15.2f", oatt6_avgprice[pv]);
        printf("  ");
        printf("avg_disc: ");
        printf("%15.2f", oatt7_avgdisc[pv]);
        printf("  ");
        printf("count_order: ");
        printf("%8i", oatt8_countord[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish3 = std::clock();

    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "krnl_lineitem1", (stop_krnl_lineitem11 - start_krnl_lineitem11) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation5", (stop_krnl_aggregation52 - start_krnl_aggregation52) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish3 - start_finish3) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime0 - start_totalKernelTime0) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
