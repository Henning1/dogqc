#include "hip/hip_runtime.h"
#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct jpayl6 {
    int att2_nnationk;
    str_t att3_nname;
};
struct jpayl5 {
    int att6_oorderke;
    int att7_ocustkey;
};
struct jpayl9 {
    str_t att3_nname;
    int att6_oorderke;
    int att15_ccustkey;
    str_t att16_cname;
    str_t att17_caddress;
    str_t att19_cphone;
    float att20_cacctbal;
    str_t att22_ccomment;
};
struct apayl11 {
    int att15_ccustkey;
    str_t att16_cname;
    float att20_cacctbal;
    str_t att19_cphone;
    str_t att3_nname;
    str_t att17_caddress;
    str_t att22_ccomment;
};

__global__ void krnl_nation1(
    int* iatt2_nnationk, size_t* iatt3_nname_offset, char* iatt3_nname_char, unique_ht<jpayl6>* jht6) {
    int att2_nnationk;
    str_t att3_nname;

    int tid_nation1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_nation1 = loopVar;
        active = (loopVar < 25);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att2_nnationk = iatt2_nnationk[tid_nation1];
            att3_nname = stringScan ( iatt3_nname_offset, iatt3_nname_char, tid_nation1);
        }
        // -------- hash join build (opId: 6) --------
        if(active) {
            jpayl6 payl6;
            payl6.att2_nnationk = att2_nnationk;
            payl6.att3_nname = att3_nname;
            uint64_t hash6;
            hash6 = 0;
            if(active) {
                hash6 = hash ( (hash6 + ((uint64_t)att2_nnationk)));
            }
            hashBuildUnique ( jht6, 50, hash6, &(payl6));
        }
        loopVar += step;
    }

}

__global__ void krnl_orders2(
    int* iatt6_oorderke, int* iatt7_ocustkey, unsigned* iatt10_oorderda, multi_ht* jht5, jpayl5* jht5_payload) {
    int att6_oorderke;
    int att7_ocustkey;
    unsigned att10_oorderda;

    int tid_orders1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_orders1 = loopVar;
        active = (loopVar < 1500000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att6_oorderke = iatt6_oorderke[tid_orders1];
            att7_ocustkey = iatt7_ocustkey[tid_orders1];
            att10_oorderda = iatt10_oorderda[tid_orders1];
        }
        // -------- selection (opId: 3) --------
        if(active) {
            active = ((att10_oorderda >= 19931001) && (att10_oorderda < 19940101));
        }
        // -------- hash join build (opId: 5) --------
        if(active) {
            uint64_t hash5 = 0;
            if(active) {
                hash5 = 0;
                if(active) {
                    hash5 = hash ( (hash5 + ((uint64_t)att7_ocustkey)));
                }
            }
            hashCountMulti ( jht5, 150000, hash5);
        }
        loopVar += step;
    }

}

__global__ void krnl_orders2_ins(
    int* iatt6_oorderke, int* iatt7_ocustkey, unsigned* iatt10_oorderda, multi_ht* jht5, jpayl5* jht5_payload, int* offs5) {
    int att6_oorderke;
    int att7_ocustkey;
    unsigned att10_oorderda;

    int tid_orders1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_orders1 = loopVar;
        active = (loopVar < 1500000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att6_oorderke = iatt6_oorderke[tid_orders1];
            att7_ocustkey = iatt7_ocustkey[tid_orders1];
            att10_oorderda = iatt10_oorderda[tid_orders1];
        }
        // -------- selection (opId: 3) --------
        if(active) {
            active = ((att10_oorderda >= 19931001) && (att10_oorderda < 19940101));
        }
        // -------- hash join build (opId: 5) --------
        if(active) {
            uint64_t hash5 = 0;
            if(active) {
                hash5 = 0;
                if(active) {
                    hash5 = hash ( (hash5 + ((uint64_t)att7_ocustkey)));
                }
            }
            jpayl5 payl;
            payl.att6_oorderke = att6_oorderke;
            payl.att7_ocustkey = att7_ocustkey;
            hashInsertMulti ( jht5, jht5_payload, offs5, 150000, hash5, &(payl));
        }
        loopVar += step;
    }

}

__global__ void krnl_customer4(
    int* iatt15_ccustkey, size_t* iatt16_cname_offset, char* iatt16_cname_char, size_t* iatt17_caddress_offset, char* iatt17_caddress_char, int* iatt18_cnationk, size_t* iatt19_cphone_offset, char* iatt19_cphone_char, float* iatt20_cacctbal, size_t* iatt22_ccomment_offset, char* iatt22_ccomment_char, multi_ht* jht5, jpayl5* jht5_payload, unique_ht<jpayl6>* jht6, unique_ht<jpayl9>* jht9) {
    int att15_ccustkey;
    str_t att16_cname;
    str_t att17_caddress;
    int att18_cnationk;
    str_t att19_cphone;
    float att20_cacctbal;
    str_t att22_ccomment;
    int att6_oorderke;
    int att7_ocustkey;
    int att2_nnationk;
    str_t att3_nname;

    int tid_customer1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_customer1 = loopVar;
        active = (loopVar < 150000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att15_ccustkey = iatt15_ccustkey[tid_customer1];
            att16_cname = stringScan ( iatt16_cname_offset, iatt16_cname_char, tid_customer1);
            att17_caddress = stringScan ( iatt17_caddress_offset, iatt17_caddress_char, tid_customer1);
            att18_cnationk = iatt18_cnationk[tid_customer1];
            att19_cphone = stringScan ( iatt19_cphone_offset, iatt19_cphone_char, tid_customer1);
            att20_cacctbal = iatt20_cacctbal[tid_customer1];
            att22_ccomment = stringScan ( iatt22_ccomment_offset, iatt22_ccomment_char, tid_customer1);
        }
        // -------- hash join probe (opId: 5) --------
        int matchEnd5 = 0;
        int matchOffset5 = 0;
        int matchStep5 = 1;
        int matchFound5 = 0;
        int probeActive5 = active;
        uint64_t hash5 = 0;
        if(probeActive5) {
            hash5 = 0;
            if(active) {
                hash5 = hash ( (hash5 + ((uint64_t)att15_ccustkey)));
            }
            probeActive5 = hashProbeMulti ( jht5, 150000, hash5, matchOffset5, matchEnd5);
        }
        active = probeActive5;
        while(__any_sync(ALL_LANES,active)) {
            probeActive5 = active;
            jpayl5 payl;
            if(probeActive5) {
                payl = jht5_payload[matchOffset5];
                att6_oorderke = payl.att6_oorderke;
                att7_ocustkey = payl.att7_ocustkey;
                active &= ((att7_ocustkey == att15_ccustkey));
                matchFound5 += active;
            }
            // -------- hash join probe (opId: 6) --------
            uint64_t hash6 = 0;
            if(active) {
                hash6 = 0;
                if(active) {
                    hash6 = hash ( (hash6 + ((uint64_t)att18_cnationk)));
                }
            }
            jpayl6* probepayl6;
            int numLookups6 = 0;
            if(active) {
                active = hashProbeUnique ( jht6, 50, hash6, numLookups6, &(probepayl6));
            }
            int bucketFound6 = 0;
            int probeActive6 = active;
            while((probeActive6 && !(bucketFound6))) {
                jpayl6 jprobepayl6 = *(probepayl6);
                att2_nnationk = jprobepayl6.att2_nnationk;
                att3_nname = jprobepayl6.att3_nname;
                bucketFound6 = 1;
                bucketFound6 &= ((att2_nnationk == att18_cnationk));
                if(!(bucketFound6)) {
                    probeActive6 = hashProbeUnique ( jht6, 50, hash6, numLookups6, &(probepayl6));
                }
            }
            active = bucketFound6;
            // -------- hash join build (opId: 9) --------
            if(active) {
                jpayl9 payl9;
                payl9.att3_nname = att3_nname;
                payl9.att6_oorderke = att6_oorderke;
                payl9.att15_ccustkey = att15_ccustkey;
                payl9.att16_cname = att16_cname;
                payl9.att17_caddress = att17_caddress;
                payl9.att19_cphone = att19_cphone;
                payl9.att20_cacctbal = att20_cacctbal;
                payl9.att22_ccomment = att22_ccomment;
                uint64_t hash9;
                hash9 = 0;
                if(active) {
                    hash9 = hash ( (hash9 + ((uint64_t)att6_oorderke)));
                }
                hashBuildUnique ( jht9, 300000, hash9, &(payl9));
            }
            matchOffset5 += matchStep5;
            probeActive5 &= ((matchOffset5 < matchEnd5));
            active = probeActive5;
        }
        loopVar += step;
    }

}

__global__ void krnl_lineitem7(
    int* iatt23_lorderke, float* iatt28_lextende, float* iatt29_ldiscoun, char* iatt31_lreturnf, unique_ht<jpayl9>* jht9, agg_ht<apayl11>* aht11, float* agg1) {
    int att23_lorderke;
    float att28_lextende;
    float att29_ldiscoun;
    char att31_lreturnf;
    str_t att3_nname;
    int att6_oorderke;
    int att15_ccustkey;
    str_t att16_cname;
    str_t att17_caddress;
    str_t att19_cphone;
    float att20_cacctbal;
    str_t att22_ccomment;
    int buffercount100009_ = 0;
    unsigned warpid = (threadIdx.x / 32);
    int bufferBase100009_ = (warpid * 32);
    // shared memory variables for divergence buffers
    __shared__ str_t att3_nname_dvgnce_buf_100009_[32];
    __shared__ int att15_ccustkey_dvgnce_buf_100009_[32];
    __shared__ str_t att16_cname_dvgnce_buf_100009_[32];
    __shared__ str_t att17_caddress_dvgnce_buf_100009_[32];
    __shared__ str_t att19_cphone_dvgnce_buf_100009_[32];
    __shared__ float att20_cacctbal_dvgnce_buf_100009_[32];
    __shared__ str_t att22_ccomment_dvgnce_buf_100009_[32];
    __shared__ float att28_lextende_dvgnce_buf_100009_[32];
    __shared__ float att29_ldiscoun_dvgnce_buf_100009_[32];
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));
    float att39_rev;

    int tid_lineitem1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_lineitem1 = loopVar;
        active = (loopVar < 6001215);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att23_lorderke = iatt23_lorderke[tid_lineitem1];
            att28_lextende = iatt28_lextende[tid_lineitem1];
            att29_ldiscoun = iatt29_ldiscoun[tid_lineitem1];
            att31_lreturnf = iatt31_lreturnf[tid_lineitem1];
        }
        // -------- selection (opId: 8) --------
        if(active) {
            active = (att31_lreturnf == 'R');
        }
        // -------- hash join probe (opId: 9) --------
        uint64_t hash9 = 0;
        if(active) {
            hash9 = 0;
            if(active) {
                hash9 = hash ( (hash9 + ((uint64_t)att23_lorderke)));
            }
        }
        jpayl9* probepayl9;
        int numLookups9 = 0;
        if(active) {
            active = hashProbeUnique ( jht9, 300000, hash9, numLookups9, &(probepayl9));
        }
        int bucketFound9 = 0;
        int probeActive9 = active;
        while((probeActive9 && !(bucketFound9))) {
            jpayl9 jprobepayl9 = *(probepayl9);
            att3_nname = jprobepayl9.att3_nname;
            att6_oorderke = jprobepayl9.att6_oorderke;
            att15_ccustkey = jprobepayl9.att15_ccustkey;
            att16_cname = jprobepayl9.att16_cname;
            att17_caddress = jprobepayl9.att17_caddress;
            att19_cphone = jprobepayl9.att19_cphone;
            att20_cacctbal = jprobepayl9.att20_cacctbal;
            att22_ccomment = jprobepayl9.att22_ccomment;
            bucketFound9 = 1;
            bucketFound9 &= ((att6_oorderke == att23_lorderke));
            if(!(bucketFound9)) {
                probeActive9 = hashProbeUnique ( jht9, 300000, hash9, numLookups9, &(probepayl9));
            }
        }
        active = bucketFound9;
        // -------- divergence buffer (opId: 100009) --------
        // ensures that the thread activity in each warp (32 threads) lies above a given threshold
        // depending on the buffer count inactive lanes are either refilled or flushed to the buffer
        int activemask100009_ = __ballot_sync(ALL_LANES,active);
        int numactive100009_ = __popc(activemask100009_);
        int scan100009_;
        int remaining100009_;
        int bufIdx100009_;
        int minTuplesInFlight100009_ = (flushPipeline) ? (0) : (28);
        while(((buffercount100009_ + numactive100009_) > minTuplesInFlight100009_)) {
            // refill inactive lanes from shared memory buffer
            if(((numactive100009_ < 28) && buffercount100009_)) {
                remaining100009_ = max(((buffercount100009_ + numactive100009_) - 32), 0);
                // prefix scan of inactive lanes
                scan100009_ = __popc((~(activemask100009_) & prefixlanes));
                // gather buffered data (tids, datastructure state, computed values)
                if((!(active) && (scan100009_ < buffercount100009_))) {
                    bufIdx100009_ = (remaining100009_ + (scan100009_ + bufferBase100009_));
                    att3_nname = att3_nname_dvgnce_buf_100009_[bufIdx100009_];
                    att15_ccustkey = att15_ccustkey_dvgnce_buf_100009_[bufIdx100009_];
                    att16_cname = att16_cname_dvgnce_buf_100009_[bufIdx100009_];
                    att17_caddress = att17_caddress_dvgnce_buf_100009_[bufIdx100009_];
                    att19_cphone = att19_cphone_dvgnce_buf_100009_[bufIdx100009_];
                    att20_cacctbal = att20_cacctbal_dvgnce_buf_100009_[bufIdx100009_];
                    att22_ccomment = att22_ccomment_dvgnce_buf_100009_[bufIdx100009_];
                    att28_lextende = att28_lextende_dvgnce_buf_100009_[bufIdx100009_];
                    att29_ldiscoun = att29_ldiscoun_dvgnce_buf_100009_[bufIdx100009_];
                    active = 1;
                }
                // decrement buffer count
                buffercount100009_ = remaining100009_;
            }
            // -------- map (opId: 10) --------
            if(active) {
                att39_rev = (att28_lextende * ((float)1.0f - att29_ldiscoun));
            }
            // -------- aggregation (opId: 11) --------
            int bucket = 0;
            if(active) {
                uint64_t hash11 = 0;
                hash11 = 0;
                if(active) {
                    hash11 = hash ( (hash11 + ((uint64_t)att15_ccustkey)));
                }
                hash11 = hash ( (hash11 + stringHash ( att16_cname)));
                if(active) {
                    hash11 = hash ( (hash11 + ((uint64_t)att20_cacctbal)));
                }
                hash11 = hash ( (hash11 + stringHash ( att19_cphone)));
                hash11 = hash ( (hash11 + stringHash ( att3_nname)));
                hash11 = hash ( (hash11 + stringHash ( att17_caddress)));
                hash11 = hash ( (hash11 + stringHash ( att22_ccomment)));
                apayl11 payl;
                payl.att15_ccustkey = att15_ccustkey;
                payl.att16_cname = att16_cname;
                payl.att20_cacctbal = att20_cacctbal;
                payl.att19_cphone = att19_cphone;
                payl.att3_nname = att3_nname;
                payl.att17_caddress = att17_caddress;
                payl.att22_ccomment = att22_ccomment;
                int bucketFound = 0;
                int numLookups = 0;
                while(!(bucketFound)) {
                    bucket = hashAggregateGetBucket ( aht11, 138748, hash11, numLookups, &(payl));
                    apayl11 probepayl = aht11[bucket].payload;
                    bucketFound = 1;
                    bucketFound &= ((payl.att15_ccustkey == probepayl.att15_ccustkey));
                    bucketFound &= (stringEquals ( payl.att16_cname, probepayl.att16_cname));
                    bucketFound &= ((payl.att20_cacctbal == probepayl.att20_cacctbal));
                    bucketFound &= (stringEquals ( payl.att19_cphone, probepayl.att19_cphone));
                    bucketFound &= (stringEquals ( payl.att3_nname, probepayl.att3_nname));
                    bucketFound &= (stringEquals ( payl.att17_caddress, probepayl.att17_caddress));
                    bucketFound &= (stringEquals ( payl.att22_ccomment, probepayl.att22_ccomment));
                }
            }
            if(active) {
                atomicAdd(&(agg1[bucket]), ((float)att39_rev));
            }
            active = 0;
            activemask100009_ = __ballot_sync(ALL_LANES,active);
            numactive100009_ = __popc(activemask100009_);
        }
        // flush to divergence buffer
        if((numactive100009_ > 0)) {
            // warp prefix scan of remaining active lanes
            scan100009_ = (__popc((activemask100009_ & prefixlanes)) + buffercount100009_);
            // write to buffer
            bufIdx100009_ = (bufferBase100009_ + scan100009_);
            if(active) {
                att3_nname_dvgnce_buf_100009_[bufIdx100009_] = att3_nname;
                att15_ccustkey_dvgnce_buf_100009_[bufIdx100009_] = att15_ccustkey;
                att16_cname_dvgnce_buf_100009_[bufIdx100009_] = att16_cname;
                att17_caddress_dvgnce_buf_100009_[bufIdx100009_] = att17_caddress;
                att19_cphone_dvgnce_buf_100009_[bufIdx100009_] = att19_cphone;
                att20_cacctbal_dvgnce_buf_100009_[bufIdx100009_] = att20_cacctbal;
                att22_ccomment_dvgnce_buf_100009_[bufIdx100009_] = att22_ccomment;
                att28_lextende_dvgnce_buf_100009_[bufIdx100009_] = att28_lextende;
                att29_ldiscoun_dvgnce_buf_100009_[bufIdx100009_] = att29_ldiscoun;
            }
            __syncwarp();
            buffercount100009_ += numactive100009_;
            active = 0;
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation11(
    agg_ht<apayl11>* aht11, float* agg1, int* nout_result, int* oatt15_ccustkey, str_offs* oatt16_cname_offset, char* iatt16_cname_char, float* oatt20_cacctbal, str_offs* oatt19_cphone_offset, char* iatt19_cphone_char, str_offs* oatt3_nname_offset, char* iatt3_nname_char, str_offs* oatt17_caddress_offset, char* iatt17_caddress_char, str_offs* oatt22_ccomment_offset, char* iatt22_ccomment_char, float* oatt1_revenue) {
    int att15_ccustkey;
    str_t att16_cname;
    float att20_cacctbal;
    str_t att19_cphone;
    str_t att3_nname;
    str_t att17_caddress;
    str_t att22_ccomment;
    float att1_revenue;
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_aggregation11 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation11 = loopVar;
        active = (loopVar < 138748);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 11) --------
        if(active) {
            active &= ((aht11[tid_aggregation11].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl11 payl = aht11[tid_aggregation11].payload;
            att15_ccustkey = payl.att15_ccustkey;
            att16_cname = payl.att16_cname;
            att20_cacctbal = payl.att20_cacctbal;
            att19_cphone = payl.att19_cphone;
            att3_nname = payl.att3_nname;
            att17_caddress = payl.att17_caddress;
            att22_ccomment = payl.att22_ccomment;
        }
        if(active) {
            att1_revenue = agg1[tid_aggregation11];
        }
        // -------- materialize (opId: 12) --------
        int wp;
        int writeMask;
        int numProj;
        writeMask = __ballot_sync(ALL_LANES,active);
        numProj = __popc(writeMask);
        if((warplane == 0)) {
            wp = atomicAdd(nout_result, numProj);
        }
        wp = __shfl_sync(ALL_LANES,wp,0);
        wp = (wp + __popc((writeMask & prefixlanes)));
        if(active) {
            oatt15_ccustkey[wp] = att15_ccustkey;
            oatt16_cname_offset[wp] = toStringOffset ( iatt16_cname_char, att16_cname);
            oatt20_cacctbal[wp] = att20_cacctbal;
            oatt19_cphone_offset[wp] = toStringOffset ( iatt19_cphone_char, att19_cphone);
            oatt3_nname_offset[wp] = toStringOffset ( iatt3_nname_char, att3_nname);
            oatt17_caddress_offset[wp] = toStringOffset ( iatt17_caddress_char, att17_caddress);
            oatt22_ccomment_offset[wp] = toStringOffset ( iatt22_ccomment_char, att22_ccomment);
            oatt1_revenue[wp] = att1_revenue;
        }
        loopVar += step;
    }

}

int main() {
    int* iatt2_nnationk;
    iatt2_nnationk = ( int*) map_memory_file ( "mmdb/nation_n_nationkey" );
    size_t* iatt3_nname_offset;
    iatt3_nname_offset = ( size_t*) map_memory_file ( "mmdb/nation_n_name_offset" );
    char* iatt3_nname_char;
    iatt3_nname_char = ( char*) map_memory_file ( "mmdb/nation_n_name_char" );
    int* iatt6_oorderke;
    iatt6_oorderke = ( int*) map_memory_file ( "mmdb/orders_o_orderkey" );
    int* iatt7_ocustkey;
    iatt7_ocustkey = ( int*) map_memory_file ( "mmdb/orders_o_custkey" );
    unsigned* iatt10_oorderda;
    iatt10_oorderda = ( unsigned*) map_memory_file ( "mmdb/orders_o_orderdate" );
    int* iatt15_ccustkey;
    iatt15_ccustkey = ( int*) map_memory_file ( "mmdb/customer_c_custkey" );
    size_t* iatt16_cname_offset;
    iatt16_cname_offset = ( size_t*) map_memory_file ( "mmdb/customer_c_name_offset" );
    char* iatt16_cname_char;
    iatt16_cname_char = ( char*) map_memory_file ( "mmdb/customer_c_name_char" );
    size_t* iatt17_caddress_offset;
    iatt17_caddress_offset = ( size_t*) map_memory_file ( "mmdb/customer_c_address_offset" );
    char* iatt17_caddress_char;
    iatt17_caddress_char = ( char*) map_memory_file ( "mmdb/customer_c_address_char" );
    int* iatt18_cnationk;
    iatt18_cnationk = ( int*) map_memory_file ( "mmdb/customer_c_nationkey" );
    size_t* iatt19_cphone_offset;
    iatt19_cphone_offset = ( size_t*) map_memory_file ( "mmdb/customer_c_phone_offset" );
    char* iatt19_cphone_char;
    iatt19_cphone_char = ( char*) map_memory_file ( "mmdb/customer_c_phone_char" );
    float* iatt20_cacctbal;
    iatt20_cacctbal = ( float*) map_memory_file ( "mmdb/customer_c_acctbal" );
    size_t* iatt22_ccomment_offset;
    iatt22_ccomment_offset = ( size_t*) map_memory_file ( "mmdb/customer_c_comment_offset" );
    char* iatt22_ccomment_char;
    iatt22_ccomment_char = ( char*) map_memory_file ( "mmdb/customer_c_comment_char" );
    int* iatt23_lorderke;
    iatt23_lorderke = ( int*) map_memory_file ( "mmdb/lineitem_l_orderkey" );
    float* iatt28_lextende;
    iatt28_lextende = ( float*) map_memory_file ( "mmdb/lineitem_l_extendedprice" );
    float* iatt29_ldiscoun;
    iatt29_ldiscoun = ( float*) map_memory_file ( "mmdb/lineitem_l_discount" );
    char* iatt31_lreturnf;
    iatt31_lreturnf = ( char*) map_memory_file ( "mmdb/lineitem_l_returnflag" );

    int nout_result;
    std::vector < int > oatt15_ccustkey(69374);
    std::vector < str_offs > oatt16_cname_offset(69374);
    std::vector < float > oatt20_cacctbal(69374);
    std::vector < str_offs > oatt19_cphone_offset(69374);
    std::vector < str_offs > oatt3_nname_offset(69374);
    std::vector < str_offs > oatt17_caddress_offset(69374);
    std::vector < str_offs > oatt22_ccomment_offset(69374);
    std::vector < float > oatt1_revenue(69374);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt2_nnationk;
    hipMalloc((void**) &d_iatt2_nnationk, 25* sizeof(int) );
    size_t* d_iatt3_nname_offset;
    hipMalloc((void**) &d_iatt3_nname_offset, (25 + 1)* sizeof(size_t) );
    char* d_iatt3_nname_char;
    hipMalloc((void**) &d_iatt3_nname_char, 186* sizeof(char) );
    int* d_iatt6_oorderke;
    hipMalloc((void**) &d_iatt6_oorderke, 1500000* sizeof(int) );
    int* d_iatt7_ocustkey;
    hipMalloc((void**) &d_iatt7_ocustkey, 1500000* sizeof(int) );
    unsigned* d_iatt10_oorderda;
    hipMalloc((void**) &d_iatt10_oorderda, 1500000* sizeof(unsigned) );
    int* d_iatt15_ccustkey;
    hipMalloc((void**) &d_iatt15_ccustkey, 150000* sizeof(int) );
    size_t* d_iatt16_cname_offset;
    hipMalloc((void**) &d_iatt16_cname_offset, (150000 + 1)* sizeof(size_t) );
    char* d_iatt16_cname_char;
    hipMalloc((void**) &d_iatt16_cname_char, 2700009* sizeof(char) );
    size_t* d_iatt17_caddress_offset;
    hipMalloc((void**) &d_iatt17_caddress_offset, (150000 + 1)* sizeof(size_t) );
    char* d_iatt17_caddress_char;
    hipMalloc((void**) &d_iatt17_caddress_char, 3753296* sizeof(char) );
    int* d_iatt18_cnationk;
    hipMalloc((void**) &d_iatt18_cnationk, 150000* sizeof(int) );
    size_t* d_iatt19_cphone_offset;
    hipMalloc((void**) &d_iatt19_cphone_offset, (150000 + 1)* sizeof(size_t) );
    char* d_iatt19_cphone_char;
    hipMalloc((void**) &d_iatt19_cphone_char, 2250009* sizeof(char) );
    float* d_iatt20_cacctbal;
    hipMalloc((void**) &d_iatt20_cacctbal, 150000* sizeof(float) );
    size_t* d_iatt22_ccomment_offset;
    hipMalloc((void**) &d_iatt22_ccomment_offset, (150000 + 1)* sizeof(size_t) );
    char* d_iatt22_ccomment_char;
    hipMalloc((void**) &d_iatt22_ccomment_char, 10836339* sizeof(char) );
    int* d_iatt23_lorderke;
    hipMalloc((void**) &d_iatt23_lorderke, 6001215* sizeof(int) );
    float* d_iatt28_lextende;
    hipMalloc((void**) &d_iatt28_lextende, 6001215* sizeof(float) );
    float* d_iatt29_ldiscoun;
    hipMalloc((void**) &d_iatt29_ldiscoun, 6001215* sizeof(float) );
    char* d_iatt31_lreturnf;
    hipMalloc((void**) &d_iatt31_lreturnf, 6001215* sizeof(char) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    int* d_oatt15_ccustkey;
    hipMalloc((void**) &d_oatt15_ccustkey, 69374* sizeof(int) );
    str_offs* d_oatt16_cname_offset;
    hipMalloc((void**) &d_oatt16_cname_offset, 69374* sizeof(str_offs) );
    float* d_oatt20_cacctbal;
    hipMalloc((void**) &d_oatt20_cacctbal, 69374* sizeof(float) );
    str_offs* d_oatt19_cphone_offset;
    hipMalloc((void**) &d_oatt19_cphone_offset, 69374* sizeof(str_offs) );
    str_offs* d_oatt3_nname_offset;
    hipMalloc((void**) &d_oatt3_nname_offset, 69374* sizeof(str_offs) );
    str_offs* d_oatt17_caddress_offset;
    hipMalloc((void**) &d_oatt17_caddress_offset, 69374* sizeof(str_offs) );
    str_offs* d_oatt22_ccomment_offset;
    hipMalloc((void**) &d_oatt22_ccomment_offset, 69374* sizeof(str_offs) );
    float* d_oatt1_revenue;
    hipMalloc((void**) &d_oatt1_revenue, 69374* sizeof(float) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    unique_ht<jpayl6>* d_jht6;
    hipMalloc((void**) &d_jht6, 50* sizeof(unique_ht<jpayl6>) );
    {
        int gridsize=100;
        int blocksize=32;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht6, 50);
    }
    multi_ht* d_jht5;
    hipMalloc((void**) &d_jht5, 150000* sizeof(multi_ht) );
    jpayl5* d_jht5_payload;
    hipMalloc((void**) &d_jht5_payload, 150000* sizeof(jpayl5) );
    {
        int gridsize=100;
        int blocksize=32;
        initMultiHT<<<gridsize, blocksize>>>(d_jht5, 150000);
    }
    int* d_offs5;
    hipMalloc((void**) &d_offs5, 1* sizeof(int) );
    {
        int gridsize=100;
        int blocksize=32;
        initArray<<<gridsize, blocksize>>>(d_offs5, 0, 1);
    }
    unique_ht<jpayl9>* d_jht9;
    hipMalloc((void**) &d_jht9, 300000* sizeof(unique_ht<jpayl9>) );
    {
        int gridsize=100;
        int blocksize=32;
        initUniqueHT<<<gridsize, blocksize>>>(d_jht9, 300000);
    }
    agg_ht<apayl11>* d_aht11;
    hipMalloc((void**) &d_aht11, 138748* sizeof(agg_ht<apayl11>) );
    {
        int gridsize=100;
        int blocksize=32;
        initAggHT<<<gridsize, blocksize>>>(d_aht11, 138748);
    }
    float* d_agg1;
    hipMalloc((void**) &d_agg1, 138748* sizeof(float) );
    {
        int gridsize=100;
        int blocksize=32;
        initArray<<<gridsize, blocksize>>>(d_agg1, 0.0f, 138748);
    }
    {
        int gridsize=100;
        int blocksize=32;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt2_nnationk, iatt2_nnationk, 25 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt3_nname_offset, iatt3_nname_offset, (25 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt3_nname_char, iatt3_nname_char, 186 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt6_oorderke, iatt6_oorderke, 1500000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt7_ocustkey, iatt7_ocustkey, 1500000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt10_oorderda, iatt10_oorderda, 1500000 * sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt15_ccustkey, iatt15_ccustkey, 150000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt16_cname_offset, iatt16_cname_offset, (150000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt16_cname_char, iatt16_cname_char, 2700009 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt17_caddress_offset, iatt17_caddress_offset, (150000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt17_caddress_char, iatt17_caddress_char, 3753296 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt18_cnationk, iatt18_cnationk, 150000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt19_cphone_offset, iatt19_cphone_offset, (150000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt19_cphone_char, iatt19_cphone_char, 2250009 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt20_cacctbal, iatt20_cacctbal, 150000 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt22_ccomment_offset, iatt22_ccomment_offset, (150000 + 1) * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt22_ccomment_char, iatt22_ccomment_char, 10836339 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt23_lorderke, iatt23_lorderke, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt28_lextende, iatt28_lextende, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt29_ldiscoun, iatt29_ldiscoun, 6001215 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt31_lreturnf, iatt31_lreturnf, 6001215 * sizeof(char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime0 = std::clock();
    std::clock_t start_krnl_nation11 = std::clock();
    {
        int gridsize=100;
        int blocksize=32;
        krnl_nation1<<<gridsize, blocksize>>>(d_iatt2_nnationk, d_iatt3_nname_offset, d_iatt3_nname_char, d_jht6);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_nation11 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_nation1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_nation1")
        }
    }

    std::clock_t start_krnl_orders22 = std::clock();
    {
        int gridsize=100;
        int blocksize=32;
        krnl_orders2<<<gridsize, blocksize>>>(d_iatt6_oorderke, d_iatt7_ocustkey, d_iatt10_oorderda, d_jht5, d_jht5_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_orders22 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_orders2! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_orders2")
        }
    }

    std::clock_t start_scanMultiHT3 = std::clock();
    {
        int gridsize=100;
        int blocksize=32;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht5, 150000, d_offs5);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT3 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT")
        }
    }

    std::clock_t start_krnl_orders2_ins4 = std::clock();
    {
        int gridsize=100;
        int blocksize=32;
        krnl_orders2_ins<<<gridsize, blocksize>>>(d_iatt6_oorderke, d_iatt7_ocustkey, d_iatt10_oorderda, d_jht5, d_jht5_payload, d_offs5);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_orders2_ins4 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_orders2_ins! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_orders2_ins")
        }
    }

    std::clock_t start_krnl_customer45 = std::clock();
    {
        int gridsize=100;
        int blocksize=32;
        krnl_customer4<<<gridsize, blocksize>>>(d_iatt15_ccustkey, d_iatt16_cname_offset, d_iatt16_cname_char, d_iatt17_caddress_offset, d_iatt17_caddress_char, d_iatt18_cnationk, d_iatt19_cphone_offset, d_iatt19_cphone_char, d_iatt20_cacctbal, d_iatt22_ccomment_offset, d_iatt22_ccomment_char, d_jht5, d_jht5_payload, d_jht6, d_jht9);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_customer45 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_customer4! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_customer4")
        }
    }

    std::clock_t start_krnl_lineitem76 = std::clock();
    {
        int gridsize=100;
        int blocksize=32;
        krnl_lineitem7<<<gridsize, blocksize>>>(d_iatt23_lorderke, d_iatt28_lextende, d_iatt29_ldiscoun, d_iatt31_lreturnf, d_jht9, d_aht11, d_agg1);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_lineitem76 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem7! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem7")
        }
    }

    std::clock_t start_krnl_aggregation117 = std::clock();
    {
        int gridsize=100;
        int blocksize=32;
        krnl_aggregation11<<<gridsize, blocksize>>>(d_aht11, d_agg1, d_nout_result, d_oatt15_ccustkey, d_oatt16_cname_offset, d_iatt16_cname_char, d_oatt20_cacctbal, d_oatt19_cphone_offset, d_iatt19_cphone_char, d_oatt3_nname_offset, d_iatt3_nname_char, d_oatt17_caddress_offset, d_iatt17_caddress_char, d_oatt22_ccomment_offset, d_iatt22_ccomment_char, d_oatt1_revenue);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation117 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation11! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation11")
        }
    }

    std::clock_t stop_totalKernelTime0 = std::clock();
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt15_ccustkey.data(), d_oatt15_ccustkey, 69374 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt16_cname_offset.data(), d_oatt16_cname_offset, 69374 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt20_cacctbal.data(), d_oatt20_cacctbal, 69374 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( oatt19_cphone_offset.data(), d_oatt19_cphone_offset, 69374 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt3_nname_offset.data(), d_oatt3_nname_offset, 69374 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt17_caddress_offset.data(), d_oatt17_caddress_offset, 69374 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt22_ccomment_offset.data(), d_oatt22_ccomment_offset, 69374 * sizeof(str_offs), hipMemcpyDeviceToHost);
    hipMemcpy( oatt1_revenue.data(), d_oatt1_revenue, 69374 * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt2_nnationk);
    hipFree( d_iatt3_nname_offset);
    hipFree( d_iatt3_nname_char);
    hipFree( d_jht6);
    hipFree( d_iatt6_oorderke);
    hipFree( d_iatt7_ocustkey);
    hipFree( d_iatt10_oorderda);
    hipFree( d_jht5);
    hipFree( d_jht5_payload);
    hipFree( d_offs5);
    hipFree( d_iatt15_ccustkey);
    hipFree( d_iatt16_cname_offset);
    hipFree( d_iatt16_cname_char);
    hipFree( d_iatt17_caddress_offset);
    hipFree( d_iatt17_caddress_char);
    hipFree( d_iatt18_cnationk);
    hipFree( d_iatt19_cphone_offset);
    hipFree( d_iatt19_cphone_char);
    hipFree( d_iatt20_cacctbal);
    hipFree( d_iatt22_ccomment_offset);
    hipFree( d_iatt22_ccomment_char);
    hipFree( d_jht9);
    hipFree( d_iatt23_lorderke);
    hipFree( d_iatt28_lextende);
    hipFree( d_iatt29_ldiscoun);
    hipFree( d_iatt31_lreturnf);
    hipFree( d_aht11);
    hipFree( d_agg1);
    hipFree( d_nout_result);
    hipFree( d_oatt15_ccustkey);
    hipFree( d_oatt16_cname_offset);
    hipFree( d_oatt20_cacctbal);
    hipFree( d_oatt19_cphone_offset);
    hipFree( d_oatt3_nname_offset);
    hipFree( d_oatt17_caddress_offset);
    hipFree( d_oatt22_ccomment_offset);
    hipFree( d_oatt1_revenue);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish8 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 69374)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("c_custkey: ");
        printf("%8i", oatt15_ccustkey[pv]);
        printf("  ");
        printf("c_name: ");
        stringPrint ( iatt16_cname_char, oatt16_cname_offset[pv]);
        printf("  ");
        printf("c_acctbal: ");
        printf("%15.2f", oatt20_cacctbal[pv]);
        printf("  ");
        printf("c_phone: ");
        stringPrint ( iatt19_cphone_char, oatt19_cphone_offset[pv]);
        printf("  ");
        printf("n_name: ");
        stringPrint ( iatt3_nname_char, oatt3_nname_offset[pv]);
        printf("  ");
        printf("c_address: ");
        stringPrint ( iatt17_caddress_char, oatt17_caddress_offset[pv]);
        printf("  ");
        printf("c_comment: ");
        stringPrint ( iatt22_ccomment_char, oatt22_ccomment_offset[pv]);
        printf("  ");
        printf("revenue: ");
        printf("%15.2f", oatt1_revenue[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish8 = std::clock();

    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "krnl_nation1", (stop_krnl_nation11 - start_krnl_nation11) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_orders2", (stop_krnl_orders22 - start_krnl_orders22) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "scanMultiHT", (stop_scanMultiHT3 - start_scanMultiHT3) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_orders2_ins", (stop_krnl_orders2_ins4 - start_krnl_orders2_ins4) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_customer4", (stop_krnl_customer45 - start_krnl_customer45) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_lineitem7 LR9", (stop_krnl_lineitem76 - start_krnl_lineitem76) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation11", (stop_krnl_aggregation117 - start_krnl_aggregation117) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish8 - start_finish8) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime0 - start_totalKernelTime0) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
