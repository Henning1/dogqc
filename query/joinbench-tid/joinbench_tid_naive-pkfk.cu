#include "hip/hip_runtime.h"
#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct jpayl3 {
    int tid;
};

__global__ void krnl_r_build1(
    int* iatt1_rbuild, int* iatt2_rlinenum, multi_ht* jht3, jpayl3* jht3_payload) {
    int att1_rbuild;
    int att2_rlinenum;

    int tid_r_build1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    // -------- scan --------
    while(!(flushPipeline)) {
        tid_r_build1 = loopVar;
      //active = (tid_r_build1 < 10000000);
        active = (tid_r_build1 < 100000000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att1_rbuild = iatt1_rbuild[tid_r_build1];
            att2_rlinenum = iatt2_rlinenum[tid_r_build1];
        }
        // -------- hash join build --------
        if(active) {
            uint64_t hash3 = 0;
            if(active) {
                hash3 = 0;
                hash3 = hash ( (hash3 + ((uint64_t)att1_rbuild)));
            }
          //hashCountMulti ( jht3, 20000000, hash3);
            hashCountMulti ( jht3, 200000000, hash3);
        }
        loopVar += step;
    }

}

__global__ void krnl_r_build1_ins(
    int* iatt1_rbuild, int* iatt2_rlinenum, multi_ht* jht3, jpayl3* jht3_payload, int* offs3) {
    int att1_rbuild;
    int att2_rlinenum;

    int tid_r_build1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    // -------- scan --------
    while(!(flushPipeline)) {
        tid_r_build1 = loopVar;
      //active = (tid_r_build1 < 10000000);
        active = (tid_r_build1 < 100000000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        // -------- hash join build --------
        if(active) {
            uint64_t hash3 = 0;
            if(active) {
                hash3 = 0;
                hash3 = hash ( (hash3 + ((uint64_t)iatt1_rbuild[tid_r_build1])));
            }
            jpayl3 payl;
            payl.tid = tid_r_build1;
          //hashInsertMulti ( jht3, jht3_payload, offs3, 20000000, hash3, &(payl));
            hashInsertMulti ( jht3, jht3_payload, offs3, 200000000, hash3, &(payl));
        }
        loopVar += step;
    }

}

__global__ void krnl_s_probe2(
    int* iatt3_sprobe, int* iatt4_slinenum, multi_ht* jht3, jpayl3* jht3_payload, int* nout_result, int* oatt1_rbuild, int* oatt2_rlinenum, int* oatt3_sprobe, int* oatt4_slinenum, int* iatt1_rbuild, int* iatt2_rlinenum) {
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_s_probe1 = 0;
    int tid_r_build1;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    // -------- scan --------
    while(!(flushPipeline)) {
        tid_s_probe1 = loopVar;
      //active = (tid_s_probe1 < 312500);
      //active = (tid_s_probe1 < 3125000);
        active = (tid_s_probe1 < 100000000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        // -------- hash join probe --------
        int matchEnd3 = 0;
        int matchOffset3 = 0;
        int matchStep3 = 1;
        int matchFound3 = 0;
        int probeActive3 = active;
        uint64_t hash3 = 0;
        if(probeActive3) {
            hash3 = 0;
            hash3 = hash ( (hash3 + ((uint64_t)iatt3_sprobe[tid_s_probe1])));
          //probeActive3 = hashProbeMulti ( jht3, 20000000, hash3, matchOffset3, matchEnd3);
            probeActive3 = hashProbeMulti ( jht3, 200000000, hash3, matchOffset3, matchEnd3);
        }
        active = probeActive3;
        while(__any_sync(ALL_LANES,active)) {
            jpayl3 payl;
            if(probeActive3) {
                payl = jht3_payload[matchOffset3];
                tid_r_build1 = payl.tid;
                active &= ((iatt1_rbuild[tid_r_build1] == iatt3_sprobe[tid_s_probe1]));
                matchFound3 += active;
            }
            // -------- materialize --------
            int wp;
            int writeMask;
            int numProj;
            writeMask = __ballot_sync(ALL_LANES,active);
            numProj = __popc(writeMask);
            if((warplane == 0)) {
                wp = atomicAdd(nout_result, numProj);
            }
            wp = __shfl_sync(ALL_LANES,wp,0);
            wp = (wp + __popc((writeMask & prefixlanes)));
            if(active) {
                oatt1_rbuild[wp] = iatt1_rbuild[tid_r_build1];
                oatt2_rlinenum[wp] = iatt2_rlinenum[tid_r_build1];
                oatt3_sprobe[wp] = iatt3_sprobe[tid_s_probe1];
                oatt4_slinenum[wp] = iatt4_slinenum[tid_s_probe1];
            }
            matchOffset3 += matchStep3;
            probeActive3 &= ((matchOffset3 < matchEnd3));
            active = probeActive3;
        }
        loopVar += step;
    }

}

int main() {
    std::clock_t start_import50 = std::clock();
    int* iatt1_rbuild;
    iatt1_rbuild = ( int*) map_memory_file ( "mmdb/r_build_r_build" );
    int* iatt2_rlinenum;
    iatt2_rlinenum = ( int*) map_memory_file ( "mmdb/r_build_r_linenumber" );
    int* iatt3_sprobe;
    iatt3_sprobe = ( int*) map_memory_file ( "mmdb/s_probe_s_probe" );
    int* iatt4_slinenum;
    iatt4_slinenum = ( int*) map_memory_file ( "mmdb/s_probe_s_linenumber" );
    std::clock_t stop_import50 = std::clock();

    std::clock_t start_declare51 = std::clock();
    int nout_result;
  //std::vector < int > oatt1_rbuild(10000000);
  //std::vector < int > oatt2_rlinenum(10000000);
  //std::vector < int > oatt3_sprobe(10000000);
  //std::vector < int > oatt4_slinenum(10000000);
    std::vector < int > oatt1_rbuild(100000000);
    std::vector < int > oatt2_rlinenum(100000000);
    std::vector < int > oatt3_sprobe(100000000);
    std::vector < int > oatt4_slinenum(100000000);
    std::clock_t stop_declare51 = std::clock();

    std::clock_t start_wake_up_gpu52 = std::clock();
    // wake up gpu
    hipDeviceSynchronize();
    std::clock_t stop_wake_up_gpu52 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    std::clock_t start_cuda_malloc53 = std::clock();
    int* d_iatt1_rbuild;
  //hipMalloc((void**) &d_iatt1_rbuild, 10000000* sizeof(int) );
    hipMalloc((void**) &d_iatt1_rbuild, 100000000* sizeof(int) );
    int* d_iatt2_rlinenum;
  //hipMalloc((void**) &d_iatt2_rlinenum, 10000000* sizeof(int) );
    hipMalloc((void**) &d_iatt2_rlinenum, 100000000* sizeof(int) );
  //fprintf(stderr, "ht jht3, type multi_ht: %lu MB\n", ((sizeof(multi_ht) * 20000000) / 1000000));
    fprintf(stderr, "ht jht3, type multi_ht: %lu MB\n", ((sizeof(multi_ht) * 200000000) / 10000000));
  //fprintf(stderr, "ht jht3_payload, type jpayl3: %lu MB\n", ((sizeof(jpayl3) * 20000000) / 1000000));
    fprintf(stderr, "ht jht3_payload, type jpayl3: %lu MB\n", ((sizeof(jpayl3) * 200000000) / 10000000));
    int* d_iatt3_sprobe;
  //hipMalloc((void**) &d_iatt3_sprobe, 10000000* sizeof(int) );
    hipMalloc((void**) &d_iatt3_sprobe, 100000000* sizeof(int) );
    int* d_iatt4_slinenum;
  //hipMalloc((void**) &d_iatt4_slinenum, 10000000* sizeof(int) );
    hipMalloc((void**) &d_iatt4_slinenum, 100000000* sizeof(int) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    int* d_oatt1_rbuild;
  //hipMalloc((void**) &d_oatt1_rbuild, 10000000* sizeof(int) );
    hipMalloc((void**) &d_oatt1_rbuild, 100000000* sizeof(int) );
    int* d_oatt2_rlinenum;
  //hipMalloc((void**) &d_oatt2_rlinenum, 10000000* sizeof(int) );
    hipMalloc((void**) &d_oatt2_rlinenum, 100000000* sizeof(int) );
    int* d_oatt3_sprobe;
  //hipMalloc((void**) &d_oatt3_sprobe, 10000000* sizeof(int) );
    hipMalloc((void**) &d_oatt3_sprobe, 100000000* sizeof(int) );
    int* d_oatt4_slinenum;
  //hipMalloc((void**) &d_oatt4_slinenum, 10000000* sizeof(int) );
    hipMalloc((void**) &d_oatt4_slinenum, 100000000* sizeof(int) );
    hipDeviceSynchronize();
    std::clock_t stop_cuda_malloc53 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    std::clock_t start_cuda_mallocHT54 = std::clock();
    multi_ht* d_jht3;
  //hipMalloc((void**) &d_jht3, 20000000* sizeof(multi_ht) );
    hipMalloc((void**) &d_jht3, 200000000* sizeof(multi_ht) );
    jpayl3* d_jht3_payload;
  //hipMalloc((void**) &d_jht3_payload, 20000000* sizeof(jpayl3) );
    hipMalloc((void**) &d_jht3_payload, 200000000* sizeof(jpayl3) );
    {
        int gridsize=920;
        int blocksize=128;
      //initMultiHT<<<gridsize, blocksize>>>(d_jht3, 20000000);
        initMultiHT<<<gridsize, blocksize>>>(d_jht3, 200000000);
    }
    int* d_offs3;
    hipMalloc((void**) &d_offs3, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs3, 0, 1);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    std::clock_t stop_cuda_mallocHT54 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    std::clock_t start_cuda_memcpy_in55 = std::clock();
  //hipMemcpy( d_iatt1_rbuild, iatt1_rbuild, 10000000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt1_rbuild, iatt1_rbuild, 100000000 * sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy( d_iatt2_rlinenum, iatt2_rlinenum, 10000000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt2_rlinenum, iatt2_rlinenum, 100000000 * sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy( d_iatt3_sprobe, iatt3_sprobe, 312500 * sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy( d_iatt3_sprobe, iatt3_sprobe, 3125000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt3_sprobe, iatt3_sprobe, 100000000 * sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy( d_iatt4_slinenum, iatt4_slinenum, 312500 * sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy( d_iatt4_slinenum, iatt4_slinenum, 3125000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt4_slinenum, iatt4_slinenum, 100000000 * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    std::clock_t stop_cuda_memcpy_in55 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime56 = std::clock();
    std::clock_t start_krnl_r_build1_920_12857 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_r_build1<<<gridsize, blocksize>>>(d_iatt1_rbuild, d_iatt2_rlinenum, d_jht3, d_jht3_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_r_build1_920_12857 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_r_build1 920 128! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_r_build1 920 128")
        }
    }

    std::clock_t start_scanMultiHT_920_12858 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
      //scanMultiHT<<<gridsize, blocksize>>>(d_jht3, 20000000, d_offs3);
        scanMultiHT<<<gridsize, blocksize>>>(d_jht3, 200000000, d_offs3);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT_920_12858 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT 920 128! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT 920 128")
        }
    }

    std::clock_t start_krnl_r_build1_ins_920_12859 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_r_build1_ins<<<gridsize, blocksize>>>(d_iatt1_rbuild, d_iatt2_rlinenum, d_jht3, d_jht3_payload, d_offs3);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_r_build1_ins_920_12859 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_r_build1_ins 920 128! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_r_build1_ins 920 128")
        }
    }

    std::clock_t start_krnl_s_probe2_920_12860 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_s_probe2<<<gridsize, blocksize>>>(d_iatt3_sprobe, d_iatt4_slinenum, d_jht3, d_jht3_payload, d_nout_result, d_oatt1_rbuild, d_oatt2_rlinenum, d_oatt3_sprobe, d_oatt4_slinenum,d_iatt1_rbuild, d_iatt2_rlinenum);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_s_probe2_920_12860 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_s_probe2 920 128! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_s_probe2 920 128")
        }
    }

    std::clock_t stop_totalKernelTime56 = std::clock();
    std::clock_t start_cuda_memcpy_out61 = std::clock();
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
  //hipMemcpy( oatt1_rbuild.data(), d_oatt1_rbuild, 10000000 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt1_rbuild.data(), d_oatt1_rbuild, 100000000 * sizeof(int), hipMemcpyDeviceToHost);
  //hipMemcpy( oatt2_rlinenum.data(), d_oatt2_rlinenum, 10000000 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt2_rlinenum.data(), d_oatt2_rlinenum, 100000000 * sizeof(int), hipMemcpyDeviceToHost);
  //hipMemcpy( oatt3_sprobe.data(), d_oatt3_sprobe, 10000000 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt3_sprobe.data(), d_oatt3_sprobe, 100000000 * sizeof(int), hipMemcpyDeviceToHost);
  //hipMemcpy( oatt4_slinenum.data(), d_oatt4_slinenum, 10000000 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt4_slinenum.data(), d_oatt4_slinenum, 100000000 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    std::clock_t stop_cuda_memcpy_out61 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    std::clock_t start_cuda_free62 = std::clock();
    hipFree( d_iatt1_rbuild);
    hipFree( d_iatt2_rlinenum);
    hipFree( d_jht3);
    hipFree( d_jht3_payload);
    hipFree( d_offs3);
    hipFree( d_iatt3_sprobe);
    hipFree( d_iatt4_slinenum);
    hipFree( d_nout_result);
    hipFree( d_oatt1_rbuild);
    hipFree( d_oatt2_rlinenum);
    hipFree( d_oatt3_sprobe);
    hipFree( d_oatt4_slinenum);
    hipDeviceSynchronize();
    std::clock_t stop_cuda_free62 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish63 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
  //if((nout_result > 10000000)) {
    if((nout_result > 100000000)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("r_build: ");
        printf("%8i", oatt1_rbuild[pv]);
        printf("  ");
        printf("r_linenumber: ");
        printf("%8i", oatt2_rlinenum[pv]);
        printf("  ");
        printf("s_probe: ");
        printf("%8i", oatt3_sprobe[pv]);
        printf("  ");
        printf("s_linenumber: ");
        printf("%8i", oatt4_slinenum[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    //FILE* outFile;
    //outFile = fopen("queryresult.csv", "w");
    //fprintf(outFile, "r_build, ");
    //fprintf(outFile, "r_linenumber, ");
    //fprintf(outFile, "s_probe, ");
    //fprintf(outFile, "s_linenumber, ");
    //fprintf(outFile, "\n");
    //for ( int pv = 0; (pv < nout_result); pv += 1) {
    //    fprintf(outFile, "%8i  ", oatt1_rbuild[pv]);
    //    fprintf(outFile, "%8i  ", oatt2_rlinenum[pv]);
    //    fprintf(outFile, "%8i  ", oatt3_sprobe[pv]);
    //    fprintf(outFile, "%8i  ", oatt4_slinenum[pv]);
    //    fprintf(outFile, "\n");
    //}
    std::clock_t stop_finish63 = std::clock();

    std::cout << "import: " << (stop_import50 - start_import50) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "declare: " << (stop_declare51 - start_declare51) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "wake up gpu: " << (stop_wake_up_gpu52 - start_wake_up_gpu52) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "cuda malloc: " << (stop_cuda_malloc53 - start_cuda_malloc53) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "cuda mallocHT: " << (stop_cuda_mallocHT54 - start_cuda_mallocHT54) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "cuda memcpy in: " << (stop_cuda_memcpy_in55 - start_cuda_memcpy_in55) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "krnl_r_build1 920 128: " << (stop_krnl_r_build1_920_12857 - start_krnl_r_build1_920_12857) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "scanMultiHT 920 128: " << (stop_scanMultiHT_920_12858 - start_scanMultiHT_920_12858) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "krnl_r_build1_ins 920 128: " << (stop_krnl_r_build1_ins_920_12859 - start_krnl_r_build1_ins_920_12859) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "krnl_s_probe2 920 128: " << (stop_krnl_s_probe2_920_12860 - start_krnl_s_probe2_920_12860) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "cuda memcpy out: " << (stop_cuda_memcpy_out61 - start_cuda_memcpy_out61) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "cuda free: " << (stop_cuda_free62 - start_cuda_free62) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "finish: " << (stop_finish63 - start_finish63) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "totalKernelTime: " << (stop_totalKernelTime56 - start_totalKernelTime56) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
}
