#include "hip/hip_runtime.h"
#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct jpayl3 {
    int tid;
};

__global__ void krnl_r_build1(
    int* iatt1_rbuild, int* iatt2_rlinenum, multi_ht* jht3, jpayl3* jht3_payload) {
    int att1_rbuild;
    int att2_rlinenum;

    int tid_r_build1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    // -------- scan --------
    while(!(flushPipeline)) {
        tid_r_build1 = loopVar;
        active = (tid_r_build1 < 100000000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att1_rbuild = iatt1_rbuild[tid_r_build1];
            att2_rlinenum = iatt2_rlinenum[tid_r_build1];
        }
        // -------- hash join build --------
        if(active) {
            uint64_t hash3 = 0;
            if(active) {
                hash3 = 0;
                hash3 = hash ( (hash3 + ((uint64_t)att1_rbuild)));
            }
            hashCountMulti ( jht3, 200000000, hash3);
        }
        loopVar += step;
    }

}

__global__ void krnl_r_build1_ins(
    int* iatt1_rbuild, int* iatt2_rlinenum, multi_ht* jht3, jpayl3* jht3_payload, int* offs3) {
    int att1_rbuild;
    int att2_rlinenum;

    int tid_r_build1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    // -------- scan --------
    while(!(flushPipeline)) {
        tid_r_build1 = loopVar;
        active = (tid_r_build1 < 100000000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        // -------- hash join build --------
        if(active) {
            uint64_t hash3 = 0;
            if(active) {
                hash3 = 0;
                hash3 = hash ( (hash3 + ((uint64_t)iatt1_rbuild[tid_r_build1])));
            }
            jpayl3 payl;
            payl.tid = tid_r_build1;
            hashInsertMulti ( jht3, jht3_payload, offs3, 200000000, hash3, &(payl));
        }
        loopVar += step;
    }

}

__global__ void krnl_s_probe2(
    int* iatt3_sprobe, int* iatt4_slinenum, multi_ht* jht3, jpayl3* jht3_payload, int* nout_result, int* oatt1_rbuild, int* oatt2_rlinenum, int* oatt3_sprobe, int* oatt4_slinenum, int* iatt1_rbuild, int* iatt2_rlinenum) {
    int att3_sprobe;
    int att4_slinenum;
    unsigned warplane = (threadIdx.x % 32);
    int att1_rbuild;
    int att2_rlinenum;
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_s_probe1 = 0;
    int tid_r_build1;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    // -------- scan --------
    while(!(flushPipeline)) {
        tid_s_probe1 = loopVar;
      //active = (tid_s_probe1 < 3125000);
      //active = (tid_s_probe1 < 100000000);
        active = (tid_s_probe1 < 12500000);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        // -------- hash join probe --------
        // -------- multiprobe multi broadcast --------
        int matchEnd3 = 0;
        int matchEndBuf3 = 0;
        int matchOffset3 = 0;
        int matchOffsetBuf3 = 0;
        int probeActive3 = active;

        //int att3_sprobe_bcbuf3;
        //int att4_slinenum_bcbuf3;
        int tid_s_probe1_bcbuf;

        uint64_t hash3 = 0;
        if(probeActive3) {
            hash3 = 0;
            hash3 = hash ( (hash3 + ((uint64_t)iatt3_sprobe[tid_s_probe1])));
            probeActive3 = hashProbeMulti ( jht3, 200000000, hash3, matchOffsetBuf3, matchEndBuf3);
        }
        unsigned activeProbes3 = __ballot_sync(ALL_LANES,probeActive3);
        int num3 = 0;
        num3 = (matchEndBuf3 - matchOffsetBuf3);
        unsigned wideProbes3 = __ballot_sync(ALL_LANES,(num3 >= 32));

        //att3_sprobe_bcbuf3 = att3_sprobe;
        //att4_slinenum_bcbuf3 = att4_slinenum;
        tid_s_probe1_bcbuf = tid_s_probe1;

        while((activeProbes3 > 0)) {
            unsigned tupleLane;
            unsigned broadcastLane;
            int numFilled = 0;
            int num = 0;
            while(((numFilled < 32) && activeProbes3)) {
                if((wideProbes3 > 0)) {
                    tupleLane = (__ffs(wideProbes3) - 1);
                    wideProbes3 -= (1 << tupleLane);
                }
                else {
                    tupleLane = (__ffs(activeProbes3) - 1);
                }
                num = __shfl_sync(ALL_LANES,num3,tupleLane);
                if((numFilled && ((numFilled + num) > 32))) {
                    break;
                }
                if((warplane >= numFilled)) {
                    broadcastLane = tupleLane;
                    matchOffset3 = (warplane - numFilled);
                }
                numFilled += num;
                activeProbes3 -= (1 << tupleLane);
            }
            matchOffset3 += __shfl_sync(ALL_LANES,matchOffsetBuf3,broadcastLane);
            matchEnd3 = __shfl_sync(ALL_LANES,matchEndBuf3,broadcastLane);

            //att3_sprobe = __shfl_sync(ALL_LANES,att3_sprobe_bcbuf3,broadcastLane);
            //att4_slinenum = __shfl_sync(ALL_LANES,att4_slinenum_bcbuf3,broadcastLane);
            tid_s_probe1 = __shfl_sync(ALL_LANES,tid_s_probe1_bcbuf,broadcastLane);

            probeActive3 = (matchOffset3 < matchEnd3);
            while(__any_sync(ALL_LANES,probeActive3)) {
                active = 0;
                jpayl3 payl;
                if(probeActive3) {
                    payl = jht3_payload[matchOffset3];
                    tid_r_build1 = payl.tid;
                    active = 1;
                    active &= ((iatt1_rbuild[tid_r_build1] == iatt3_sprobe[tid_s_probe1]));
                    matchOffset3 += 32;
                    probeActive3 &= ((matchOffset3 < matchEnd3));
                }
                // -------- materialize --------
                int wp;
                int writeMask;
                int numProj;
                writeMask = __ballot_sync(ALL_LANES,active);
                numProj = __popc(writeMask);
                if((warplane == 0)) {
                    wp = atomicAdd(nout_result, numProj);
                }
                wp = __shfl_sync(ALL_LANES,wp,0);
                wp = (wp + __popc((writeMask & prefixlanes)));
                if(active) {
                    oatt1_rbuild[wp] = iatt1_rbuild[tid_r_build1];
                    oatt2_rlinenum[wp] = iatt2_rlinenum[tid_r_build1];
                    oatt3_sprobe[wp] = iatt3_sprobe[tid_s_probe1];
                    oatt4_slinenum[wp] = iatt4_slinenum[tid_s_probe1];
                }
            }
        }
        loopVar += step;
    }

}

int main() {
    std::clock_t start_import2 = std::clock();
    int* iatt1_rbuild;
    iatt1_rbuild = ( int*) map_memory_file ( "mmdb/r_build_r_build" );
    int* iatt2_rlinenum;
    iatt2_rlinenum = ( int*) map_memory_file ( "mmdb/r_build_r_linenumber" );
    int* iatt3_sprobe;
    iatt3_sprobe = ( int*) map_memory_file ( "mmdb/s_probe_s_probe" );
    int* iatt4_slinenum;
    iatt4_slinenum = ( int*) map_memory_file ( "mmdb/s_probe_s_linenumber" );
    std::clock_t stop_import2 = std::clock();

    std::clock_t start_declare3 = std::clock();
    int nout_result;
    std::vector < int > oatt1_rbuild(100000000);
    std::vector < int > oatt2_rlinenum(100000000);
    std::vector < int > oatt3_sprobe(100000000);
    std::vector < int > oatt4_slinenum(100000000);
    std::clock_t stop_declare3 = std::clock();

    std::clock_t start_wake_up_gpu4 = std::clock();
    // wake up gpu
    hipDeviceSynchronize();
    std::clock_t stop_wake_up_gpu4 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    std::clock_t start_cuda_malloc5 = std::clock();
    int* d_iatt1_rbuild;
    hipMalloc((void**) &d_iatt1_rbuild, 100000000* sizeof(int) );
    int* d_iatt2_rlinenum;
    hipMalloc((void**) &d_iatt2_rlinenum, 100000000* sizeof(int) );
    int* d_iatt3_sprobe;
  //hipMalloc((void**) &d_iatt3_sprobe, 3125000* sizeof(int) );
  //hipMalloc((void**) &d_iatt3_sprobe, 100000000* sizeof(int) );
    hipMalloc((void**) &d_iatt3_sprobe, 12500000* sizeof(int) );
    int* d_iatt4_slinenum;
  //hipMalloc((void**) &d_iatt4_slinenum, 3125000* sizeof(int) );
  //hipMalloc((void**) &d_iatt4_slinenum, 100000000* sizeof(int) );
    hipMalloc((void**) &d_iatt4_slinenum, 12500000* sizeof(int) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    int* d_oatt1_rbuild;
    hipMalloc((void**) &d_oatt1_rbuild, 100000000* sizeof(int) );
    int* d_oatt2_rlinenum;
    hipMalloc((void**) &d_oatt2_rlinenum, 100000000* sizeof(int) );
    int* d_oatt3_sprobe;
    hipMalloc((void**) &d_oatt3_sprobe, 100000000* sizeof(int) );
    int* d_oatt4_slinenum;
    hipMalloc((void**) &d_oatt4_slinenum, 100000000* sizeof(int) );
    hipDeviceSynchronize();
    std::clock_t stop_cuda_malloc5 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    std::clock_t start_cuda_mallocHT6 = std::clock();
    multi_ht* d_jht3;
    hipMalloc((void**) &d_jht3, 200000000* sizeof(multi_ht) );
    jpayl3* d_jht3_payload;
    hipMalloc((void**) &d_jht3_payload, 200000000* sizeof(jpayl3) );
    {
        int gridsize=920;
        int blocksize=128;
        initMultiHT<<<gridsize, blocksize>>>(d_jht3, 200000000);
    }
    int* d_offs3;
    hipMalloc((void**) &d_offs3, 1* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_offs3, 0, 1);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    std::clock_t stop_cuda_mallocHT6 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    std::clock_t start_cuda_memcpy_in7 = std::clock();
    hipMemcpy( d_iatt1_rbuild, iatt1_rbuild, 100000000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt2_rlinenum, iatt2_rlinenum, 100000000 * sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy( d_iatt3_sprobe, iatt3_sprobe, 3125000 * sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy( d_iatt3_sprobe, iatt3_sprobe, 100000000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt3_sprobe, iatt3_sprobe, 12500000 * sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy( d_iatt4_slinenum, iatt4_slinenum, 3125000 * sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy( d_iatt4_slinenum, iatt4_slinenum, 100000000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( d_iatt4_slinenum, iatt4_slinenum, 12500000 * sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy( d_iatt3_sprobe, iatt3_sprobe, 100000000 * sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy( d_iatt4_slinenum, iatt4_slinenum, 100000000 * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    std::clock_t stop_cuda_memcpy_in7 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime8 = std::clock();
    std::clock_t start_krnl_r_build1_920_1289 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_r_build1<<<gridsize, blocksize>>>(d_iatt1_rbuild, d_iatt2_rlinenum, d_jht3, d_jht3_payload);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_r_build1_920_1289 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_r_build1 920 128! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_r_build1 920 128")
        }
    }

    std::clock_t start_scanMultiHT_920_12810 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        scanMultiHT<<<gridsize, blocksize>>>(d_jht3, 200000000, d_offs3);
    }
    hipDeviceSynchronize();
    std::clock_t stop_scanMultiHT_920_12810 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in scanMultiHT 920 128! " << hipGetErrorString( err ) << std::endl;
            ERROR("scanMultiHT 920 128")
        }
    }

    std::clock_t start_krnl_r_build1_ins_920_12811 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_r_build1_ins<<<gridsize, blocksize>>>(d_iatt1_rbuild, d_iatt2_rlinenum, d_jht3, d_jht3_payload, d_offs3);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_r_build1_ins_920_12811 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_r_build1_ins 920 128! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_r_build1_ins 920 128")
        }
    }

    std::clock_t start_krnl_s_probe2_920_12812 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_s_probe2<<<gridsize, blocksize>>>(d_iatt3_sprobe, d_iatt4_slinenum, d_jht3, d_jht3_payload, d_nout_result, d_oatt1_rbuild, d_oatt2_rlinenum, d_oatt3_sprobe, d_oatt4_slinenum,d_iatt1_rbuild, d_iatt2_rlinenum);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_s_probe2_920_12812 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_s_probe2 920 128! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_s_probe2 920 128")
        }
    }

    std::clock_t stop_totalKernelTime8 = std::clock();
    std::clock_t start_cuda_memcpy_out13 = std::clock();
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt1_rbuild.data(), d_oatt1_rbuild, 100000000 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt2_rlinenum.data(), d_oatt2_rlinenum, 100000000 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt3_sprobe.data(), d_oatt3_sprobe, 100000000 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt4_slinenum.data(), d_oatt4_slinenum, 100000000 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    std::clock_t stop_cuda_memcpy_out13 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    std::clock_t start_cuda_free14 = std::clock();
    hipFree( d_iatt1_rbuild);
    hipFree( d_iatt2_rlinenum);
    hipFree( d_jht3);
    hipFree( d_jht3_payload);
    hipFree( d_offs3);
    hipFree( d_iatt3_sprobe);
    hipFree( d_iatt4_slinenum);
    hipFree( d_nout_result);
    hipFree( d_oatt1_rbuild);
    hipFree( d_oatt2_rlinenum);
    hipFree( d_oatt3_sprobe);
    hipFree( d_oatt4_slinenum);
    hipDeviceSynchronize();
    std::clock_t stop_cuda_free14 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish15 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 100000000)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("r_build: ");
        printf("%8i", oatt1_rbuild[pv]);
        printf("  ");
        printf("r_linenumber: ");
        printf("%8i", oatt2_rlinenum[pv]);
        printf("  ");
        printf("s_probe: ");
        printf("%8i", oatt3_sprobe[pv]);
        printf("  ");
        printf("s_linenumber: ");
        printf("%8i", oatt4_slinenum[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    //FILE* outFile;
    //outFile = fopen("queryresult.csv", "w");
    //fprintf(outFile, "r_build, ");
    //fprintf(outFile, "r_linenumber, ");
    //fprintf(outFile, "s_probe, ");
    //fprintf(outFile, "s_linenumber, ");
    //fprintf(outFile, "\n");
    //for ( int pv = 0; (pv < nout_result); pv += 1) {
    //    fprintf(outFile, "%8i  ", oatt1_rbuild[pv]);
    //    fprintf(outFile, "%8i  ", oatt2_rlinenum[pv]);
    //    fprintf(outFile, "%8i  ", oatt3_sprobe[pv]);
    //    fprintf(outFile, "%8i  ", oatt4_slinenum[pv]);
    //    fprintf(outFile, "\n");
    //}
    std::clock_t stop_finish15 = std::clock();

    std::cout << "import: " << (stop_import2 - start_import2) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "declare: " << (stop_declare3 - start_declare3) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "wake up gpu: " << (stop_wake_up_gpu4 - start_wake_up_gpu4) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "cuda malloc: " << (stop_cuda_malloc5 - start_cuda_malloc5) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "cuda mallocHT: " << (stop_cuda_mallocHT6 - start_cuda_mallocHT6) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "cuda memcpy in: " << (stop_cuda_memcpy_in7 - start_cuda_memcpy_in7) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "krnl_r_build1 920 128: " << (stop_krnl_r_build1_920_1289 - start_krnl_r_build1_920_1289) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "scanMultiHT 920 128: " << (stop_scanMultiHT_920_12810 - start_scanMultiHT_920_12810) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "krnl_r_build1_ins 920 128: " << (stop_krnl_r_build1_ins_920_12811 - start_krnl_r_build1_ins_920_12811) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "krnl_s_probe2 920 128: " << (stop_krnl_s_probe2_920_12812 - start_krnl_s_probe2_920_12812) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "cuda memcpy out: " << (stop_cuda_memcpy_out13 - start_cuda_memcpy_out13) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "cuda free: " << (stop_cuda_free14 - start_cuda_free14) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "finish: " << (stop_finish15 - start_finish15) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    std::cout << "totalKernelTime: " << (stop_totalKernelTime8 - start_totalKernelTime8) / (double) (CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
}
